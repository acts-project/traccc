#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <cuda/seeding/triplet_counting.cuh>
#include <cuda/utils/cuda_helper.cuh>

namespace traccc {
namespace cuda {

/// Forward declaration of triplet counting kernel
/// The number of triplets per mid-bot doublets are counted and recorded into
/// triplet counter container
///
/// @param config seed finder config
/// @param internal_sp_view vecmem container for internal spacepoint
/// @param doublet_count_view vecmem container for doublet_counter
/// @param mid_bot_doublet_container vecmem container for mid-bot doublets
/// @param mid_top_doublet_container vecmem container for mid-top doublets
/// @param triplet_counter_container vecmem container for triplet counters
/// @resource vecmem memory resource
__global__ void triplet_counting_kernel(
    const seedfinder_config config,
    internal_spacepoint_container_view internal_sp_view,
    doublet_counter_container_view doublet_counter_view,
    doublet_container_view mid_bot_doublet_view,
    doublet_container_view mid_top_doublet_view,
    triplet_counter_container_view triplet_counter_view);

void triplet_counting(const seedfinder_config& config,
                      host_internal_spacepoint_container& internal_sp_container,
                      host_doublet_counter_container& doublet_counter_container,
                      host_doublet_container& mid_bot_doublet_container,
                      host_doublet_container& mid_top_doublet_container,
                      host_triplet_counter_container& triplet_counter_container,
                      vecmem::memory_resource* resource) {
    auto internal_sp_view = get_data(internal_sp_container, resource);
    auto doublet_counter_container_view =
        get_data(doublet_counter_container, resource);
    auto mid_bot_doublet_view = get_data(mid_bot_doublet_container, resource);
    auto mid_top_doublet_view = get_data(mid_top_doublet_container, resource);
    auto triplet_counter_container_view =
        get_data(triplet_counter_container, resource);

    // The thread-block is desinged to make each thread count triplets per
    // middle-bot doublet

    // -- Num threads
    // The dimension of block is the integer multiple of WARP_SIZE (=32)
    unsigned int num_threads = WARP_SIZE * 8;

    // -- Num blocks
    // The dimension of grid is = sum_i{N_i}, where:
    // i is the spacepoint bin index
    // N_i is the number of blocks for i-th bin, defined as
    // num_mid_bot_doublet_per_bin / num_threads + 1
    unsigned int num_blocks = 0;
    for (size_t i = 0; i < internal_sp_view.headers.size(); ++i) {
        num_blocks += mid_bot_doublet_container.headers[i] / num_threads + 1;
    }

    // run the kernel
    triplet_counting_kernel<<<num_blocks, num_threads>>>(
        config, internal_sp_view, doublet_counter_container_view,
        mid_bot_doublet_view, mid_top_doublet_view,
        triplet_counter_container_view);

    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void triplet_counting_kernel(
    const seedfinder_config config,
    internal_spacepoint_container_view internal_sp_view,
    doublet_counter_container_view doublet_counter_view,
    doublet_container_view mid_bot_doublet_view,
    doublet_container_view mid_top_doublet_view,
    triplet_counter_container_view triplet_counter_view) {
    device_internal_spacepoint_container internal_sp_device(
        {internal_sp_view.headers, internal_sp_view.items});
    device_doublet_counter_container doublet_counter_device(
        {doublet_counter_view.headers, doublet_counter_view.items});
    device_doublet_container mid_bot_doublet_device(
        {mid_bot_doublet_view.headers, mid_bot_doublet_view.items});
    device_doublet_container mid_top_doublet_device(
        {mid_top_doublet_view.headers, mid_top_doublet_view.items});
    device_triplet_counter_container triplet_counter_device(
        {triplet_counter_view.headers, triplet_counter_view.items});

    // Get the bin index of spacepoint binning and reference block idx for the
    // bin index
    unsigned int bin_idx = 0;
    unsigned int ref_block_idx = 0;
    cuda_helper::get_header_idx(mid_bot_doublet_device, bin_idx, ref_block_idx);

    // Header of internal spacepoint container : spacepoint bin information
    // Item of internal spacepoint container : internal spacepoint objects per
    // bin
    auto internal_sp_per_bin = internal_sp_device.items.at(bin_idx);
    auto& num_compat_spM_per_bin = doublet_counter_device.headers.at(bin_idx);

    // Header of doublet counter : number of compatible middle sp per bin
    // Item of doublet counter : doublet counter objects per bin
    auto doublet_counter_per_bin = doublet_counter_device.items.at(bin_idx);

    // Header of doublet: number of mid_bot doublets per bin
    // Item of doublet: doublet objects per bin
    const auto& num_mid_bot_doublets_per_bin =
        mid_bot_doublet_device.headers.at(bin_idx);
    auto mid_bot_doublets_per_bin = mid_bot_doublet_device.items.at(bin_idx);

    // Header of doublet: number of mid_top doublets per bin
    // Item of doublet: doublet objects per bin
    const auto& num_mid_top_doublets_per_bin =
        mid_top_doublet_device.headers.at(bin_idx);
    auto mid_top_doublets_per_bin = mid_top_doublet_device.items.at(bin_idx);

    // Header of triplet counter: number of compatible mid_top doublets per bin
    // Item of triplet counter: triplet counter objects per bin
    auto& num_compat_mb_per_bin = triplet_counter_device.headers.at(bin_idx);
    auto triplet_counter_per_bin = triplet_counter_device.items.at(bin_idx);

    // index of middle-bot doublet in the item vector
    auto mb_idx = (blockIdx.x - ref_block_idx) * blockDim.x + threadIdx.x;

    // prevent the tail threads referring the null doublet counter
    if (mb_idx >= num_mid_bot_doublets_per_bin) {
        return;
    }

    // middle-bot doublet
    const auto& mid_bot_doublet = mid_bot_doublets_per_bin[mb_idx];
    // middle spacepoint index
    const auto& spM_idx = mid_bot_doublet.sp1.sp_idx;
    // middle spacepoint
    const auto& spM = internal_sp_per_bin[spM_idx];
    // bin index of bottom spacepoint
    const auto& spB_bin = mid_bot_doublet.sp2.bin_idx;
    // bottom spacepoint index
    const auto& spB_idx = mid_bot_doublet.sp2.sp_idx;
    // bottom spacepoint
    const auto& spB = internal_sp_device.items.at(spB_bin)[spB_idx];

    // Apply the conformal transformation to middle-bot doublet
    auto lb = doublet_finding_helper::transform_coordinates(spM, spB, true);

    // Calculate some physical quantities required for triplet compatibility
    // check
    scalar iSinTheta2 = 1 + lb.cotTheta() * lb.cotTheta();
    scalar scatteringInRegion2 = config.maxScatteringAngle2 * iSinTheta2;
    scatteringInRegion2 *= config.sigmaScattering * config.sigmaScattering;
    scalar curvature, impact_parameter;

    // find the reference (start) index of the mid-top doublet container item
    // vector, where the doublets are recorded The start index is calculated by
    // accumulating the number of mid-top doublets of all previous compatible
    // middle spacepoints
    unsigned int mb_end_idx = 0;
    unsigned int mt_start_idx = 0;
    unsigned int mt_end_idx = 0;

    for (unsigned int i = 0; i < num_compat_spM_per_bin; ++i) {
        mb_end_idx += doublet_counter_per_bin[i].n_mid_bot;
        mt_end_idx += doublet_counter_per_bin[i].n_mid_top;

        if (mb_end_idx > mb_idx) {
            break;
        }
        mt_start_idx += doublet_counter_per_bin[i].n_mid_top;
    }

    if (mt_end_idx >= mid_top_doublets_per_bin.size()) {
        mt_end_idx = fmin(mid_top_doublets_per_bin.size(), mt_end_idx);
    }

    if (mt_start_idx >= mid_top_doublets_per_bin.size()) {
        return;
    }

    // number of triplets per thread (or per middle-bot doublet)
    unsigned int num_triplets_per_mb = 0;

    // iterate over mid-top doublets
    for (unsigned int i = mt_start_idx; i < mt_end_idx; ++i) {
        const auto& mid_top_doublet = mid_top_doublets_per_bin[i];

        const auto& spT_bin = mid_top_doublet.sp2.bin_idx;
        const auto& spT_idx = mid_top_doublet.sp2.sp_idx;
        const auto& spT = internal_sp_device.items.at(spT_bin)[spT_idx];

        // Apply the conformal transformation to middle-top doublet
        auto lt =
            doublet_finding_helper::transform_coordinates(spM, spT, false);

        // Check if mid-bot and mid-top doublets can form a triplet
        if (triplet_finding_helper::isCompatible(
                spM, lb, lt, config, iSinTheta2, scatteringInRegion2, curvature,
                impact_parameter)) {
            num_triplets_per_mb++;
        }
    }

    // if the number of triplets per mb is larger than 0, write the triplet
    // counter into the container
    if (num_triplets_per_mb > 0) {
        auto pos = atomicAdd(&num_compat_mb_per_bin, 1);
        triplet_counter_per_bin[pos] = {mid_bot_doublet, num_triplets_per_mb};
    }
}

}  // namespace cuda
}  // namespace traccc
