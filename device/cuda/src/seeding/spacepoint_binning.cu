#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021-2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "traccc/cuda/seeding/spacepoint_binning.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// Project include(s).
#include "traccc/device/get_prefix_sum.hpp"
#include "traccc/seeding/device/count_grid_capacities.hpp"
#include "traccc/seeding/device/populate_grid.hpp"

// VecMem include(s).
#include <vecmem/utils/copy.hpp>
#include <vecmem/utils/cuda/copy.hpp>

namespace traccc::cuda {
namespace kernels {

/// CUDA kernel for running @c traccc::device::count_grid_capacities
__global__ void count_grid_capacities(
    seedfinder_config config, sp_grid::axis_p0_type phi_axis,
    sp_grid::axis_p1_type z_axis,
    spacepoint_container_types::const_view spacepoints,
    vecmem::data::vector_view<const device::prefix_sum_element_t> sp_prefix_sum,
    vecmem::data::vector_view<unsigned int> grid_capacities) {

    device::count_grid_capacities(threadIdx.x + blockIdx.x * blockDim.x, config,
                                  phi_axis, z_axis, spacepoints, sp_prefix_sum,
                                  grid_capacities);
}

/// CUDA kernel for running @c traccc::device::populate_grid
__global__ void populate_grid(
    seedfinder_config config,
    spacepoint_container_types::const_view spacepoints,
    vecmem::data::vector_view<const device::prefix_sum_element_t> sp_prefix_sum,
    sp_grid_view grid) {

    device::populate_grid(threadIdx.x + blockIdx.x * blockDim.x, config,
                          spacepoints, sp_prefix_sum, grid);
}

}  // namespace kernels

spacepoint_binning::spacepoint_binning(
    const seedfinder_config& config, const spacepoint_grid_config& grid_config,
    const traccc::memory_resource& mr)
    : m_config(config.toInternalUnits()),
      m_axes(get_axes(grid_config.toInternalUnits(),
                      (mr.host ? *(mr.host) : mr.main))),
      m_mr(mr) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

sp_grid_buffer spacepoint_binning::operator()(
    const spacepoint_container_types::const_view& spacepoints_view) const {

    // Get the spacepoint sizes from the view
    auto sp_sizes = m_copy->get_sizes(spacepoints_view.items);

    return this->operator()(spacepoints_view, sp_sizes);
}

sp_grid_buffer spacepoint_binning::operator()(
    const spacepoint_container_types::buffer& spacepoints_buffer) const {

    // Get the spacepoint sizes from the buffer
    auto sp_sizes = m_copy->get_sizes(spacepoints_buffer.items);

    return this->operator()(spacepoints_buffer, sp_sizes);
}

sp_grid_buffer spacepoint_binning::operator()(
    const spacepoint_container_types::const_view& spacepoints_view,
    const std::vector<unsigned int>& sp_sizes) const {

    // Get the prefix sum for the spacepoints using buffer.
    const device::prefix_sum_t sp_prefix_sum = device::get_prefix_sum(
        sp_sizes, (m_mr.host ? *(m_mr.host) : m_mr.main));

    // Set up the buffer of the prefix sum and its view
    vecmem::data::vector_buffer<device::prefix_sum_element_t>
        sp_prefix_sum_buff(sp_prefix_sum.size(), m_mr.main);
    m_copy->setup(sp_prefix_sum_buff);
    (*m_copy)(vecmem::get_data(sp_prefix_sum), sp_prefix_sum_buff,
              vecmem::copy::type::copy_type::host_to_device);
    vecmem::data::vector_view<device::prefix_sum_element_t> sp_prefix_sum_view =
        sp_prefix_sum_buff;

    // Set up the container that will be filled with the required capacities for
    // the spacepoint grid.
    const std::size_t grid_bins = m_axes.first.n_bins * m_axes.second.n_bins;
    vecmem::data::vector_buffer<unsigned int> grid_capacities_buff(grid_bins,
                                                                   m_mr.main);
    m_copy->setup(grid_capacities_buff);
    m_copy->memset(grid_capacities_buff, 0);
    vecmem::data::vector_view<unsigned int> grid_capacities_view =
        grid_capacities_buff;

    // Calculate the number of threads and thread blocks to run the kernels for.
    const unsigned int num_threads = WARP_SIZE * 8;
    const unsigned int num_blocks = sp_prefix_sum.size() / num_threads + 1;

    // Fill the grid capacity container.
    kernels::count_grid_capacities<<<num_blocks, num_threads>>>(
        m_config, m_axes.first, m_axes.second, spacepoints_view,
        sp_prefix_sum_view, grid_capacities_view);
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Copy grid capacities back to the host
    vecmem::vector<unsigned int> grid_capacities_host(m_mr.host ? m_mr.host
                                                                : &(m_mr.main));
    (*m_copy)(grid_capacities_buff, grid_capacities_host);

    // Create the grid buffer.
    sp_grid_buffer grid_buffer(
        m_axes.first, m_axes.second, std::vector<std::size_t>(grid_bins, 0),
        std::vector<std::size_t>(grid_capacities_host.begin(),
                                 grid_capacities_host.end()),
        m_mr.main, m_mr.host);
    m_copy->setup(grid_buffer._buffer);
    sp_grid_view grid_view = grid_buffer;

    // Populate the grid.
    kernels::populate_grid<<<num_blocks, num_threads>>>(
        m_config, spacepoints_view, sp_prefix_sum_view, grid_view);
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Return the freshly filled buffer.
    return grid_buffer;
}

}  // namespace traccc::cuda
