#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <cuda/seeding/weight_updating.cuh>
#include <cuda/utils/cuda_helper.cuh>
#include <cuda/utils/definitions.hpp>

namespace traccc {
namespace cuda {

/// Forward declaration of weight updating kernel
/// The weight of triplets are updated by iterating over triplets which share
/// the same middle spacepoint
///
/// @param config seed finder config
/// @param internal_sp_container vecmem container for internal spacepoint
/// @param triplet_counter_container vecmem container for triplet counters
/// @param triplet_container vecmem container for triplets
__global__ void weight_updating_kernel(
    const seedfilter_config filter_config,
    internal_spacepoint_container_view internal_sp_view,
    triplet_counter_container_view triplet_counter_view,
    triplet_container_view triplet_view);

void weight_updating(const seedfilter_config& filter_config,
                     host_internal_spacepoint_container& internal_sp_container,
                     host_triplet_counter_container& triplet_counter_container,
                     host_triplet_container& triplet_container,
                     vecmem::memory_resource* resource) {
    auto internal_sp_view = get_data(internal_sp_container, resource);
    auto triplet_counter_view = get_data(triplet_counter_container, resource);
    auto triplet_view = get_data(triplet_container, resource);

    // The thread-block is desinged to make each thread update the weight of eac
    // triplet

    // -- Num threads
    // The dimension of block is the integer multiple of WARP_SIZE (=32)
    unsigned int num_threads = WARP_SIZE * 2;

    // -- Num blocks
    // The dimension of grid is = sum_i{N_i}, where:
    // i is the spacepoint bin index
    // N_i is the number of blocks for i-th bin, defined as num_triplets_per_bin
    // / num_threads + 1
    unsigned int num_blocks = 0;
    for (size_t i = 0; i < internal_sp_view.headers.size(); ++i) {
        num_blocks += triplet_container.headers[i] / num_threads + 1;
    }

    // shared memory assignment for the radius of the compatible top spacepoints
    unsigned int sh_mem = sizeof(float) * filter_config.compatSeedLimit;

    // run the kernel
    weight_updating_kernel<<<num_blocks, num_threads, sh_mem>>>(
        filter_config, internal_sp_view, triplet_counter_view, triplet_view);

    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void weight_updating_kernel(
    const seedfilter_config filter_config,
    internal_spacepoint_container_view internal_sp_view,
    triplet_counter_container_view triplet_counter_view,
    triplet_container_view triplet_view) {
    device_internal_spacepoint_container internal_sp_device(
        {internal_sp_view.headers, internal_sp_view.items});

    device_triplet_counter_container triplet_counter_device(
        {triplet_counter_view.headers, triplet_counter_view.items});
    device_triplet_container triplet_device(
        {triplet_view.headers, triplet_view.items});

    // Get the bin index of spacepoint binning and reference block idx for the
    // bin index
    unsigned int bin_idx = 0;
    unsigned int ref_block_idx = 0;
    cuda_helper::get_header_idx(triplet_device, bin_idx, ref_block_idx);

    // Header of internal spacepoint container : spacepoint bin information
    // Item of internal spacepoint container : internal spacepoint objects per
    // bin
    auto internal_sp_per_bin = internal_sp_device.items.at(bin_idx);

    // Header of triplet counter: number of compatible mid_top doublets per bin
    // Item of triplet counter: triplet counter objects per bin
    auto& num_compat_mb_per_bin = triplet_counter_device.headers.at(bin_idx);
    auto triplet_counter_per_bin = triplet_counter_device.items.at(bin_idx);

    // Header of triplet: number of triplets per bin
    // Item of triplet: triplet objects per bin
    auto& num_triplets_per_bin = triplet_device.headers.at(bin_idx);
    auto triplets_per_bin = triplet_device.items.at(bin_idx);

    extern __shared__ float compat_seedR[];
    __syncthreads();

    // index of triplet in the item vector
    auto tr_idx = (blockIdx.x - ref_block_idx) * blockDim.x + threadIdx.x;
    auto& triplet = triplets_per_bin[tr_idx];
    auto& spB_idx = triplet.sp1;
    auto& spM_idx = triplet.sp2;
    auto& spT_idx = triplet.sp3;

    // prevent overflow
    if (tr_idx >= num_triplets_per_bin) {
        return;
    }

    // find the reference index (start and end) of the triplet container item
    // vector
    size_t start_idx = 0;
    size_t end_idx = 0;

    for (auto triplet_counter : triplet_counter_per_bin) {
        end_idx += triplet_counter.n_triplets;

        if (triplet_counter.mid_bot_doublet.sp1 == spM_idx &&
            triplet_counter.mid_bot_doublet.sp2 == spB_idx) {
            break;
        }

        start_idx += triplet_counter.n_triplets;
    }

    if (end_idx >= triplets_per_bin.size()) {
        end_idx = fmin(triplets_per_bin.size(), end_idx);
    }

    // prevent overflow
    if (start_idx >= triplets_per_bin.size()) {
        return;
    }

    auto& current_spT =
        internal_sp_device.items[spT_idx.bin_idx][spT_idx.sp_idx];

    float currentTop_r = current_spT.radius();

    // if two compatible seeds with high distance in r are found, compatible
    // seeds span 5 layers
    // -> very good seed
    float lowerLimitCurv =
        triplet.curvature - filter_config.deltaInvHelixDiameter;
    float upperLimitCurv =
        triplet.curvature + filter_config.deltaInvHelixDiameter;
    int num_compat_seedR = 0;

    // iterate over triplets
    for (auto tr_it = triplets_per_bin.begin() + start_idx;
         tr_it != triplets_per_bin.begin() + end_idx; tr_it++) {
        if (triplet == *tr_it) {
            continue;
        }

        auto& other_triplet = *tr_it;
        auto other_spT_idx = (*tr_it).sp3;
        auto other_spT =
            internal_sp_device
                .items[other_spT_idx.bin_idx][other_spT_idx.sp_idx];

        // compared top SP should have at least deltaRMin distance
        float otherTop_r = other_spT.radius();
        float deltaR = currentTop_r - otherTop_r;
        if (std::abs(deltaR) < filter_config.deltaRMin) {
            continue;
        }

        // curvature difference within limits?
        // TODO: how much slower than sorting all vectors by curvature
        // and breaking out of loop? i.e. is vector size large (e.g. in
        // jets?)
        if (other_triplet.curvature < lowerLimitCurv) {
            continue;
        }
        if (other_triplet.curvature > upperLimitCurv) {
            continue;
        }

        bool newCompSeed = true;

        for (size_t i_s = 0; i_s < num_compat_seedR; ++i_s) {
            float previousDiameter = compat_seedR[i_s];

            // original ATLAS code uses higher min distance for 2nd found
            // compatible seed (20mm instead of 5mm) add new compatible seed
            // only if distance larger than rmin to all other compatible
            // seeds
            if (std::abs(previousDiameter - otherTop_r) <
                filter_config.deltaRMin) {
                newCompSeed = false;
                break;
            }
        }

        if (newCompSeed) {
            compat_seedR[num_compat_seedR] = otherTop_r;
            triplet.weight += filter_config.compatSeedWeight;
            num_compat_seedR++;
        }

        if (num_compat_seedR >= filter_config.compatSeedLimit) {
            break;
        }
    }
}

}  // namespace cuda
}  // namespace traccc
