#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021-2023 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../utils/utils.hpp"
#include "traccc/cuda/seeding/seed_finding.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// Project include(s).
#include "traccc/cuda/utils/make_prefix_sum_buff.hpp"
#include "traccc/device/fill_prefix_sum.hpp"
#include "traccc/device/make_prefix_sum_buffer.hpp"
#include "traccc/edm/device/device_doublet.hpp"
#include "traccc/edm/device/device_triplet.hpp"
#include "traccc/edm/device/doublet_counter.hpp"
#include "traccc/edm/device/seeding_global_counter.hpp"
#include "traccc/edm/device/triplet_counter.hpp"
#include "traccc/seeding/device/count_doublets.hpp"
#include "traccc/seeding/device/count_triplets.hpp"
#include "traccc/seeding/device/find_doublets.hpp"
#include "traccc/seeding/device/find_triplets.hpp"
#include "traccc/seeding/device/reduce_triplet_counts.hpp"
#include "traccc/seeding/device/select_seeds.hpp"
#include "traccc/seeding/device/update_triplet_weights.hpp"

// VecMem include(s).
#include "vecmem/utils/cuda/copy.hpp"

// System include(s).
#include <algorithm>
#include <vector>

namespace traccc::cuda {
namespace kernels {

/// CUDA kernel for running @c traccc::device::count_doublets
__global__ void count_doublets(
    seedfinder_config config, sp_grid_const_view sp_grid,
    vecmem::data::vector_view<const device::prefix_sum_element_t> sp_prefix_sum,
    device::doublet_counter_collection_types::view doublet_counter,
    unsigned int& nMidBot, unsigned int& nMidTop) {

    device::count_doublets(threadIdx.x + blockIdx.x * blockDim.x, config,
                           sp_grid, sp_prefix_sum, doublet_counter, nMidBot,
                           nMidTop);
}

/// CUDA kernel for running @c traccc::device::find_doublets
__global__ void find_doublets(
    seedfinder_config config, sp_grid_const_view sp_grid,
    device::doublet_counter_collection_types::const_view doublet_counter,
    device::device_doublet_collection_types::view mb_doublets,
    device::device_doublet_collection_types::view mt_doublets) {

    device::find_doublets(threadIdx.x + blockIdx.x * blockDim.x, config,
                          sp_grid, doublet_counter, mb_doublets, mt_doublets);
}

/// CUDA kernel for running @c traccc::device::count_triplets
__global__ void count_triplets(
    seedfinder_config config, sp_grid_const_view sp_grid,
    device::doublet_counter_collection_types::const_view doublet_counter,
    device::device_doublet_collection_types::const_view mb_doublets,
    device::device_doublet_collection_types::const_view mt_doublets,
    device::triplet_counter_spM_collection_types::view spM_counter,
    device::triplet_counter_collection_types::view midBot_counter) {

    device::count_triplets(threadIdx.x + blockIdx.x * blockDim.x, config,
                           sp_grid, doublet_counter, mb_doublets, mt_doublets,
                           spM_counter, midBot_counter);
}

/// CUDA kernel for running @c traccc::device::reduce_triplet_counts
__global__ void reduce_triplet_counts(
    device::doublet_counter_collection_types::const_view doublet_counter,
    device::triplet_counter_spM_collection_types::view spM_counter,
    unsigned int& num_triplets) {

    device::reduce_triplet_counts(threadIdx.x + blockIdx.x * blockDim.x,
                                  doublet_counter, spM_counter, num_triplets);
}

/// CUDA kernel for running @c traccc::device::find_triplets
__global__ void find_triplets(
    seedfinder_config config, seedfilter_config filter_config,
    sp_grid_const_view sp_grid,
    device::doublet_counter_collection_types::const_view doublet_counter,
    device::device_doublet_collection_types::const_view mt_doublets,
    device::triplet_counter_spM_collection_types::const_view spM_tc,
    device::triplet_counter_collection_types::const_view midBot_tc,
    device::device_triplet_collection_types::view triplet_view) {

    device::find_triplets(threadIdx.x + blockIdx.x * blockDim.x, config,
                          filter_config, sp_grid, doublet_counter, mt_doublets,
                          spM_tc, midBot_tc, triplet_view);
}

/// CUDA kernel for running @c traccc::device::update_triplet_weights
__global__ void update_triplet_weights(
    seedfilter_config filter_config, sp_grid_const_view sp_grid,
    device::triplet_counter_spM_collection_types::const_view spM_tc,
    device::triplet_counter_collection_types::const_view midBot_tc,
    device::device_triplet_collection_types::view triplet_view) {

    // Array for temporary storage of quality parameters for comparing triplets
    // within weight updating kernel
    extern __shared__ scalar data[];
    // Each thread uses compatSeedLimit elements of the array
    scalar* dataPos = &data[threadIdx.x * filter_config.compatSeedLimit];

    device::update_triplet_weights(threadIdx.x + blockIdx.x * blockDim.x,
                                   filter_config, sp_grid, spM_tc, midBot_tc,
                                   dataPos, triplet_view);
}

/// CUDA kernel for running @c traccc::device::select_seeds
__global__ void select_seeds(
    seedfilter_config filter_config,
    spacepoint_collection_types::const_view spacepoints_view,
    sp_grid_const_view internal_sp_view,
    device::triplet_counter_spM_collection_types::const_view spM_tc,
    device::triplet_counter_collection_types::const_view midBot_tc,
    device::device_triplet_collection_types::view triplet_view,
    seed_collection_types::view seed_view) {

    // Array for temporary storage of triplets for comparing within seed
    // selecting kernel
    extern __shared__ triplet data2[];
    // Each thread uses max_triplets_per_spM elements of the array
    triplet* dataPos = &data2[threadIdx.x * filter_config.max_triplets_per_spM];

    device::select_seeds(threadIdx.x + blockIdx.x * blockDim.x, filter_config,
                         spacepoints_view, internal_sp_view, spM_tc, midBot_tc,
                         triplet_view, dataPos, seed_view);
}

}  // namespace kernels

seed_finding::seed_finding(const seedfinder_config& config,
                           const seedfilter_config& filter_config,
                           const traccc::memory_resource& mr,
                           vecmem::copy& copy, stream& str)
    : m_seedfinder_config(config),
      m_seedfilter_config(filter_config),
      m_mr(mr),
      m_copy(copy),
      m_stream(str) {}

seed_finding::output_type seed_finding::operator()(
    const spacepoint_collection_types::const_view& spacepoints_view,
    const sp_grid_const_view& g2_view) const {

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // Get the sizes from the grid view
    auto grid_sizes = m_copy.get_sizes(g2_view._data_view);

    // Create prefix sum buffer
    vecmem::data::vector_buffer sp_grid_prefix_sum_buff =
        make_prefix_sum_buff(grid_sizes, m_copy, m_mr, m_stream);

    // Set up the doublet counter buffer.
    device::doublet_counter_collection_types::buffer doublet_counter_buffer = {
        m_copy.get_size(sp_grid_prefix_sum_buff), m_mr.main,
        vecmem::data::buffer_type::resizable};
    m_copy.setup(doublet_counter_buffer);

    // Calculate the number of threads and thread blocks to run the doublet
    // counting kernel for.
    const unsigned int nDoubletCountThreads = WARP_SIZE * 2;
    const unsigned int nDoubletCountBlocks =
        (m_copy.get_size(sp_grid_prefix_sum_buff) + nDoubletCountThreads - 1) /
        nDoubletCountThreads;

    // Counter for the total number of doublets and triplets
    vecmem::unique_alloc_ptr<device::seeding_global_counter>
        globalCounter_device =
            vecmem::make_unique_alloc<device::seeding_global_counter>(
                m_mr.main);
    CUDA_ERROR_CHECK(hipMemsetAsync(globalCounter_device.get(), 0,
                                     sizeof(device::seeding_global_counter),
                                     stream));

    // Count the number of doublets that we need to produce.
    kernels::count_doublets<<<nDoubletCountBlocks, nDoubletCountThreads, 0,
                              stream>>>(
        m_seedfinder_config, g2_view, sp_grid_prefix_sum_buff,
        doublet_counter_buffer, (*globalCounter_device).m_nMidBot,
        (*globalCounter_device).m_nMidTop);
    CUDA_ERROR_CHECK(hipGetLastError());

    // Get the summary values.
    vecmem::unique_alloc_ptr<device::seeding_global_counter>
        globalCounter_host =
            vecmem::make_unique_alloc<device::seeding_global_counter>(
                (m_mr.host != nullptr) ? *(m_mr.host) : m_mr.main);
    CUDA_ERROR_CHECK(hipMemcpyAsync(globalCounter_host.get(),
                                     globalCounter_device.get(),
                                     sizeof(device::seeding_global_counter),
                                     hipMemcpyDeviceToHost, stream));
    m_stream.synchronize();

    // Set up the doublet counter buffers.
    device::device_doublet_collection_types::buffer doublet_buffer_mb = {
        globalCounter_host->m_nMidBot, m_mr.main};
    m_copy.setup(doublet_buffer_mb);
    device::device_doublet_collection_types::buffer doublet_buffer_mt = {
        globalCounter_host->m_nMidTop, m_mr.main};
    m_copy.setup(doublet_buffer_mt);

    // Calculate the number of threads and thread blocks to run the doublet
    // finding kernel for.
    const unsigned int nDoubletFindThreads = WARP_SIZE * 2;
    const unsigned int doublet_counter_buffer_size =
        m_copy.get_size(doublet_counter_buffer);
    const unsigned int nDoubletFindBlocks =
        (doublet_counter_buffer_size + nDoubletFindThreads - 1) /
        nDoubletFindThreads;

    // Find all of the spacepoint doublets.
    kernels::
        find_doublets<<<nDoubletFindBlocks, nDoubletFindThreads, 0, stream>>>(
            m_seedfinder_config, g2_view, doublet_counter_buffer,
            doublet_buffer_mb, doublet_buffer_mt);
    CUDA_ERROR_CHECK(hipGetLastError());

    // Set up the triplet counter buffers
    device::triplet_counter_spM_collection_types::buffer
        triplet_counter_spM_buffer = {doublet_counter_buffer_size, m_mr.main};
    m_copy.setup(triplet_counter_spM_buffer);
    m_copy.memset(triplet_counter_spM_buffer, 0);
    device::triplet_counter_collection_types::buffer
        triplet_counter_midBot_buffer = {globalCounter_host->m_nMidBot,
                                         m_mr.main,
                                         vecmem::data::buffer_type::resizable};
    m_copy.setup(triplet_counter_midBot_buffer);

    // Calculate the number of threads and thread blocks to run the doublet
    // counting kernel for.
    const unsigned int nTripletCountThreads = WARP_SIZE * 2;
    const unsigned int nTripletCountBlocks =
        (globalCounter_host->m_nMidBot + nTripletCountThreads - 1) /
        nTripletCountThreads;

    // Count the number of triplets that we need to produce.
    kernels::count_triplets<<<nTripletCountBlocks, nTripletCountThreads, 0,
                              stream>>>(
        m_seedfinder_config, g2_view, doublet_counter_buffer, doublet_buffer_mb,
        doublet_buffer_mt, triplet_counter_spM_buffer,
        triplet_counter_midBot_buffer);
    CUDA_ERROR_CHECK(hipGetLastError());

    // Calculate the number of threads and thread blocks to run the triplet
    // count reduction kernel for.
    const unsigned int nTcReductionThreads = WARP_SIZE * 2;
    const unsigned int nTcReductionBlocks =
        (doublet_counter_buffer_size + nTcReductionThreads - 1) /
        nTcReductionThreads;

    // Reduce the triplet counts per spM.
    kernels::reduce_triplet_counts<<<nTcReductionBlocks, nTcReductionThreads, 0,
                                     stream>>>(
        doublet_counter_buffer, triplet_counter_spM_buffer,
        (*globalCounter_device).m_nTriplets);
    CUDA_ERROR_CHECK(hipGetLastError());

    CUDA_ERROR_CHECK(hipMemcpyAsync(globalCounter_host.get(),
                                     globalCounter_device.get(),
                                     sizeof(device::seeding_global_counter),
                                     hipMemcpyDeviceToHost, stream));
    m_stream.synchronize();

    // Set up the triplet buffer.
    device::device_triplet_collection_types::buffer triplet_buffer = {
        globalCounter_host->m_nTriplets, m_mr.main};
    m_copy.setup(triplet_buffer);

    // Calculate the number of threads and thread blocks to run the triplet
    // finding kernel for.
    const unsigned int nTripletFindThreads = WARP_SIZE * 2;
    const unsigned int nTripletFindBlocks =
        (m_copy.get_size(triplet_counter_midBot_buffer) + nTripletFindThreads -
         1) /
        nTripletFindThreads;

    // Find all of the spacepoint triplets.
    kernels::
        find_triplets<<<nTripletFindBlocks, nTripletFindThreads, 0, stream>>>(
            m_seedfinder_config, m_seedfilter_config, g2_view,
            doublet_counter_buffer, doublet_buffer_mt,
            triplet_counter_spM_buffer, triplet_counter_midBot_buffer,
            triplet_buffer);
    CUDA_ERROR_CHECK(hipGetLastError());

    // Calculate the number of threads and thread blocks to run the weight
    // updating kernel for.
    const unsigned int nWeightUpdatingThreads = WARP_SIZE * 2;
    const unsigned int nWeightUpdatingBlocks =
        (globalCounter_host->m_nTriplets + nWeightUpdatingThreads - 1) /
        nWeightUpdatingThreads;

    // Update the weights of all spacepoint triplets.
    kernels::update_triplet_weights<<<
        nWeightUpdatingBlocks, nWeightUpdatingThreads,
        sizeof(scalar) * m_seedfilter_config.compatSeedLimit *
            nWeightUpdatingThreads,
        stream>>>(m_seedfilter_config, g2_view, triplet_counter_spM_buffer,
                  triplet_counter_midBot_buffer, triplet_buffer);
    CUDA_ERROR_CHECK(hipGetLastError());

    // Create result object: collection of seeds
    seed_collection_types::buffer seed_buffer(
        globalCounter_host->m_nTriplets, m_mr.main,
        vecmem::data::buffer_type::resizable);
    m_copy.setup(seed_buffer);

    // Calculate the number of threads and thread blocks to run the seed
    // selecting kernel for.
    const unsigned int nSeedSelectingThreads = WARP_SIZE * 2;
    const unsigned int nSeedSelectingBlocks =
        (doublet_counter_buffer_size + nSeedSelectingThreads - 1) /
        nSeedSelectingThreads;

    // Create seeds out of selected triplets
    kernels::select_seeds<<<nSeedSelectingBlocks, nSeedSelectingThreads,
                            sizeof(triplet) *
                                m_seedfilter_config.max_triplets_per_spM *
                                nSeedSelectingThreads,
                            stream>>>(m_seedfilter_config, spacepoints_view,
                                      g2_view, triplet_counter_spM_buffer,
                                      triplet_counter_midBot_buffer,
                                      triplet_buffer, seed_buffer);
    CUDA_ERROR_CHECK(hipGetLastError());

    return seed_buffer;
}

}  // namespace traccc::cuda
