#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021-2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "traccc/cuda/seeding/counting_grid_capacities.hpp"
#include "traccc/cuda/utils/definitions.hpp"

namespace traccc {
namespace cuda {

__global__ void counting_grid_capacities_kernel(
    const seedfinder_config config, const sp_grid_buffer::axis_p0_type phi_axis,
    const sp_grid_buffer::axis_p1_type z_axis,
    spacepoint_container_view spacepoints_view,
    vecmem::data::vector_view<std::pair<unsigned int, unsigned int>>
        sp_container_indices_view,
    vecmem::data::vector_view<unsigned int> grid_capacities_view);

void counting_grid_capacities(
    const seedfinder_config config, const sp_grid_buffer::axis_p0_type phi_axis,
    const sp_grid_buffer::axis_p1_type z_axis,
    host_spacepoint_container& spacepoints,
    vecmem::vector<std::pair<unsigned int, unsigned int>>& sp_container_indices,
    vecmem::vector<unsigned int>& grid_capacities,
    vecmem::memory_resource& resource) {

    auto spacepoints_view = get_data(spacepoints, &resource);
    auto sp_container_indices_view = vecmem::get_data(sp_container_indices);
    auto grid_capacities_view = vecmem::get_data(grid_capacities);

    // number of threads is the integer multiple of WARP_SIZE (=32)
    unsigned int num_threads = WARP_SIZE * 8;
    unsigned int num_blocks = spacepoints.total_size() / num_threads + 1;

    // run the kernel
    counting_grid_capacities_kernel<<<num_blocks, num_threads>>>(
        config, phi_axis, z_axis, spacepoints_view, sp_container_indices_view,
        grid_capacities_view);

    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void counting_grid_capacities_kernel(
    const seedfinder_config config, const sp_grid_buffer::axis_p0_type phi_axis,
    const sp_grid_buffer::axis_p1_type z_axis,
    spacepoint_container_view spacepoints_view,
    vecmem::data::vector_view<std::pair<unsigned int, unsigned int>>
        sp_container_indices_view,
    vecmem::data::vector_view<unsigned int> grid_capacities_view) {

    // Get device container for input parameters
    device_spacepoint_container spacepoints_device(
        {spacepoints_view.headers, spacepoints_view.items});
    vecmem::device_vector<std::pair<unsigned int, unsigned int>>
        sp_container_indices(sp_container_indices_view);
    vecmem::device_vector<unsigned int> grid_capacities_device(
        grid_capacities_view);

    auto gid = threadIdx.x + blockIdx.x * blockDim.x;

    /// kill the process before overflow
    if (gid >= sp_container_indices.size()) {
        return;
    }

    const auto& header_idx = sp_container_indices[gid].first;
    const auto& sp_idx = sp_container_indices[gid].second;

    auto spacepoints_per_module = spacepoints_device.get_items().at(header_idx);
    const auto& sp = spacepoints_per_module[sp_idx];

    /// Check out if the spacepoints can be used for seeding
    size_t r_index = is_valid_sp(config, sp);

    /// Ignore is radius index is invalid value
    if (r_index != detray::invalid_value<size_t>()) {

        auto isp = internal_spacepoint<spacepoint>(
            spacepoints_device, {header_idx, sp_idx}, config.beamPos);

        /// Get bin index in grid
        size_t bin_index =
            phi_axis.bin(isp.phi()) + phi_axis.bins() * z_axis.bin(isp.z());

        /// increase the capacity for the grid bin
        atomicAdd(&grid_capacities_device[bin_index], 1);
    }
}

}  // namespace cuda
}  // namespace traccc
