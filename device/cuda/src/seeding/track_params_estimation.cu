#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021-2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "traccc/cuda/seeding/track_params_estimation.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// VecMem include(s).
#include <vecmem/utils/cuda/copy.hpp>

namespace traccc {
namespace cuda {

/// Forward declaration of track parameter estimating kernel
/// The bound track parameters at the bottom spacepoints are obtained
///
/// @param seeds_view seeds found by seed finding
/// @param params_view vector of bound track parameters at the bottom
/// spacepoints
__global__ void track_params_estimating_kernel(
    spacepoint_container_types::const_view spacepoints_view,
    vecmem::data::vector_view<const seed> seeds_view,
    vecmem::data::vector_view<bound_track_parameters> params_view);

track_params_estimation::track_params_estimation(
    const traccc::memory_resource& mr)
    : m_mr(mr) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

host_bound_track_parameters_collection track_params_estimation::operator()(
    const spacepoint_container_types::const_view& spacepoints_view,
    const vecmem::data::vector_view<const seed>& seeds_view) const {

    // Get the size of the seeds view
    auto seeds_size = m_copy->get_size(seeds_view);

    return this->operator()(spacepoints_view, seeds_view, seeds_size);
}

host_bound_track_parameters_collection track_params_estimation::operator()(
    const spacepoint_container_types::buffer& spacepoints_buffer,
    const vecmem::data::vector_buffer<seed>& seeds_buffer) const {

    // Get the size of the seeds buffer
    auto seeds_size = m_copy->get_size(seeds_buffer);

    return this->operator()(spacepoints_buffer, seeds_buffer, seeds_size);
}

host_bound_track_parameters_collection track_params_estimation::operator()(
    const spacepoint_container_types::const_view& spacepoints_view,
    const vecmem::data::vector_view<const seed>& seeds_view,
    std::size_t seeds_size) const {

    // Create output host container
    host_bound_track_parameters_collection params(
        seeds_size, (m_mr.host ? m_mr.host : &(m_mr.main)));

    // Check if anything needs to be done.
    if (seeds_size == 0) {
        return params;
    }

    // Create device buffer for the parameters
    vecmem::data::vector_buffer<bound_track_parameters> params_buffer(
        seeds_size, m_mr.main);
    m_copy->setup(params_buffer);

    // -- Num threads
    // The dimension of block is the integer multiple of WARP_SIZE (=32)
    unsigned int num_threads = WARP_SIZE * 2;

    // -- Num blocks
    // The dimension of grid is number_of_seeds / num_threads + 1
    unsigned int num_blocks = seeds_size / num_threads + 1;

    // run the kernel
    track_params_estimating_kernel<<<num_blocks, num_threads>>>(
        spacepoints_view, seeds_view, params_buffer);

    // cuda error check
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Copy the results back to the host
    (*m_copy)(params_buffer, params);

    return params;
}

__global__ void track_params_estimating_kernel(
    spacepoint_container_types::const_view spacepoints_view,
    vecmem::data::vector_view<const seed> seeds_view,
    vecmem::data::vector_view<bound_track_parameters> params_view) {

    // Get device container for input parameters
    const spacepoint_container_types::const_device spacepoints_device(
        spacepoints_view);
    vecmem::device_vector<const seed> seeds_device(seeds_view);
    device_bound_track_parameters_collection params_device(params_view);

    // vector index for threads
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // prevent overflow
    if (gid >= seeds_device.size()) {
        return;
    }

    // convenient assumption on bfield and mass
    // TODO: make use of bfield extension for the future
    vector3 bfield = {0, 0, 2};

    const auto& seed = seeds_device.at(gid);
    auto& param = params_device[gid].vector();

    // Get bound track parameter
    param =
        seed_to_bound_vector(spacepoints_device, seed, bfield, PION_MASS_MEV);
}

}  // namespace cuda
}  // namespace traccc
