#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2024-2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../utils/cuda_error_handling.hpp"
#include "../utils/get_size.hpp"
#include "../utils/global_index.hpp"
#include "../utils/utils.hpp"
#include "traccc/cuda/seeding/spacepoint_formation_algorithm.hpp"

// Project include(s).
#include "traccc/geometry/detector.hpp"
#include "traccc/seeding/device/form_spacepoints.hpp"

namespace traccc::cuda {
namespace kernels {

template <typename detector_t>
__global__ void __launch_bounds__(1024, 1)
    form_spacepoints(typename detector_t::view det_view,
                     measurement_collection_types::const_view measurements_view,
                     edm::spacepoint_collection::view spacepoints_view)
    requires(traccc::is_detector_traits<detector_t>)
{

    device::form_spacepoints<detector_t>(details::global_index1(), det_view,
                                         measurements_view, spacepoints_view);
}

}  // namespace kernels

spacepoint_formation_algorithm::spacepoint_formation_algorithm(
    const traccc::memory_resource& mr, vecmem::copy& copy, stream& str,
    std::unique_ptr<const Logger> logger)
    : messaging(std::move(logger)), m_mr(mr), m_copy(copy), m_stream(str) {}

edm::spacepoint_collection::buffer spacepoint_formation_algorithm::operator()(
    const detector_buffer& detector,
    const measurement_collection_types::const_view& measurements_view) const {

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // Staging area for copying sizes from device to host
    vecmem::unique_alloc_ptr<unsigned int> size_staging_ptr =
        vecmem::make_unique_alloc<unsigned int>(*(m_mr.host));

    // Get the number of measurements.
    const measurement_collection_types::const_view::size_type num_measurements =
        get_size(measurements_view, size_staging_ptr.get(), stream);

    // Create the result buffer.
    edm::spacepoint_collection::buffer spacepoints(
        num_measurements, m_mr.main, vecmem::data::buffer_type::resizable);
    m_copy.get().setup(spacepoints)->ignore();

    // If there are no measurements, we can conclude here.
    if (num_measurements == 0) {
        return spacepoints;
    }

    // Launch parameters for the kernel.
    const unsigned int blockSize = 256;
    const unsigned int nBlocks = (num_measurements + blockSize - 1) / blockSize;

    // Launch the spacepoint formation kernel.
    detector_buffer_visitor<detector_type_list>(
        detector, [&]<typename detector_traits_t>(
                      const typename detector_traits_t::view& det) {
            kernels::form_spacepoints<detector_traits_t>
                <<<nBlocks, blockSize, 0, stream>>>(det, measurements_view,
                                                    spacepoints);
        });

    TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

    // Return the reconstructed spacepoints.
    return spacepoints;
}
}  // namespace traccc::cuda
