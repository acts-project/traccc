#include "hip/hip_runtime.h"
/*
 * TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "traccc/cuda/cca/component_connection.hpp"
#include "traccc/cuda/utils/definitions.hpp"
#include "traccc/edm/cell.hpp"
#include "traccc/edm/cluster.hpp"
#include "traccc/edm/measurement.hpp"
#include "vecmem/containers/vector.hpp"
#include "vecmem/memory/allocator.hpp"
#include "vecmem/memory/binary_page_memory_resource.hpp"
#include "vecmem/memory/cuda/device_memory_resource.hpp"
#include "vecmem/memory/cuda/managed_memory_resource.hpp"

namespace {
static constexpr std::size_t MAX_CELLS_PER_PARTITION = 2048;
static constexpr std::size_t THREADS_PER_BLOCK = 256;
using index_t = unsigned short;
}  // namespace

namespace traccc::cuda {
namespace details {

/*
 * Convenience structure to work with flattened data arrays instead of
 * an array/vector of cells.
 */
struct cell_container {
    std::size_t size = 0;
    channel_id* channel0 = nullptr;
    channel_id* channel1 = nullptr;
    scalar* activation = nullptr;
    scalar* time = nullptr;
    geometry_id* module_id = nullptr;
};

/*
 * Convenience structure to work with flattened data arrays instead of
 * an array/vector of measures.
 */
struct measurement_container {
    unsigned int size = 0;
    scalar* channel0 = nullptr;
    scalar* channel1 = nullptr;
    scalar* variance0 = nullptr;
    scalar* variance1 = nullptr;
    geometry_id* module_id = nullptr;
};

/*
 * Check if two cells are considered close enough to be part of the same
 * cluster.
 */
__device__ bool is_adjacent(channel_id ac0, channel_id ac1, channel_id bc0,
                            channel_id bc1) {
    unsigned int p0 = (ac0 - bc0);
    unsigned int p1 = (ac1 - bc1);

    return p0 * p0 <= 1 && p1 * p1 <= 1;
}

__device__ void reduce_problem_cell(cell_container& cells, index_t tid,
                                    unsigned char& adjc, index_t adjv[]) {
    /*
     * The number of adjacent cells for each cell must start at zero, to
     * avoid uninitialized memory. adjv does not need to be zeroed, as
     * we will only access those values if adjc indicates that the value
     * is set.
     */
    adjc = 0;

    channel_id c0 = cells.channel0[tid];
    channel_id c1 = cells.channel1[tid];
    geometry_id gid = cells.module_id[tid];

    /*
     * First, we traverse the cells backwards, starting from the current
     * cell and working back to the first, collecting adjacent cells
     * along the way.
     */
    for (index_t j = tid - 1; j < tid; --j) {
        /*
         * Since the data is sorted, we can assume that if we see a cell
         * sufficiently far away in both directions, it becomes
         * impossible for that cell to ever be adjacent to this one.
         * This is a small optimisation.
         */
        if (cells.channel1[j] + 1 < c1 || cells.module_id[j] != gid) {
            break;
        }

        /*
         * If the cell examined is adjacent to the current cell, save it
         * in the current cell's adjacency set.
         */
        if (is_adjacent(c0, c1, cells.channel0[j], cells.channel1[j])) {
            adjv[adjc++] = j;
        }
    }

    /*
     * Now we examine all the cells past the current one, using almost
     * the same logic as in the backwards pass.
     */
    for (index_t j = tid + 1; j < cells.size; ++j) {
        /*
         * Note that this check now looks in the opposite direction! An
         * important difference.
         */
        if (cells.channel1[j] > c1 + 1 || cells.module_id[j] != gid) {
            break;
        }

        if (is_adjacent(c0, c1, cells.channel0[j], cells.channel1[j])) {
            adjv[adjc++] = j;
        }
    }
}

/*
 * Implementation of a FastSV algorithm with the following steps:
 *   1) mix of stochastic and aggressive hooking
 *   2) shortcutting
 *
 * The implementation corresponds to an adapted versiion of Algorithm 3 of
 * the following paper:
 * https://www.sciencedirect.com/science/article/pii/S0743731520302689
 *
 * f      = array holding the parent cell ID for the current iteration.
 * gf     = array holding grandparent cell ID from the previous iteration.
            This array only gets updated at the end of the iteration to prevent
            race conditions.
 */
__device__ void fast_sv_1(index_t* f, index_t* gf, unsigned char adjc[],
                          index_t adjv[][8], unsigned int size) {
    /*
     * The algorithm finishes if an iteration leaves the arrays unchanged.
     * This varible will be set if a change is made, and dictates if another
     * loop is necessary.
     */
    bool gf_changed;

    do {
        /*
         * Reset the end-parameter to false, so we can set it to true if we
         * make a change to the gf array.
         */
        gf_changed = false;

        /*
         * The algorithm executes in a loop of three distinct parallel
         * stages. In this first one, a mix of stochastic and aggressive
         * hooking, we examine adjacent cells and copy their grand parents
         * cluster ID if it is lower than ours, essentially merging the two
         * together.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            __builtin_assume(adjc[tst] <= 8);

            for (unsigned char k = 0; k < adjc[tst]; ++k) {
                index_t q = gf[adjv[tst][k]];

                if (gf[tid] > q) {
                    f[f[tid]] = q;
                    f[tid] = q;
                }
            }
        }

        /*
         * Each stage in this algorithm must be preceded by a
         * synchronization barrier!
         */
        __syncthreads();

        /*
         * The second stage is shortcutting, which is an optimisation that
         * allows us to look at any shortcuts in the cluster IDs that we
         * can merge without adjacency information.
         */
        for (index_t tid = threadIdx.x; tid < size; tid += blockDim.x) {
            if (f[tid] > gf[tid]) {
                f[tid] = gf[tid];
            }
        }

        /*
         * Synchronize before the final stage.
         */
        __syncthreads();

        /*
         * Update the array for the next generation, keeping track of any
         * changes we make.
         */
        for (index_t tid = threadIdx.x; tid < size; tid += blockDim.x) {
            if (gf[tid] != f[f[tid]]) {
                gf[tid] = f[f[tid]];
                gf_changed = true;
            }
        }

        /*
         * To determine whether we need another iteration, we use block
         * voting mechanics. Each thread checks if it has made any changes
         * to the arrays, and votes. If any thread votes true, all threads
         * will return a true value and go to the next iteration. Only if
         * all threads return false will the loop exit.
         */
    } while (__syncthreads_or(gf_changed));
}

/*
 * Implementation of a FastSV algorithm with the following steps:
 *   1) stochastic hooking
 *   2) aggressive hooking
 *   3) shortcutting
 *
 * The implementation corresponds to Algorithm 2 of the following paper:
 * https://epubs.siam.org/doi/pdf/10.1137/1.9781611976137.5
 *
 * f      = array holding the parent cell ID for the current iteration.
 * f_next = buffer array holding updated information for the next iteration.
 */
__device__ void fast_sv_2(index_t* f, index_t* f_next, unsigned char adjc[],
                          index_t adjv[][8], unsigned int size) {
    /*
     * The algorithm finishes if an iteration leaves the array for the next
     * iteration unchanged.
     * This varible will be set if a change is made, and dictates if another
     * loop is necessary.
     */
    bool f_next_changed;

    do {
        /*
         * Reset the end-parameter to false, so we can set it to true if we
         * make a change to the f_next array.
         */
        f_next_changed = false;

        /*
         * The algorithm executes in a loop of four distinct parallel
         * stages. In this first one, stochastic hooking, we examine the
         * grandparents of adjacent cells and copy cluster ID if it
         * is lower than our, essentially merging the two together.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            for (unsigned char k = 0; k < adjc[tst]; ++k) {
                index_t q = f[f[adjv[tst][k]]];

                if (q < f_next[f[tid]]) {
                    // hook to grandparent of adjacent cell
                    f_next[f[tid]] = q;
                    f_next_changed = true;
                }
            }
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * The second stage performs aggressive hooking, during which each
         * cell might be hooked to the grand parent of an adjacent cell.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            for (unsigned char k = 0; k < adjc[tst]; ++k) {
                index_t q = f[f[adjv[tst][k]]];

                if (q < f_next[tid]) {
                    f_next[tid] = q;
                    f_next_changed = true;
                }
            }
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * The third stage is shortcutting, which is an optimisation that
         * allows us to look at any shortcuts in the cluster IDs that we
         * can merge without adjacency information.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            if (f[f[tid]] < f_next[tid]) {
                f_next[tid] = f[f[tid]];
                f_next_changed = true;
            }
        }

        /*
         * Synchronize before the final stage.
         */
        __syncthreads();

        /*
         * Update the array for the next generation.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            f[tid] = f_next[tid];
        }

        /*
         * To determine whether we need another iteration, we use block
         * voting mechanics. Each thread checks if it has made any changes
         * to the arrays, and votes. If any thread votes true, all threads
         * will return a true value and go to the next iteration. Only if
         * all threads return false will the loop exit.
         */
    } while (__syncthreads_or(f_next_changed));
}

/*
 * Implementation of a simplified SV algorithm with the following steps:
 *   1) tree hooking
 *   2) shortcutting
 *
 * The implementation corresponds to Algorithm 1 of the following paper:
 * https://epubs.siam.org/doi/pdf/10.1137/1.9781611976137.5
 *
 * f      = array holding the parent cell ID for the current iteration.
 * f_next = buffer array holding updated information for the next iteration.
 */
__device__ void simplified_sv(index_t* f, index_t* f_next, unsigned char adjc[],
                              index_t adjv[][8], unsigned int size) {
    /*
     * The algorithm finishes if an iteration leaves the array for the next
     * iteration unchanged.
     * This varible will be set if a change is made, and dictates if another
     * loop is necessary.
     */
    bool f_changed;

    do {
        /*
         * Reset the end-parameter to false, so we can set it to true if we
         * make a change to the f_next array.
         */
        f_changed = false;

        /*
         * The algorithm executes in a loop of four distinct parallel
         * stages. In this first one, tree hooking, we examine adjacent cells of
         * cluster roots and copy their cluster ID if it is lower than our,
         * essentially merging the two together.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            if (f[tid] == f[f[tid]]) {  // only perform for roots of clusters
                for (unsigned char k = 0; k < adjc[tst]; ++k) {
                    index_t q = f[adjv[tst][k]];
                    if (q < f[tid]) {
                        f_next[f[tid]] = q;
                        f_changed = true;
                    }
                }
            }
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * Update the array for the next stage of the iteration.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            f[tid] = f_next[tid];
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * The third stage is shortcutting, which is an optimisation that
         * allows us to look at any shortcuts in the cluster IDs that we
         * can merge without adjacency information.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            if (f[f[tid]] < f[tid]) {
                f_next[tid] = f[f[tid]];
                f_changed = true;
            }
        }

        /*
         * Synchronize before the final stage.
         */
        __syncthreads();

        /*
         * Update the array for the next generation.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            f[tid] = f_next[tid];
        }

        /*
         * To determine whether we need another iteration, we use block
         * voting mechanics. Each thread checks if it has made any changes
         * to the arrays, and votes. If any thread votes true, all threads
         * will return a true value and go to the next iteration. Only if
         * all threads return false will the loop exit.
         */
    } while (__syncthreads_or(f_changed));
}

__device__ void aggregate_clusters(const cell_container& cells,
                                   measurement_container& out, index_t* f) {
    __shared__ unsigned int outi;

    if (threadIdx.x == 0) {
        outi = 0;
    }

    __syncthreads();

    /*
     * This is the post-processing stage, where we merge the clusters into a
     * single measurement and write it to the output.
     */
    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {

        /*
         * If and only if the value in the work arrays is equal to the index
         * of a cell, that cell is the "parent" of a cluster of cells. If
         * they are not, there is nothing for us to do. Easy!
         */
        if (f[tid] == tid) {
            /*
             * If we are a cluster owner, atomically claim a position in the
             * output array which we can write to.
             */
            unsigned int id = atomicAdd(&outi, 1);

            /*
             * These variables keep track of the sums of X and Y coordinates
             * for the final coordinates, the total activation weight, as
             * well as the sum of squares of positions, which we use to
             * calculate the variance.
             */
            float sw = 0.0;
            float mx = 0.0, my = 0.0;
            float vx = 0.0, vy = 0.0;

            /*
             * Now, we iterate over all other cells to check if they belong
             * to our cluster. Note that we can start at the current index
             * because no cell is every a child of a cluster owned by a cell
             * with a higher ID.
             */
            for (index_t j = tid; j < cells.size; j++) {
                /*
                 * If the value of this cell is equal to our, that means it
                 * is part of our cluster. In that case, we take its values
                 * for position and add them to our accumulators.
                 */
                if (f[j] == tid) {
                    float w = cells.activation[j];

                    sw += w;

                    float pmx = mx, pmy = my;
                    float dx = cells.channel0[j] - pmx;
                    float dy = cells.channel1[j] - pmy;
                    float wf = w / sw;

                    mx = pmx + wf * dx;
                    my = pmy + wf * dy;

                    vx += w * dx * (cells.channel0[j] - mx);
                    vy += w * dy * (cells.channel1[j] - my);
                }
            }

            /*
             * Write the average weighted x and y coordinates, as well as
             * the weighted average square position, to the output array.
             */
            out.channel0[id] = mx;
            out.channel1[id] = my;
            out.variance0[id] = vx / sw;
            out.variance1[id] = vy / sw;
            out.module_id[id] = cells.module_id[tid];
        }
    }
}

__global__ __launch_bounds__(THREADS_PER_BLOCK) void ccl_kernel(
    const cell_container container, const unsigned* partitions,
    measurement_container& _out_ctnr) {
    const unsigned start = partitions[blockIdx.x];

    /*
     * Seek the correct cell region in the input data. Again, this is all a
     * contiguous block of memory for now, and we use the blocks array to
     * define the different ranges per block/module. At the end of this we
     * have the starting address of the block of cells dedicated to this
     * module, and we have its size.
     */
    cell_container cells;
    cells.size = partitions[blockIdx.x + 1] - partitions[blockIdx.x];
    cells.channel0 = &container.channel0[start];
    cells.channel1 = &container.channel1[start];
    cells.activation = &container.activation[start];
    cells.time = &container.time[start];
    cells.module_id = &container.module_id[start];

    assert(cells.size <= MAX_CELLS_PER_PARTITION);

    /*
     * As an optimisation, we will keep track of which cells are adjacent to
     * each other cell. To do this, we define, in thread-local memory or
     * registers, up to eight adjacent cell indices and we keep track of how
     * many adjacent cells there are (i.e. adjc[i] determines how many of
     * the eight values in adjv[i] are actually meaningful).
     *
     * The implementation is such that a thread might need to process more
     * than one hit. As such, we keep one counter and eight indices _per_
     * hit the thread is processing. This number is never larger than
     * the max number of activations per module divided by the threads per
     * block.
     *
     * adjc = adjecency count
     * adjv = adjecency vector
     */
    index_t adjv[MAX_CELLS_PER_PARTITION / THREADS_PER_BLOCK][8];
    unsigned char adjc[MAX_CELLS_PER_PARTITION / THREADS_PER_BLOCK];

    /*
     * After this is all done, we synchronise the block. I am not absolutely
     * certain that this is necessary here, but the overhead is not that big
     * and we might as well be safe rather than sorry.
     */
    __syncthreads();

    /*
     * This loop initializes the adjacency cache, which essentially
     * translates the sparse CCL problem into a graph CCL problem which we
     * can tackle with well-studied algorithms. This loop pattern is often
     * found throughout this code. We iterate over the number of activations
     * each thread must process. Sadly, the CUDA limit is 1024 threads per
     * block and we cannot guarantee that there will be fewer than 1024
     * activations in a module. So each thread must be able to do more than
     * one.
     */
    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {
        reduce_problem_cell(cells, tid, adjc[tst], adjv[tst]);
    }

    // if (threadIdx.x == 0) asm("mov.u32 %0, %clock;" : "=r"(c2) );

    /*
     * These arrays are the meat of the pudding of this algorithm, and we
     * will constantly be writing and reading from them which is why we
     * declare them to be in the fast shared memory. Note that this places a
     * limit on the maximum activations per module, as the amount of shared
     * memory is limited. These could always be moved to global memory, but
     * the algorithm would be decidedly slower in that case.
     */
    __shared__ index_t f[MAX_CELLS_PER_PARTITION],
        f_next[MAX_CELLS_PER_PARTITION];

    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {
        /*
         * At the start, the values of f and f_next should be equal to the
         * ID of the cell.
         */
        f[tid] = tid;
        f_next[tid] = tid;
    }

    /*
     * Now that the data has initialized, we synchronize again before we
     * move onto the actual processing part.
     */
    __syncthreads();

    fast_sv_1(f, f_next, adjc, adjv, cells.size);

    /*
     * This variable will be used to write to the output later.
     */
    __shared__ unsigned int outi;

    /*
     * Initialize the counter of clusters per thread block
     */
    if (threadIdx.x == 0) {
        outi = 0;
    }

    __syncthreads();

    /*
     * Count the number of clusters by checking how many cells have
     * themself assigned as a parent.
     */
    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {
        if (f[tid] == tid) {
            atomicAdd(&outi, 1);
        }
    }

    __syncthreads();

    /*
     * Add the number of clusters of each thread block to the total
     * number of clusters. At the same time, a cluster id is retrieved
     * for the next data processing step.
     * Note that this might be not the same cluster as has been treated
     * previously. However, since each thread block spawns a the maximum
     * amount of threads per block, this has no sever implications.
     */
    if (threadIdx.x == 0) {
        outi = atomicAdd(&_out_ctnr.size, outi);
    }

    __syncthreads();

    measurement_container out;
    out.channel0 = &_out_ctnr.channel0[outi];
    out.channel1 = &_out_ctnr.channel1[outi];
    out.variance0 = &_out_ctnr.variance0[outi];
    out.variance1 = &_out_ctnr.variance1[outi];
    out.module_id = &_out_ctnr.module_id[outi];

    aggregate_clusters(cells, out, f);
}

std::tuple<vecmem::unique_alloc_ptr<unsigned[]>, std::size_t> partition_cpu(
    const cell_container_types::host& data, vecmem::memory_resource& mem,
    const details::cell_container cells) {
    vecmem::unique_alloc_ptr<unsigned[]> partitions =
        vecmem::make_unique_alloc<unsigned[]>(mem, cells.size);
    std::size_t index = 0;
    std::size_t size = 0;
    std::size_t elements = 0;
    std::size_t pidx = 0;

    /*
     * Iterate over every cell module in the current data set.
     */
    for (std::size_t i = 0; i < data.size(); ++i) {
        /*
         * We start at 0 since this is the origin of the local coordinate
         * system within a cell module.
         */
        channel_id last_mid = 0;

        for (const cell& c : data.at(i).items) {
            /*
             * Create a new partition if an "empty" row is detected. A row
             * is considered "empty" if the channel1 value between two
             * consecutive cells have a difference > 1.
             * To prevent creating many small partitions, the current partition
             * must have at least twice the size of threads per block. This
             * guarantees that each thread handles later at least two cells.
             */
            if (c.channel1 > last_mid + 1 && size >= 2 * THREADS_PER_BLOCK) {
                partitions[pidx++] = index;

                index += size;
                size = 0;
            }

            last_mid = c.channel1;
            size += 1;
            elements += 1;
        }

        /*
         * If a cell module has many activations and therefore no empty
         * rows, it is possible that partitions reach a considerable
         * size. To prevent very big partitions, we check at the end of each
         * module if the current partition is not above a threshold, and end the
         * current partition if necessary here.
         */
        if (size >= 2 * THREADS_PER_BLOCK) {
            partitions[pidx++] = index;

            index += size;
            size = 0;
        }
    }

    /*
     * Create the very last partition after having iterated over all cell
     * modules and cells.
     */
    if (size > 0) {
        partitions[pidx++] = index;
    }

    partitions[pidx++] = elements;

    return {std::move(partitions), pidx};
}

__global__ void partition_kernel(const cell_container cells, unsigned* out,
                                 unsigned long long int* idx, unsigned slots) {
    /*
     * We will use shared memory as intermediate storage for our partitions.
     * All of this is mostly setup.
     */
    extern __shared__ unsigned tmp[];
    __shared__ unsigned tmp_idx;
    __shared__ unsigned out_idx;

    if (threadIdx.x == 0) {
        tmp_idx = 0;
    }

    __syncthreads();

    /*
     * In the first segment of the kernel, we will identify all cells for which
     * the next cell skips a row, or is on a different module. This marks a
     * valid partition point, even if this leads to an extremely fine
     * partition.
     */
    for (unsigned cid = blockIdx.x * slots + threadIdx.x;
         cid < (blockIdx.x + 1) * slots; cid += blockDim.x) {
        if (cid == 0 || cid == cells.size) {
            /*
             * We always need a partition that starts at the beginning, and a
             * trailing partition at the end. This clause ensures that.
             */
            tmp[atomicAdd(&tmp_idx, 1u)] = cid;
        } else if (cid + 1 < cells.size &&
                   (cells.channel1[cid + 1] > cells.channel1[cid] + 1 ||
                    cells.module_id[cid + 1] != cells.module_id[cid])) {
            /*
             * In this case, we have found an intermediate partition point: a
             * switch to a new module, or the next hit is more than a full row
             * away!
             */
            tmp[atomicAdd(&tmp_idx, 1u)] = cid + 1;
        }
    }

    __syncthreads();

    /*
     * We proceed with the next segment. The first segment finds partition
     * points, but the GPU does not guarantee that warps execute in order, so
     * the partitions may be scrambled. This implementation of odd-even sort
     * quickly sorts them.
     */
    bool sorted;

    do {
        sorted = true;

        /*
         * Odd component.
         */
        for (uint32_t j = 2 * threadIdx.x + 1; j + 1 < tmp_idx;
             j += 2 * blockDim.x) {

            if (tmp[j] > tmp[j + 1]) {
                unsigned k = tmp[j];
                tmp[j] = tmp[j + 1];
                tmp[j + 1] = k;
                sorted = false;
            }
        }

        __syncthreads();

        /*
         * Even component.
         */
        for (uint32_t j = 2 * threadIdx.x; j + 1 < tmp_idx;
             j += 2 * blockDim.x) {
            if (tmp[j] > tmp[j + 1]) {
                unsigned k = tmp[j];
                tmp[j] = tmp[j + 1];
                tmp[j + 1] = k;
                sorted = false;
            }
        }

        /*
         * We keep running until no thread reports that the array is unsorted!
         */
    } while (__syncthreads_or(!sorted));

    /*
     * Next, we will combine partitions to more evenly spread the load on the
     * actual CCL kernel.
     *
     * This code works by overriding the existing array of partition indices.
     * The `old_idx` variable denotes the end of the old array, the `base_idx`
     * variable denotes the current starting index in the old array, and the
     * `tmp_idx` variable denotes the index we write partitions to in the new
     * array. The old and new array are actually the same memory, but the
     * writing index for the new points will always be behind the reading
     * indices in the old part, so this is safe!
     */
    const unsigned old_idx = tmp_idx;

    __syncthreads();

    /*
     * Note that the first element always remains as it is, so we can simply
     * start the process from index 1; that means the first element is never
     * touched.
     */
    unsigned base_idx = 1;

    if (threadIdx.x == 0) {
        tmp_idx = 1;
    }

    __syncthreads();

    /*
     * Now, try to merge partitions. Note once again that the base index is the
     * position we look at in the old array. If this reaches the old index, we
     * have reached the final point in the array and we are done.
     */
    while (base_idx < old_idx) {
        /*
         * Retrieve the cell index of the last partition in the new segment
         * of the array; we will compare against this point to check whether
         * the size of the partition conforms with the maximum size.
         */
        unsigned base_val = tmp[tmp_idx - 1];

        /*
         * Each thread might need to check multiple partitions. We check in
         * blocks starting from the beginning of the array and moving towards
         * the end of it.
         */
        unsigned j = 0;
        int rem;

        do {
            unsigned i = base_idx + j * blockDim.x + threadIdx.x;

            /*
             * Each thread computes whether the partition it is investigating
             * lies within the boundaries of the permissible partition size.
             * A useful consequence of this is that the number of threads that
             * satisfy this condition is also the delta that we must apply to
             * the index to find the first partition that does _not_ satify the
             * requirement. This works because the array is sorted.
             *
             * In case all threads report that they are within reach, the split
             * may be in the next chunk. Thus, we consider a return equal to
             * the size of the block to mean that we need to try this process
             * again.
             */
            rem = __syncthreads_count(
                i + 1 < old_idx && tmp[i] < base_val + MAX_CELLS_PER_PARTITION);

            ++j;
        } while (rem == blockDim.x);

        /*
         * Compute the new base index.
         */
        base_idx += (j - 1) * blockDim.x + rem + 1;

        /*
         * The lead thread inserts the partition into the new array.
         */
        if (threadIdx.x == 0) {
            tmp[tmp_idx++] = tmp[base_idx - 1];
        }

        __syncthreads();
    }

    /*
     * Next, we reserve space in the output array in global memory.
     */
    if (threadIdx.x == 0) {
        if (tmp_idx > 0) {
            out_idx = atomicAdd(idx, tmp_idx);
        }
    }

    __syncthreads();

    /*
     * The remaining threads now wake up, and all threads proceed to write the
     * array of partitions from shared memory to global memory in a coalesced
     * fashion.
     */
    for (unsigned i = threadIdx.x; i < tmp_idx; i += blockDim.x) {
        out[out_idx + i] = tmp[i];
    }
}

__global__ void partition_sorting_kernel(unsigned* out,
                                         const unsigned long long int* count) {
    /*
     * This should only EVER be launched with a single block!
     */
    assert(gridDim.x == 1);

    /*
     * Another implementation of odd-even sorting. But can I say, despite its
     * O(n^2) worst case performance it's perfect for sorting small arrays on
     * parallel shared memory machines!
     */
    bool sorted;

    do {
        sorted = true;

        for (uint32_t j = 2 * threadIdx.x + 1; j + 1 < *count;
             j += 2 * blockDim.x) {

            if (out[j] > out[j + 1]) {
                unsigned k = out[j];
                out[j] = out[j + 1];
                out[j + 1] = k;
                sorted = false;
            }
        }

        __syncthreads();

        for (uint32_t j = 2 * threadIdx.x; j + 1 < *count;
             j += 2 * blockDim.x) {
            if (out[j] > out[j + 1]) {
                unsigned k = out[j];
                out[j] = out[j + 1];
                out[j + 1] = k;
                sorted = false;
            }
        }
    } while (__syncthreads_or(!sorted));
}

std::tuple<vecmem::unique_alloc_ptr<unsigned[]>, std::size_t> partition_gpu(
    const cell_container_types::host& data, vecmem::memory_resource& mem,
    const details::cell_container cells) {
    /*
     * First, we allocate memory for our partitions, as well as memory for
     * an integer in which to store the partition counts.
     */
    vecmem::unique_alloc_ptr<unsigned[]> partitions =
        vecmem::make_unique_alloc<unsigned[]>(mem, cells.size + 1);
    vecmem::unique_alloc_ptr<unsigned long long int> pidx =
        vecmem::make_unique_alloc<unsigned long long int>(mem);

    /*
     * The partition counter must be set to zero.
     */
    CUDA_ERROR_CHECK(hipMemset(pidx.get(), 0, sizeof(unsigned long long int)));

    /*
     * The partitioning kernel merges partitions within the same thread block.
     * This works better, in principle, if there are more partitions to
     * examine, because it reduces fragmentation of partitions. This means that
     * it is sometimes desirable to process more than one cell per thread. This
     * slots variable determines the number of cells that is examined per
     * block.
     */
    const unsigned slots = 512;

    /*
     * Launch the actual partitioning kernel and wait for it to finish.
     */
    const int grid_size =
        std::max(1ul, cells.size / slots + (cells.size % slots == 0 ? 0 : 1));
    const int blck_size = 256;
    const int smem_size = slots * sizeof(unsigned);

    partition_kernel<<<grid_size, blck_size, smem_size>>>(
        cells, partitions.get(), pidx.get(), slots);

    CUDA_ERROR_CHECK(hipPeekAtLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    /*
     * Next, we need to make sure that the partitions are sorted. Because there
     * are usually very few partitions (less than 1000) we can do this fairly
     * efficiently with a single block running odd-even sort.
     */
    partition_sorting_kernel<<<1, 1024>>>(partitions.get(), pidx.get());

    CUDA_ERROR_CHECK(hipPeekAtLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    /*
     * Finally, we copy the number of partitions back to the host.
     *
     * TODO: Replace this with dynamic parallelism to obviate the need for the
     * copy back to the host.
     */
    unsigned long long int hpidx;

    CUDA_ERROR_CHECK(hipMemcpy(&hpidx, pidx.get(),
                                sizeof(unsigned long long int),
                                hipMemcpyDeviceToHost));

    return {std::move(partitions), hpidx};
}
}  // namespace details

component_connection::output_type component_connection::operator()(
    const cell_container_types::host& data) const {
    vecmem::cuda::managed_memory_resource upstream;
    vecmem::cuda::device_memory_resource dmem;
    vecmem::binary_page_memory_resource mem(upstream);

    std::size_t total_cells = 0;

    for (std::size_t i = 0; i < data.size(); ++i) {
        total_cells += data.at(i).items.size();
    }

    /*
     * Flatten the data to handle memory access (fetch and cache)
     * more efficiently. This removes the hierarchy level that
     * references to the cell module.
     */
    vecmem::vector<channel_id> channel0(&mem);
    channel0.reserve(total_cells);
    vecmem::vector<channel_id> channel1(&mem);
    channel1.reserve(total_cells);
    vecmem::vector<scalar> activation(&mem);
    activation.reserve(total_cells);
    vecmem::vector<scalar> time(&mem);
    time.reserve(total_cells);
    vecmem::vector<geometry_id> module_id(&mem);
    module_id.reserve(total_cells);

    for (std::size_t i = 0; i < data.size(); ++i) {
        for (std::size_t j = 0; j < data.at(i).items.size(); ++j) {
            channel0.push_back(data.at(i).items.at(j).channel0);
            channel1.push_back(data.at(i).items.at(j).channel1);
            activation.push_back(data.at(i).items.at(j).activation);
            time.push_back(data.at(i).items.at(j).time);
            module_id.push_back(data.at(i).header.module);
        }
    }

    /*
     * Store the flattened arrays in a convenience data container.
     */
    details::cell_container container;
    container.size = total_cells;
    container.channel0 = channel0.data();
    container.channel1 = channel1.data();
    container.activation = activation.data();
    container.time = time.data();
    container.module_id = module_id.data();

    /*
     * Separate the problem into various subproblems (partitions).
     * We know that the input data is sorted primarily on channel1 (y-axis),
     * and secondarily on channel0 (x-axis). This allows the cheap creation
     * of partitions based on the distance of the y-value between two
     * consecutive cells. If this distance is above a threshold, we have the
     * guarantee that the two cells belong not to the same cluster.
     *
     * Runs on the GPU, but a CPU implementation is also available!
     */
    std::tuple<vecmem::unique_alloc_ptr<unsigned[]>, std::size_t> partitions =
        details::partition_gpu(data, dmem, container);

    /*
     * Reserve space for the result of the algorithm. Currently, there is
     * enough space allocated that (in theory) each cell could be a single
     * cluster, but this should not be the case with real experiment data.
     */
    vecmem::allocator alloc(mem);

    details::measurement_container* mctnr =
        alloc.new_object<details::measurement_container>();

    mctnr->channel0 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->channel1 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->variance0 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->variance1 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->module_id = static_cast<geometry_id*>(
        alloc.allocate_bytes(total_cells * sizeof(geometry_id)));

    /*
     * Run the connected component labeling algorithm to retrieve the clusters.
     *
     * This step includes the measurement (hit) creation for each cluster.
     */
    if (std::get<1>(partitions) > 1) {
        ccl_kernel<<<std::get<1>(partitions) - 1, THREADS_PER_BLOCK>>>(
            container, std::get<0>(partitions).get(), *mctnr);

        CUDA_ERROR_CHECK(hipPeekAtLastError());
        CUDA_ERROR_CHECK(hipDeviceSynchronize());
    }

    /*
     * Copy back the data from our flattened data structure into the traccc EDM.
     */
    output_type out;

    for (std::size_t i = 0; i < data.size(); ++i) {
        vecmem::vector<measurement> v(&mem);

        for (std::size_t j = 0; j < mctnr->size; ++j) {
            if (mctnr->module_id[j] == data.at(i).header.module) {
                measurement m;

                m.local = {mctnr->channel0[j], mctnr->channel1[j]};
                m.variance = {mctnr->variance0[j], mctnr->variance1[j]};

                v.push_back(m);
            }
        }

        out.push_back(cell_module(data.at(i).header), std::move(v));
    }

    return out;
}
}  // namespace traccc::cuda
