#include "hip/hip_runtime.h"
/*
 * TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "traccc/cuda/cca/component_connection.hpp"
#include "traccc/cuda/utils/definitions.hpp"
#include "vecmem/containers/vector.hpp"
#include "vecmem/memory/allocator.hpp"
#include "vecmem/memory/binary_page_memory_resource.hpp"
#include "vecmem/memory/cuda/device_memory_resource.hpp"
#include "vecmem/memory/cuda/managed_memory_resource.hpp"

namespace {
static constexpr std::size_t MAX_CELLS_PER_PARTITION = 2048;
static constexpr std::size_t TARGET_CELLS_PER_PARTITION = 1024;
static constexpr std::size_t THREADS_PER_BLOCK = 256;
using index_t = unsigned short;
}  // namespace

namespace traccc::cuda {
namespace details {

/*
 * Convenience structure to work with flattened data arrays instead of
 * an array/vector of cells.
 */
struct cell_container {
    std::size_t size = 0;
    channel_id* channel0 = nullptr;
    channel_id* channel1 = nullptr;
    scalar* activation = nullptr;
    scalar* time = nullptr;
    unsigned int* module_link = nullptr;
};

/*
 * Convenience structure to work with flattened data arrays instead of
 * an array/vector of measures.
 */
struct measurement_container {
    unsigned int size = 0;
    scalar* channel0 = nullptr;
    scalar* channel1 = nullptr;
    scalar* variance0 = nullptr;
    scalar* variance1 = nullptr;
    unsigned int* module_link = nullptr;
};

/*
 * Check if two cells are considered close enough to be part of the same
 * cluster.
 */
__device__ bool is_adjacent(channel_id ac0, channel_id ac1, channel_id bc0,
                            channel_id bc1) {
    unsigned int p0 = (ac0 - bc0);
    unsigned int p1 = (ac1 - bc1);

    return p0 * p0 <= 1 && p1 * p1 <= 1;
}

__device__ void reduce_problem_cell(cell_container& cells, index_t tid,
                                    unsigned char& adjc, index_t adjv[]) {
    /*
     * The number of adjacent cells for each cell must start at zero, to
     * avoid uninitialized memory. adjv does not need to be zeroed, as
     * we will only access those values if adjc indicates that the value
     * is set.
     */
    adjc = 0;

    channel_id c0 = cells.channel0[tid];
    channel_id c1 = cells.channel1[tid];
    unsigned int mlink = cells.module_link[tid];

    /*
     * First, we traverse the cells backwards, starting from the current
     * cell and working back to the first, collecting adjacent cells
     * along the way.
     */
    for (index_t j = tid - 1; j < tid; --j) {
        /*
         * Since the data is sorted, we can assume that if we see a cell
         * sufficiently far away in both directions, it becomes
         * impossible for that cell to ever be adjacent to this one.
         * This is a small optimisation.
         */
        if (cells.channel1[j] + 1 < c1 || cells.module_link[j] != mlink) {
            break;
        }

        /*
         * If the cell examined is adjacent to the current cell, save it
         * in the current cell's adjacency set.
         */
        if (is_adjacent(c0, c1, cells.channel0[j], cells.channel1[j])) {
            adjv[adjc++] = j;
        }
    }

    /*
     * Now we examine all the cells past the current one, using almost
     * the same logic as in the backwards pass.
     */
    for (index_t j = tid + 1; j < cells.size; ++j) {
        /*
         * Note that this check now looks in the opposite direction! An
         * important difference.
         */
        if (cells.channel1[j] > c1 + 1 || cells.module_link[j] != mlink) {
            break;
        }

        if (is_adjacent(c0, c1, cells.channel0[j], cells.channel1[j])) {
            adjv[adjc++] = j;
        }
    }
}

/*
 * Implementation of a FastSV algorithm with the following steps:
 *   1) mix of stochastic and aggressive hooking
 *   2) shortcutting
 *
 * The implementation corresponds to an adapted versiion of Algorithm 3 of
 * the following paper:
 * https://www.sciencedirect.com/science/article/pii/S0743731520302689
 *
 * f      = array holding the parent cell ID for the current iteration.
 * gf     = array holding grandparent cell ID from the previous iteration.
            This array only gets updated at the end of the iteration to prevent
            race conditions.
 */
__device__ void fast_sv_1(index_t* f, index_t* gf, unsigned char adjc[],
                          index_t adjv[][8], unsigned int size) {
    /*
     * The algorithm finishes if an iteration leaves the arrays unchanged.
     * This varible will be set if a change is made, and dictates if another
     * loop is necessary.
     */
    bool gf_changed;

    do {
        /*
         * Reset the end-parameter to false, so we can set it to true if we
         * make a change to the gf array.
         */
        gf_changed = false;

        /*
         * The algorithm executes in a loop of three distinct parallel
         * stages. In this first one, a mix of stochastic and aggressive
         * hooking, we examine adjacent cells and copy their grand parents
         * cluster ID if it is lower than ours, essentially merging the two
         * together.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            __builtin_assume(adjc[tst] <= 8);

            for (unsigned char k = 0; k < adjc[tst]; ++k) {
                index_t q = gf[adjv[tst][k]];

                if (gf[tid] > q) {
                    f[f[tid]] = q;
                    f[tid] = q;
                }
            }
        }

        /*
         * Each stage in this algorithm must be preceded by a
         * synchronization barrier!
         */
        __syncthreads();

        /*
         * The second stage is shortcutting, which is an optimisation that
         * allows us to look at any shortcuts in the cluster IDs that we
         * can merge without adjacency information.
         */
        for (index_t tid = threadIdx.x; tid < size; tid += blockDim.x) {
            if (f[tid] > gf[tid]) {
                f[tid] = gf[tid];
            }
        }

        /*
         * Synchronize before the final stage.
         */
        __syncthreads();

        /*
         * Update the array for the next generation, keeping track of any
         * changes we make.
         */
        for (index_t tid = threadIdx.x; tid < size; tid += blockDim.x) {
            if (gf[tid] != f[f[tid]]) {
                gf[tid] = f[f[tid]];
                gf_changed = true;
            }
        }

        /*
         * To determine whether we need another iteration, we use block
         * voting mechanics. Each thread checks if it has made any changes
         * to the arrays, and votes. If any thread votes true, all threads
         * will return a true value and go to the next iteration. Only if
         * all threads return false will the loop exit.
         */
    } while (__syncthreads_or(gf_changed));
}

/*
 * Implementation of a FastSV algorithm with the following steps:
 *   1) stochastic hooking
 *   2) aggressive hooking
 *   3) shortcutting
 *
 * The implementation corresponds to Algorithm 2 of the following paper:
 * https://epubs.siam.org/doi/pdf/10.1137/1.9781611976137.5
 *
 * f      = array holding the parent cell ID for the current iteration.
 * f_next = buffer array holding updated information for the next iteration.
 */
__device__ void fast_sv_2(index_t* f, index_t* f_next, unsigned char adjc[],
                          index_t adjv[][8], unsigned int size) {
    /*
     * The algorithm finishes if an iteration leaves the array for the next
     * iteration unchanged.
     * This varible will be set if a change is made, and dictates if another
     * loop is necessary.
     */
    bool f_next_changed;

    do {
        /*
         * Reset the end-parameter to false, so we can set it to true if we
         * make a change to the f_next array.
         */
        f_next_changed = false;

        /*
         * The algorithm executes in a loop of four distinct parallel
         * stages. In this first one, stochastic hooking, we examine the
         * grandparents of adjacent cells and copy cluster ID if it
         * is lower than our, essentially merging the two together.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            for (unsigned char k = 0; k < adjc[tst]; ++k) {
                index_t q = f[f[adjv[tst][k]]];

                if (q < f_next[f[tid]]) {
                    // hook to grandparent of adjacent cell
                    f_next[f[tid]] = q;
                    f_next_changed = true;
                }
            }
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * The second stage performs aggressive hooking, during which each
         * cell might be hooked to the grand parent of an adjacent cell.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            for (unsigned char k = 0; k < adjc[tst]; ++k) {
                index_t q = f[f[adjv[tst][k]]];

                if (q < f_next[tid]) {
                    f_next[tid] = q;
                    f_next_changed = true;
                }
            }
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * The third stage is shortcutting, which is an optimisation that
         * allows us to look at any shortcuts in the cluster IDs that we
         * can merge without adjacency information.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            if (f[f[tid]] < f_next[tid]) {
                f_next[tid] = f[f[tid]];
                f_next_changed = true;
            }
        }

        /*
         * Synchronize before the final stage.
         */
        __syncthreads();

        /*
         * Update the array for the next generation.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            f[tid] = f_next[tid];
        }

        /*
         * To determine whether we need another iteration, we use block
         * voting mechanics. Each thread checks if it has made any changes
         * to the arrays, and votes. If any thread votes true, all threads
         * will return a true value and go to the next iteration. Only if
         * all threads return false will the loop exit.
         */
    } while (__syncthreads_or(f_next_changed));
}

/*
 * Implementation of a simplified SV algorithm with the following steps:
 *   1) tree hooking
 *   2) shortcutting
 *
 * The implementation corresponds to Algorithm 1 of the following paper:
 * https://epubs.siam.org/doi/pdf/10.1137/1.9781611976137.5
 *
 * f      = array holding the parent cell ID for the current iteration.
 * f_next = buffer array holding updated information for the next iteration.
 */
__device__ void simplified_sv(index_t* f, index_t* f_next, unsigned char adjc[],
                              index_t adjv[][8], unsigned int size) {
    /*
     * The algorithm finishes if an iteration leaves the array for the next
     * iteration unchanged.
     * This varible will be set if a change is made, and dictates if another
     * loop is necessary.
     */
    bool f_changed;

    do {
        /*
         * Reset the end-parameter to false, so we can set it to true if we
         * make a change to the f_next array.
         */
        f_changed = false;

        /*
         * The algorithm executes in a loop of four distinct parallel
         * stages. In this first one, tree hooking, we examine adjacent cells of
         * cluster roots and copy their cluster ID if it is lower than our,
         * essentially merging the two together.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            if (f[tid] == f[f[tid]]) {  // only perform for roots of clusters
                for (unsigned char k = 0; k < adjc[tst]; ++k) {
                    index_t q = f[adjv[tst][k]];
                    if (q < f[tid]) {
                        f_next[f[tid]] = q;
                        f_changed = true;
                    }
                }
            }
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * Update the array for the next stage of the iteration.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            f[tid] = f_next[tid];
        }

        /*
         * Synchronize before the next stage.
         */
        __syncthreads();

        /*
         * The third stage is shortcutting, which is an optimisation that
         * allows us to look at any shortcuts in the cluster IDs that we
         * can merge without adjacency information.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            if (f[f[tid]] < f[tid]) {
                f_next[tid] = f[f[tid]];
                f_changed = true;
            }
        }

        /*
         * Synchronize before the final stage.
         */
        __syncthreads();

        /*
         * Update the array for the next generation.
         */
        for (index_t tst = 0, tid;
             (tid = tst * blockDim.x + threadIdx.x) < size; ++tst) {
            f[tid] = f_next[tid];
        }

        /*
         * To determine whether we need another iteration, we use block
         * voting mechanics. Each thread checks if it has made any changes
         * to the arrays, and votes. If any thread votes true, all threads
         * will return a true value and go to the next iteration. Only if
         * all threads return false will the loop exit.
         */
    } while (__syncthreads_or(f_changed));
}

__device__ void aggregate_clusters(const cell_container& cells,
                                   measurement_container& out, index_t* f) {
    __shared__ unsigned int outi;

    if (threadIdx.x == 0) {
        outi = 0;
    }

    __syncthreads();

    /*
     * This is the post-processing stage, where we merge the clusters into a
     * single measurement and write it to the output.
     */
    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {

        /*
         * If and only if the value in the work arrays is equal to the index
         * of a cell, that cell is the "parent" of a cluster of cells. If
         * they are not, there is nothing for us to do. Easy!
         */
        if (f[tid] == tid) {
            /*
             * If we are a cluster owner, atomically claim a position in the
             * output array which we can write to.
             */
            unsigned int id = atomicAdd(&outi, 1);

            /*
             * These variables keep track of the sums of X and Y coordinates
             * for the final coordinates, the total activation weight, as
             * well as the sum of squares of positions, which we use to
             * calculate the variance.
             */
            float sw = 0.0;
            float mx = 0.0, my = 0.0;
            float vx = 0.0, vy = 0.0;

            /*
             * Now, we iterate over all other cells to check if they belong
             * to our cluster. Note that we can start at the current index
             * because no cell is every a child of a cluster owned by a cell
             * with a higher ID.
             */
            for (index_t j = tid; j < cells.size; j++) {
                /*
                 * If the value of this cell is equal to our, that means it
                 * is part of our cluster. In that case, we take its values
                 * for position and add them to our accumulators.
                 */
                if (f[j] == tid) {
                    float w = cells.activation[j];

                    sw += w;

                    float pmx = mx, pmy = my;
                    float dx = cells.channel0[j] - pmx;
                    float dy = cells.channel1[j] - pmy;
                    float wf = w / sw;

                    mx = pmx + wf * dx;
                    my = pmy + wf * dy;

                    vx += w * dx * (cells.channel0[j] - mx);
                    vy += w * dy * (cells.channel1[j] - my);
                }
            }

            /*
             * Write the average weighted x and y coordinates, as well as
             * the weighted average square position, to the output array.
             */
            out.channel0[id] = mx;
            out.channel1[id] = my;
            out.variance0[id] = vx / sw;
            out.variance1[id] = vy / sw;
            out.module_link[id] = cells.module_link[tid];
        }
    }
}

__global__ __launch_bounds__(THREADS_PER_BLOCK) void ccl_kernel(
    const cell_container container, measurement_container& _out_ctnr,
    unsigned long num_cells) {
    __shared__ unsigned start, end;

    /*
     * First, we determine the exact range of cells that is to be examined by
     * this block of threads. We start from an initial range determined by the
     * block index multiplied by the target number of cells per block. We then
     * shift both the start and the end of the block forward (to a later point
     * in the array); start and end may be moved different amounts.
     */
    if (threadIdx.x == 0) {
        /*
         * Start off by naively determining the size of this block's partition.
         */
        start = blockIdx.x * TARGET_CELLS_PER_PARTITION;
        end =
            std::min(num_cells, (blockIdx.x + 1) * TARGET_CELLS_PER_PARTITION);

        /*
         * Next, shift the starting point to a position further in the array;
         * the purpose of this is to ensure that we are not operating on any
         * cells that have been claimed by the previous block (if any).
         */
        while (start != 0 &&
               container.module_link[start - 1] ==
                   container.module_link[start] &&
               container.channel1[start] <= container.channel1[start - 1] + 1) {
            ++start;
        }

        /*
         * Then, claim as many cells as we need past the naive end of the
         * current block to ensure that we do not end our partition on a cell
         * that is not a possible boundary!
         */
        while (end < num_cells &&
               container.module_link[end - 1] == container.module_link[end] &&
               container.channel1[end] <= container.channel1[end - 1] + 1) {
            ++end;
        }
    }

    __syncthreads();

    /*
     * Seek the correct cell region in the input data. Again, this is all a
     * contiguous block of memory for now, and we use the blocks array to
     * define the different ranges per block/module. At the end of this we
     * have the starting address of the block of cells dedicated to this
     * module, and we have its size.
     */
    cell_container cells;
    cells.size = end - start;
    cells.channel0 = &container.channel0[start];
    cells.channel1 = &container.channel1[start];
    cells.activation = &container.activation[start];
    cells.time = &container.time[start];
    cells.module_link = &container.module_link[start];

    assert(cells.size <= MAX_CELLS_PER_PARTITION);

    /*
     * As an optimisation, we will keep track of which cells are adjacent to
     * each other cell. To do this, we define, in thread-local memory or
     * registers, up to eight adjacent cell indices and we keep track of how
     * many adjacent cells there are (i.e. adjc[i] determines how many of
     * the eight values in adjv[i] are actually meaningful).
     *
     * The implementation is such that a thread might need to process more
     * than one hit. As such, we keep one counter and eight indices _per_
     * hit the thread is processing. This number is never larger than
     * the max number of activations per module divided by the threads per
     * block.
     *
     * adjc = adjecency count
     * adjv = adjecency vector
     */
    index_t adjv[MAX_CELLS_PER_PARTITION / THREADS_PER_BLOCK][8];
    unsigned char adjc[MAX_CELLS_PER_PARTITION / THREADS_PER_BLOCK];

    /*
     * After this is all done, we synchronise the block. I am not absolutely
     * certain that this is necessary here, but the overhead is not that big
     * and we might as well be safe rather than sorry.
     */
    __syncthreads();

    /*
     * This loop initializes the adjacency cache, which essentially
     * translates the sparse CCL problem into a graph CCL problem which we
     * can tackle with well-studied algorithms. This loop pattern is often
     * found throughout this code. We iterate over the number of activations
     * each thread must process. Sadly, the CUDA limit is 1024 threads per
     * block and we cannot guarantee that there will be fewer than 1024
     * activations in a module. So each thread must be able to do more than
     * one.
     */
    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {
        reduce_problem_cell(cells, tid, adjc[tst], adjv[tst]);
    }

    // if (threadIdx.x == 0) asm("mov.u32 %0, %clock;" : "=r"(c2) );

    /*
     * These arrays are the meat of the pudding of this algorithm, and we
     * will constantly be writing and reading from them which is why we
     * declare them to be in the fast shared memory. Note that this places a
     * limit on the maximum activations per module, as the amount of shared
     * memory is limited. These could always be moved to global memory, but
     * the algorithm would be decidedly slower in that case.
     */
    __shared__ index_t f[MAX_CELLS_PER_PARTITION],
        f_next[MAX_CELLS_PER_PARTITION];

    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {
        /*
         * At the start, the values of f and f_next should be equal to the
         * ID of the cell.
         */
        f[tid] = tid;
        f_next[tid] = tid;
    }

    /*
     * Now that the data has initialized, we synchronize again before we
     * move onto the actual processing part.
     */
    __syncthreads();

    fast_sv_1(f, f_next, adjc, adjv, cells.size);

    /*
     * This variable will be used to write to the output later.
     */
    __shared__ unsigned int outi;

    /*
     * Initialize the counter of clusters per thread block
     */
    if (threadIdx.x == 0) {
        outi = 0;
    }

    __syncthreads();

    /*
     * Count the number of clusters by checking how many cells have
     * themself assigned as a parent.
     */
    for (index_t tst = 0, tid;
         (tid = tst * blockDim.x + threadIdx.x) < cells.size; ++tst) {
        if (f[tid] == tid) {
            atomicAdd(&outi, 1);
        }
    }

    __syncthreads();

    /*
     * Add the number of clusters of each thread block to the total
     * number of clusters. At the same time, a cluster id is retrieved
     * for the next data processing step.
     * Note that this might be not the same cluster as has been treated
     * previously. However, since each thread block spawns a the maximum
     * amount of threads per block, this has no sever implications.
     */
    if (threadIdx.x == 0) {
        outi = atomicAdd(&_out_ctnr.size, outi);
    }

    __syncthreads();

    measurement_container out;
    out.channel0 = &_out_ctnr.channel0[outi];
    out.channel1 = &_out_ctnr.channel1[outi];
    out.variance0 = &_out_ctnr.variance0[outi];
    out.variance1 = &_out_ctnr.variance1[outi];
    out.module_link = &_out_ctnr.module_link[outi];

    aggregate_clusters(cells, out, f);
}
}  // namespace details

component_connection::output_type component_connection::operator()(
    const cell_collection_types::host& cells) const {
    vecmem::cuda::managed_memory_resource upstream;
    vecmem::cuda::device_memory_resource dmem;
    vecmem::binary_page_memory_resource mem(upstream);

    std::size_t total_cells = cells.size();

    /*
     * Flatten the data to handle memory access (fetch and cache)
     * more efficiently. This removes the hierarchy level that
     * references to the cell module.
     */
    vecmem::vector<channel_id> channel0(&mem);
    channel0.reserve(total_cells);
    vecmem::vector<channel_id> channel1(&mem);
    channel1.reserve(total_cells);
    vecmem::vector<scalar> activation(&mem);
    activation.reserve(total_cells);
    vecmem::vector<scalar> time(&mem);
    time.reserve(total_cells);
    vecmem::vector<unsigned int> module_link(&mem);
    module_link.reserve(total_cells);

    for (std::size_t i = 0; i < cells.size(); ++i) {
        channel0.push_back(cells.at(i).channel0);
        channel1.push_back(cells.at(i).channel1);
        activation.push_back(cells.at(i).activation);
        time.push_back(cells.at(i).time);
        module_link.push_back(cells.at(i).module_link);
    }

    /*
     * Store the flattened arrays in a convenience data container.
     */
    details::cell_container container;
    container.size = total_cells;
    container.channel0 = channel0.data();
    container.channel1 = channel1.data();
    container.activation = activation.data();
    container.time = time.data();
    container.module_link = module_link.data();

    /*
     * Reserve space for the result of the algorithm. Currently, there is
     * enough space allocated that (in theory) each cell could be a single
     * cluster, but this should not be the case with real experiment data.
     */
    vecmem::allocator alloc(mem);

    details::measurement_container* mctnr =
        alloc.new_object<details::measurement_container>();

    mctnr->channel0 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->channel1 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->variance0 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->variance1 = static_cast<scalar*>(
        alloc.allocate_bytes(total_cells * sizeof(scalar)));
    mctnr->module_link = static_cast<unsigned int*>(
        alloc.allocate_bytes(total_cells * sizeof(unsigned int)));

    /*
     * Run the connected component labeling algorithm to retrieve the clusters.
     *
     * This step includes the measurement (hit) creation for each cluster.
     */
    ccl_kernel<<<
        std::max(1ul,
                 (total_cells / TARGET_CELLS_PER_PARTITION) +
                     (total_cells % TARGET_CELLS_PER_PARTITION != 0 ? 1 : 0)),
        THREADS_PER_BLOCK>>>(container, *mctnr, total_cells);

    CUDA_ERROR_CHECK(hipPeekAtLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    /*
     * Copy back the data from our SoA data structure into the traccc EDM.
     */
    output_type out;

    for (std::size_t i = 0; i < mctnr->size; ++i) {
        measurement m;
        m.local = {mctnr->channel0[i], mctnr->channel1[i]};
        m.variance = {mctnr->variance0[i], mctnr->variance1[i]};
        m.module_link = mctnr->module_link[i];

        out.push_back(m);
    }

    return out;
}
}  // namespace traccc::cuda
