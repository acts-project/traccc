#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022-2024 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "../utils/cuda_error_handling.hpp"
#include "../utils/utils.hpp"
#include "traccc/cuda/fitting/fitting_algorithm.hpp"
#include "traccc/fitting/device/fill_sort_keys.hpp"
#include "traccc/fitting/device/fit.hpp"
#include "traccc/fitting/kalman_filter/kalman_fitter.hpp"

// detray include(s).
#include "detray/core/detector_metadata.hpp"
#include "detray/detectors/bfield.hpp"
#include "detray/propagator/rk_stepper.hpp"

// Thrust include(s).
#include <thrust/sort.h>

// System include(s).
#include <vector>

namespace traccc::cuda {

namespace kernels {

__global__ void fill_sort_keys(
    track_candidate_container_types::const_view track_candidates_view,
    vecmem::data::vector_view<device::sort_key> keys_view,
    vecmem::data::vector_view<unsigned int> ids_view) {

    device::fill_sort_keys(threadIdx.x + blockIdx.x * blockDim.x,
                           track_candidates_view, keys_view, ids_view);
}

template <typename fitter_t, typename detector_view_t>
__global__ void fit(
    detector_view_t det_data, const typename fitter_t::bfield_type field_data,
    const typename fitter_t::config_type cfg,
    track_candidate_container_types::const_view track_candidates_view,
    vecmem::data::vector_view<const unsigned int> param_ids_view,
    track_state_container_types::view track_states_view) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    device::fit<fitter_t>(gid, det_data, field_data, cfg, track_candidates_view,
                          param_ids_view, track_states_view);
}

}  // namespace kernels

template <typename fitter_t>
fitting_algorithm<fitter_t>::fitting_algorithm(
    const config_type& cfg, const traccc::memory_resource& mr,
    vecmem::copy& copy, stream& str)
    : m_cfg(cfg),
      m_mr(mr),
      m_copy(copy),
      m_stream(str),
      m_warp_size(details::get_warp_size(str.device())) {}

template <typename fitter_t>
track_state_container_types::buffer fitting_algorithm<fitter_t>::operator()(
    const typename fitter_t::detector_type::view_type& det_view,
    const typename fitter_t::bfield_type& field_view,
    const typename track_candidate_container_types::const_view&
        track_candidates_view) const {

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // Number of tracks
    const track_candidate_container_types::const_device::header_vector::
        size_type n_tracks = m_copy.get_size(track_candidates_view.headers);

    // Get the sizes of the track candidates in each track
    const std::vector<track_candidate_container_types::const_device::
                          item_vector::value_type::size_type>
        candidate_sizes = m_copy.get_sizes(track_candidates_view.items);

    track_state_container_types::buffer track_states_buffer{
        {n_tracks, m_mr.main},
        {candidate_sizes, m_mr.main, m_mr.host,
         vecmem::data::buffer_type::resizable}};

    m_copy.setup(track_states_buffer.headers)->ignore();
    m_copy.setup(track_states_buffer.items)->ignore();

    // Calculate the number of threads and thread blocks to run the track
    // fitting
    if (n_tracks > 0) {
        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks = (n_tracks + nThreads - 1) / nThreads;

        vecmem::data::vector_buffer<device::sort_key> keys_buffer(n_tracks,
                                                                  m_mr.main);
        vecmem::data::vector_buffer<unsigned int> param_ids_buffer(n_tracks,
                                                                   m_mr.main);

        // Get key and value for sorting
        kernels::fill_sort_keys<<<nBlocks, nThreads, 0, stream>>>(
            track_candidates_view, keys_buffer, param_ids_buffer);
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

        // Sort the key to get the sorted parameter ids
        vecmem::device_vector<device::sort_key> keys_device(keys_buffer);
        vecmem::device_vector<unsigned int> param_ids_device(param_ids_buffer);

        thrust::sort_by_key(thrust::cuda::par.on(stream), keys_device.begin(),
                            keys_device.end(), param_ids_device.begin());

        // Run the track fitting
        kernels::fit<fitter_t><<<nBlocks, nThreads, 0, stream>>>(
            det_view, field_view, m_cfg, track_candidates_view,
            param_ids_buffer, track_states_buffer);
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());
    }

    m_stream.synchronize();

    return track_states_buffer;
}

// Explicit template instantiation
using default_detector_type =
    detray::detector<detray::default_metadata, detray::device_container_types>;
using default_stepper_type =
    detray::rk_stepper<covfie::field<detray::bfield::const_bknd_t>::view_t,
                       default_algebra, detray::constrained_step<>>;
using default_navigator_type = detray::navigator<const default_detector_type>;
using default_fitter_type =
    kalman_fitter<default_stepper_type, default_navigator_type>;
template class fitting_algorithm<default_fitter_type>;

}  // namespace traccc::cuda
