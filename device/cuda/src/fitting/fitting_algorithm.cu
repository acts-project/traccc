#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "traccc/cuda/fitting/fitting_algorithm.hpp"
#include "traccc/cuda/utils/definitions.hpp"
#include "traccc/fitting/device/fit.hpp"
#include "traccc/fitting/kalman_filter/kalman_fitter.hpp"

// detray include(s).
#include "detray/detectors/detector_metadata.hpp"
#include "detray/masks/unbounded.hpp"
#include "detray/propagator/rk_stepper.hpp"

// System include(s).
#include <vector>

namespace traccc::cuda {

namespace kernels {

template <typename fitter_t, typename detector_view_t>
__global__ void fit(
    detector_view_t det_data, const typename fitter_t::config_type cfg,
    vecmem::data::jagged_vector_view<typename fitter_t::intersection_type>
        nav_candidates_buffer,
    track_candidate_container_types::const_view track_candidates_view,
    track_state_container_types::view track_states_view) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    device::fit<fitter_t>(gid, det_data, cfg, nav_candidates_buffer,
                          track_candidates_view, track_states_view);
}

}  // namespace kernels

template <typename fitter_t>
fitting_algorithm<fitter_t>::fitting_algorithm(
    const config_type& cfg, const traccc::memory_resource& mr)
    : m_cfg(cfg), m_mr(mr) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
};

template <typename fitter_t>
track_state_container_types::buffer fitting_algorithm<fitter_t>::operator()(
    const typename fitter_t::detector_type::detector_view_type& det_view,
    const vecmem::data::jagged_vector_view<
        typename fitter_t::intersection_type>& navigation_buffer,
    const typename track_candidate_container_types::const_view&
        track_candidates_view) const {

    // Number of tracks
    const track_candidate_container_types::const_device::header_vector::
        size_type n_tracks = m_copy->get_size(track_candidates_view.headers);

    // Get the sizes of the track candidates in each track
    const std::vector<track_candidate_container_types::const_device::
                          item_vector::value_type::size_type>
        candidate_sizes = m_copy->get_sizes(track_candidates_view.items);

    track_state_container_types::buffer track_states_buffer{
        {n_tracks, m_mr.main},
        {candidate_sizes, m_mr.main, m_mr.host,
         vecmem::data::buffer_type::resizable}};

    m_copy->setup(track_states_buffer.headers);
    m_copy->setup(track_states_buffer.items);
    m_copy->setup(navigation_buffer);

    // Calculate the number of threads and thread blocks to run the track
    // fitting
    if (n_tracks > 0) {
        const unsigned int nThreads = WARP_SIZE * 2;
        const unsigned int nBlocks = (n_tracks + nThreads - 1) / nThreads;

        // Run the track fitting
        kernels::fit<fitter_t>
            <<<nBlocks, nThreads>>>(det_view, m_cfg, navigation_buffer,
                                    track_candidates_view, track_states_buffer);
        CUDA_ERROR_CHECK(hipGetLastError());
        CUDA_ERROR_CHECK(hipDeviceSynchronize());
    }
    return track_states_buffer;
}

// Explicit template instantiation
using toy_detector_type =
    detray::detector<detray::detector_registry::toy_detector,
                     covfie::field_view, detray::device_container_types>;
using toy_stepper_type = detray::rk_stepper<
    covfie::field<toy_detector_type::bfield_backend_type>::view_t, transform3,
    detray::constrained_step<>>;
using toy_navigator_type = detray::navigator<const toy_detector_type>;
using toy_fitter_type = kalman_fitter<toy_stepper_type, toy_navigator_type>;
template class fitting_algorithm<toy_fitter_type>;

using device_detector_type =
    detray::detector<detray::detector_registry::template telescope_detector<
                         detray::rectangle2D<>>,
                     covfie::field_view, detray::device_container_types>;
using rk_stepper_type = detray::rk_stepper<
    covfie::field<device_detector_type::bfield_backend_type>::view_t,
    transform3, detray::constrained_step<>>;
using device_navigator_type = detray::navigator<const device_detector_type>;
using device_fitter_type =
    kalman_fitter<rk_stepper_type, device_navigator_type>;
template class fitting_algorithm<device_fitter_type>;

}  // namespace traccc::cuda