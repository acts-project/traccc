#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022-2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../../utils/cuda_error_handling.hpp"
#include "../../utils/global_index.hpp"
#include "fill_sort_keys.hpp"

// Project include(s).
#include "traccc/fitting/device/fill_sort_keys.hpp"

namespace traccc::cuda {
namespace kernels {

__global__ void fill_sort_keys(
    edm::track_candidate_collection<default_algebra>::const_view
        track_candidates_view,
    vecmem::data::vector_view<device::sort_key> keys_view,
    vecmem::data::vector_view<unsigned int> ids_view) {

    device::fill_sort_keys(details::global_index1(), track_candidates_view,
                           keys_view, ids_view);
}

}  // namespace kernels

void fill_sort_keys(const dim3& grid_size, const dim3& block_size,
                    hipStream_t stream,
                    edm::track_candidate_collection<default_algebra>::const_view
                        track_candidates_view,
                    vecmem::data::vector_view<device::sort_key> keys_view,
                    vecmem::data::vector_view<unsigned int> ids_view) {

    kernels::fill_sort_keys<<<block_size, grid_size, 0, stream>>>(
        track_candidates_view, keys_view, ids_view);
    TRACCC_CUDA_ERROR_CHECK(hipGetLastError());
}

}  // namespace traccc::cuda
