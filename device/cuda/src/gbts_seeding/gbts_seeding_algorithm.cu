#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "../utils/cuda_error_handling.hpp"
#include "../utils/utils.hpp"
#include "./kernels/GbtsNodesMakingKernels.cuh"
//#include "./kernels/GbtsGraphMaking.cuh"
//#include "./kernels/GbtsGraphProcessing.cuh"

#include "traccc/cuda/gbts_seeding/gbts_seeding_algorithm.hpp"

namespace traccc::cuda {

struct gbts_ctx {
	//counters
	unsigned int nSp{};
	unsigned int nEdges{};
	unsigned int nConnectedEdges{};
	unsigned int nSeeds{};
	//nEdges, nConnections, nConnectedEdges, .., nSeeds
	unsigned int* d_counters;	

	//NodeMaking
	unsigned int* d_layerCounts{};
	unsigned short* d_spacepointsLayer{};
	//begin_idx + 1 for the surfaceToLayerMap or -layerBin if one to one
	int* d_volumeToLayerMap{};	
	//surface_id, layerBin
	uint2* d_surfaceToLayerMap{};

	traccc::device::gbts_layerInfo* d_layerInfo{};

	//x,y,z,cluster width in eta
	float4* d_reducedSP{};
	//output of layer binning
	float4* d_sp_params{};	

	//GraphMaking

	//GraphProccessing

	//output
};

gbts_seeding_algorithm::gbts_seeding_algorithm(const gbts_seedfinder_config& cfg, traccc::memory_resource& mr, vecmem::copy& copy, stream& str, std::unique_ptr<const Logger> logger)
                                               : messaging(logger->clone()), m_config(cfg), m_mr(mr), m_copy(copy), m_stream(str) {}

gbts_seeding_algorithm::output_type gbts_seeding_algorithm::operator()(const traccc::edm::spacepoint_collection::const_view& spacepoints, const traccc::measurement_collection_types::const_view& measurements) const {

	gbts_seeding_algorithm::output_type output_seeds;

	gbts_ctx ctx;

	hipStream_t stream = details::get_stream(m_stream);

	//0. bin spacepoints by layer(disk) or any other maping supplied to the config.m_surfaceToLayerMap
	ctx.nSp = m_copy.get().get_size(spacepoints); //why is get needed?

	unsigned int nThreads = 1024;
	unsigned int nBlocks = 1+(ctx.nSp-1)/nThreads;

	hipMalloc(&ctx.d_layerCounts, m_config.nLayers*sizeof(unsigned int));	
	hipMalloc(&ctx.d_spacepointsLayer, ctx.nSp*sizeof(unsigned char));	
	hipMalloc(&ctx.d_reducedSP, ctx.nSp*sizeof(float4));	

	hipMalloc(&ctx.d_volumeToLayerMap, sizeof(int)*m_config.maxVolIndex);	
	hipMalloc(&ctx.d_surfaceToLayerMap, sizeof(uint2)*m_config.surfaceMapSize);	

	hipMemcpyAsync(ctx.d_volumeToLayerMap, m_config.volumeToLayerMap.get(), sizeof(int)*m_config.maxVolIndex, hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(ctx.d_surfaceToLayerMap, m_config.surfaceToLayerMap.data(), sizeof(uint2)*m_config.surfaceMapSize, hipMemcpyHostToDevice, stream);

	hipMalloc(&ctx.d_layerInfo, sizeof(traccc::device::gbts_layerInfo)*m_config.nLayers);
	hipMemcpyAsync(ctx.d_layerInfo, m_config.layerInfo.data(), sizeof(traccc::device::gbts_layerInfo)*m_config.nLayers, hipMemcpyHostToDevice, stream);	

	kernels::count_sp_by_layer<<<nBlocks,nThreads,0,stream>>>(spacepoints,measurements,
	                            ctx.d_volumeToLayerMap,ctx.d_surfaceToLayerMap,ctx.d_layerInfo, 
                                ctx.d_reducedSP, ctx.d_layerCounts, ctx.d_spacepointsLayer,
								ctx.nSp, m_config.surfaceMapSize);
	//prefix sum layerCounts
	std::unique_ptr<unsigned int[]> layerCounts = std::make_unique<unsigned int[]>(m_config.nLayers);

	hipMemcpyAsync(ctx.d_layerCounts, layerCounts.get(), m_config.nLayers*sizeof(unsigned int), hipMemcpyDeviceToHost, stream);	
	for(int layer = 1; layer<m_config.nLayers; layer++) {
		layerCounts[layer] += layerCounts[layer-1];
	}
	hipMemcpyAsync(layerCounts.get(), ctx.d_layerCounts, m_config.nLayers*sizeof(unsigned int), hipMemcpyHostToDevice, stream);	

	hipMalloc(&ctx.d_sp_params, ctx.nSp*sizeof(float4));	

	kernels::bin_sp_by_layer<<<nBlocks, nThreads, 0, stream>>>(ctx.d_sp_params, ctx.d_reducedSP, ctx.d_layerCounts, ctx.d_spacepointsLayer, ctx.nSp);	

	//1. histogram spacepoints by layer->eta->phi and convert to nodes phi,r,z,tau_min,tau_max

	//2. Find edges between spacepoint pairs

	//3. Link edges into graph

	//4. Prune unlinked edges from graph

	//5. Find longest segments with CCA

	//6. extract seeds, longest segment first

	return output_seeds;
}

} //namespace traccc::cuda
