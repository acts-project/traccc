#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "../utils/cuda_error_handling.hpp"
#include "../utils/utils.hpp"
#include "./kernels/GbtsGraphMakingKernels.cuh"
#include "./kernels/GbtsGraphProcessingKernels.cuh"
#include "./kernels/GbtsNodesMakingKernels.cuh"
#include "traccc/cuda/gbts_seeding/gbts_seeding_algorithm.hpp"

// C++ include(s)
#include <ranges>

namespace traccc::cuda {

struct gbts_ctx {
    // counters
    unsigned int nSp{};
    unsigned int nNodes{};
    unsigned int nUsedBinPairs{};
    unsigned int nMaxEdges{};

    unsigned int nEdges{};
    unsigned int nConnections{};
    unsigned int nConnectedEdges{};
    unsigned int nSeeds{};
    // nEdges, nConnections, nConnectedEdges, .., nSeeds
    unsigned int* d_counters{};

    // device side graph building cuts
    gbts_graph_building_params* d_graph_building_params;

    // node making and binning
    int* d_layerCounts{};
    short* d_spacepointsLayer{};
    // begin_idx + 1 for the surfaceToLayerMap or -layerBin if one to one
    short* d_volumeToLayerMap{};
    uint2* d_surfaceToLayerMap{};  // surface_index, layerBin
    char* d_layerType{};
    // conversion to original sp from post layer binning index
    int* d_original_sp_idx{};
    // conversion to orignal sp/node index from post binning index
    int* d_node_index{};

    // x,y,z,cluster width in eta
    float4* d_reducedSP{};
    // layer binned reducedSP
    float4* d_sp_params{};

    int2* d_layer_info{};
    float2* d_layer_geo{};

    int* d_node_eta_index{};
    int* d_node_phi_index{};

    int* d_eta_phi_histo{};     // for data binning
    int* d_phi_cusums{};        // for data binning
    int* d_eta_node_counter{};  // for data binning

    int2* d_eta_bin_views{};  // views of the nodes
    // eta-bin views of the node_params array
    std::unique_ptr<int[]> h_eta_bin_views{};

    float2* d_bin_rads{};  // minimum and maximum r of nodes inside an eta-bin
    std::unique_ptr<float[]> h_bin_rads{};

    uint4* d_bin_pair_views{};
    std::unique_ptr<int[]> h_bin_pair_views{};

    std::unique_ptr<float[]> h_bin_pair_dphi{};
    float* d_bin_pair_dphi{};
    // node making output
    float* d_node_params{};

    // GraphMaking
    int2* d_edge_nodes{};
    kernels::half4* d_edge_params{};

    int* d_num_incoming_edges{};
    int* d_edge_links{};

    unsigned char* d_num_neighbours{};
    int* d_reIndexer{};
    int* d_neighbours{};
    // offload this for CPU-side seed extraction
    int* d_output_graph{};

    // message-passing CCA
    // holds indices of the edges that need more CCA iterations
    int* d_active_edges{};
    // d_levels[edge_idx] = the maxium length of seeds starting with this edge
    char* d_levels{};
    // #paths, is terminus
    short2* d_outgoing_paths{};

    // seed-extraction walkthrough

    // edge_idx and prev path_store idx forms a uniuqe path through the graph
    int2* d_path_store{};
    int2* d_seed_proposals{};  // int quality and final mini_state_idx
    // first 32 bits are seed quality second 32 bits are seed_proposals index
    unsigned long long int* d_edge_bids{};
    // 0 as default/is real seed, 1 as maybe seed,
    //-1 as maybe fake seed, -2 as fake
    char* d_seed_ambiguity{};
};

gbts_seeding_algorithm::gbts_seeding_algorithm(
    const gbts_seedfinder_config& cfg, traccc::memory_resource& mr,
    vecmem::copy& copy, stream& str, std::unique_ptr<const Logger> logger)
    : messaging(logger->clone()),
      m_config(cfg),
      m_mr(mr),
      m_copy(copy),
      m_stream(str) {}

gbts_seeding_algorithm::output_type gbts_seeding_algorithm::operator()(
    const traccc::edm::spacepoint_collection::const_view& spacepoints,
    const traccc::measurement_collection_types::const_view& measurements)
    const {

    gbts_ctx ctx;

    hipStream_t stream = details::get_stream(m_stream);

    hipMalloc(&ctx.d_graph_building_params,
               sizeof(m_config.graph_building_params));
    hipMemcpyAsync(
        ctx.d_graph_building_params, &m_config.graph_building_params,
        sizeof(m_config.graph_building_params), hipMemcpyHostToDevice);

    // 0. bin spacepoints by the maping supplied to config.m_surfaceToLayerMap
    ctx.nSp = m_copy.get().get_size(spacepoints);
    if (ctx.nSp == 0) {
        return {0, m_mr.main};
    }
    unsigned int nThreads = 128;
    unsigned int nBlocks = 1 + (ctx.nSp - 1) / nThreads;

    hipMalloc(&ctx.d_layerCounts, (m_config.nLayers + 1) * sizeof(int));
    hipMemsetAsync(ctx.d_layerCounts, 0, (m_config.nLayers + 1) * sizeof(int),
                    stream);

    hipMalloc(&ctx.d_spacepointsLayer, ctx.nSp * sizeof(short));
    hipMalloc(&ctx.d_reducedSP, ctx.nSp * sizeof(float4));

    hipMalloc(&ctx.d_volumeToLayerMap,
               sizeof(short) * m_config.volumeToLayerMap.size());

    hipMemcpyAsync(ctx.d_volumeToLayerMap, m_config.volumeToLayerMap.data(),
                    sizeof(short) * m_config.volumeToLayerMap.size(),
                    hipMemcpyHostToDevice, stream);

    if (m_config.surfaceToLayerMap.size() != 0) {
        hipMalloc(&ctx.d_surfaceToLayerMap,
                   sizeof(uint2) * m_config.surfaceToLayerMap.size());

        hipMemcpyAsync(ctx.d_surfaceToLayerMap,
                        m_config.surfaceToLayerMap.data(),
                        sizeof(uint2) * m_config.surfaceToLayerMap.size(),
                        hipMemcpyHostToDevice, stream);
    }  // may be zero and correct, volumeMapSize, nLayers are checked at config

    hipMalloc(&ctx.d_layerType, sizeof(char) * m_config.nLayers);
    hipMemcpyAsync(ctx.d_layerType, m_config.layerInfo.type.data(),
                    sizeof(char) * m_config.nLayers, hipMemcpyHostToDevice,
                    stream);

    kernels::count_sp_by_layer<<<nBlocks, nThreads, 0, stream>>>(
        spacepoints, measurements, ctx.d_volumeToLayerMap,
        ctx.d_surfaceToLayerMap, ctx.d_layerType, ctx.d_reducedSP,
        ctx.d_layerCounts, ctx.d_spacepointsLayer,
        m_config.graph_building_params.type1_max_width, ctx.nSp,
        m_config.volumeToLayerMap.size(), m_config.surfaceToLayerMap.size());

    hipStreamSynchronize(stream);

    hipFree(ctx.d_volumeToLayerMap);
    hipFree(ctx.d_surfaceToLayerMap);
    hipFree(ctx.d_layerType);

    // prefix sum layerCounts
    std::unique_ptr<int[]> layerCounts =
        std::make_unique<int[]>(m_config.nLayers + 1);

    hipMemcpyAsync(layerCounts.get(), ctx.d_layerCounts,
                    (m_config.nLayers + 1) * sizeof(int),
                    hipMemcpyDeviceToHost, stream);

    for (size_t layer = 0; layer < m_config.nLayers; layer++) {
        layerCounts[layer + 1] += layerCounts[layer];
    }
    hipMemcpyAsync(ctx.d_layerCounts, layerCounts.get(),
                    m_config.nLayers * sizeof(int), hipMemcpyHostToDevice,
                    stream);

    ctx.nNodes = static_cast<unsigned int>(layerCounts[m_config.nLayers]);
    if (ctx.nNodes == 0)
        return {0, m_mr.main};
    layerCounts.reset();

    hipMalloc(&ctx.d_sp_params, ctx.nSp * sizeof(float4));
    hipMalloc(&ctx.d_original_sp_idx, ctx.nSp * sizeof(int));

    kernels::bin_sp_by_layer<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_sp_params, ctx.d_reducedSP, ctx.d_layerCounts,
        ctx.d_spacepointsLayer, ctx.d_original_sp_idx, ctx.nSp);

    hipStreamSynchronize(stream);
    hipError_t error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR("spacepoint layer binning: CUDA error: "
                     << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    hipFree(ctx.d_spacepointsLayer);

    // 1. histogram spacepoints by layer->eta->phi and convert to nodes
    // do this in config setup?
    hipMalloc(&ctx.d_layer_info, sizeof(int2) * m_config.nLayers);
    hipMemcpyAsync(ctx.d_layer_info, m_config.layerInfo.info.data(),
                    sizeof(int2) * m_config.nLayers, hipMemcpyHostToDevice,
                    stream);

    hipMalloc(&ctx.d_layer_geo, sizeof(float2) * m_config.nLayers);
    hipMemcpyAsync(ctx.d_layer_geo, m_config.layerInfo.geo.data(),
                    sizeof(float2) * m_config.nLayers, hipMemcpyHostToDevice,
                    stream);

    hipMalloc(&ctx.d_node_phi_index, sizeof(int) * ctx.nNodes);

    nThreads = 256;
    unsigned int nNodesPerBlock = nThreads * 64;

    nBlocks = 1 + (ctx.nNodes - 1) / nNodesPerBlock;

    kernels::node_phi_binning_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_sp_params, ctx.d_node_phi_index, nNodesPerBlock, ctx.nNodes,
        m_config.n_phi_bins);

    hipStreamSynchronize(stream);

    hipMalloc(&ctx.d_node_eta_index, sizeof(int) * ctx.nNodes);

    nBlocks = m_config.nLayers;

    kernels::node_eta_binning_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_sp_params, ctx.d_layer_info, ctx.d_layer_geo,
        ctx.d_node_eta_index, ctx.d_layerCounts, m_config.nLayers);

    hipStreamSynchronize(stream);

    hipFree(ctx.d_layerCounts);
    hipFree(ctx.d_layer_info);
    hipFree(ctx.d_layer_geo);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR(
            "eta-phi binning: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }
    size_t hist_size = sizeof(int) * m_config.n_eta_bins * m_config.n_phi_bins;
    hipMalloc(&ctx.d_eta_phi_histo, hist_size);
    hipMemsetAsync(ctx.d_eta_phi_histo, 0, hist_size, stream);
    hipMalloc(&ctx.d_phi_cusums, hist_size);

    nBlocks = 1 + (ctx.nNodes - 1) / nNodesPerBlock;

    kernels::eta_phi_histo_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_node_phi_index, ctx.d_node_eta_index, ctx.d_eta_phi_histo,
        nNodesPerBlock, ctx.nNodes, m_config.n_phi_bins);

    hipStreamSynchronize(stream);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR(
            "eta-phi histo: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    hipMalloc(&ctx.d_eta_node_counter, sizeof(int) * m_config.n_eta_bins);

    unsigned int nBinsPerBlock = 128;

    nThreads = nBinsPerBlock;

    nBlocks = 1 + (m_config.n_eta_bins - 1) / nBinsPerBlock;

    kernels::eta_phi_counting_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_eta_phi_histo, ctx.d_eta_node_counter, ctx.d_phi_cusums,
        nBinsPerBlock, m_config.n_eta_bins, m_config.n_phi_bins);

    hipStreamSynchronize(stream);
    hipFree(ctx.d_eta_phi_histo);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR(
            "eta-phi counting: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    std::unique_ptr<int[]> eta_sums =
        std::make_unique<int[]>(m_config.n_eta_bins);

    hipMemcpyAsync(&eta_sums[0], &ctx.d_eta_node_counter[0],
                    sizeof(int) * m_config.n_eta_bins, hipMemcpyDeviceToHost,
                    stream);

    hipStreamSynchronize(stream);

    for (unsigned int k = 0; k < m_config.n_eta_bins; k++) {
        eta_sums[k + 1] += eta_sums[k];
    }
    // send back
    hipMemcpyAsync(&ctx.d_eta_node_counter[0], &eta_sums[0],
                    sizeof(int) * m_config.n_eta_bins, hipMemcpyHostToDevice,
                    stream);

    ctx.h_eta_bin_views = std::make_unique<int[]>(2 * m_config.n_eta_bins);

    for (unsigned view_idx = 0; view_idx < m_config.n_eta_bins; view_idx++) {
        unsigned int pos = 2 * view_idx;
        ctx.h_eta_bin_views[pos] = (view_idx == 0) ? 0 : eta_sums[view_idx - 1];
        ctx.h_eta_bin_views[pos + 1] = eta_sums[view_idx];
    }
    eta_sums.reset();

    hipStreamSynchronize(stream);

    kernels::eta_phi_prefix_sum_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_eta_node_counter, ctx.d_phi_cusums, nBinsPerBlock,
        m_config.n_eta_bins, m_config.n_phi_bins);

    hipStreamSynchronize(stream);
    hipFree(ctx.d_eta_node_counter);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR(
            "eta-phi cusum: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    hipMalloc(&ctx.d_node_params, 5 * sizeof(float) * ctx.nNodes);
    hipMalloc(&ctx.d_node_index, sizeof(int) * ctx.nNodes);

    nThreads = 256;
    nNodesPerBlock = nThreads * 64;

    nBlocks = 1 + (ctx.nNodes - 1) / nNodesPerBlock;

    kernels::node_sorting_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_sp_params, ctx.d_node_eta_index, ctx.d_node_phi_index,
        ctx.d_phi_cusums, ctx.d_node_params, ctx.d_node_index,
        ctx.d_original_sp_idx, ctx.d_graph_building_params, nNodesPerBlock,
        ctx.nNodes, m_config.n_phi_bins);

    hipStreamSynchronize(stream);
    hipFree(ctx.d_sp_params);
    hipFree(ctx.d_original_sp_idx);
    hipFree(ctx.d_phi_cusums);
    hipFree(ctx.d_node_eta_index);
    hipFree(ctx.d_node_phi_index);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR("node sorting: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    hipMalloc(&ctx.d_eta_bin_views, sizeof(int2) * m_config.n_eta_bins);
    hipMalloc(&ctx.d_bin_rads, sizeof(float2) * m_config.n_eta_bins);

    hipMemcpyAsync(&ctx.d_eta_bin_views[0], ctx.h_eta_bin_views.get(),
                    2 * m_config.n_eta_bins * sizeof(int),
                    hipMemcpyHostToDevice, stream);

    hipStreamSynchronize(stream);

    nBinsPerBlock = 128;

    nThreads = nBinsPerBlock;

    nBlocks = 1 + (m_config.n_eta_bins - 1) / nBinsPerBlock;

    kernels::minmax_rad_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_eta_bin_views, ctx.d_node_params, ctx.d_bin_rads, nBinsPerBlock,
        m_config.n_eta_bins);

    hipStreamSynchronize(stream);
    hipFree(ctx.d_eta_bin_views);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR("node sorting: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }
    ctx.h_bin_rads = std::make_unique<float[]>(2 * m_config.n_eta_bins);

    hipMemcpyAsync(ctx.h_bin_rads.get(), &ctx.d_bin_rads[0],
                    2 * sizeof(float) * m_config.n_eta_bins,
                    hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    hipFree(ctx.d_bin_rads);

    // 2. prepare input for the graph making part of the code:

    int int_nBinPairs = 0;  // the number of eta bin pairs

    for (std::pair<unsigned int, unsigned int> binPair : m_config.binTables) {
        // loop over bin pairs defined by the layer
        // connection table and geometry settings

        int bin1_begin = ctx.h_eta_bin_views[2 * binPair.first];
        int bin1_end = ctx.h_eta_bin_views[2 * binPair.first + 1];
        // large bins will be split into smaller sub-views

        int nNodesInBin1 = bin1_end - bin1_begin;
        if (bin1_begin > bin1_end) {
            nNodesInBin1 = bin1_begin - bin1_end;
        }
        int_nBinPairs +=
            1 + (nNodesInBin1 - 1) /
                    traccc::device::gbts_consts::node_buffer_length;
    }
    unsigned int nBinPairs = static_cast<unsigned int>(int_nBinPairs);

    ctx.h_bin_pair_views = std::make_unique<int[]>(4 * nBinPairs);
    ctx.h_bin_pair_dphi = std::make_unique<float[]>(nBinPairs);

    unsigned int pairIdx = 0;
    for (std::pair<unsigned int, unsigned int> binPair : m_config.binTables) {
        float rb1 = ctx.h_bin_rads[2 * binPair.first];  // min radius

        int begin_bin1 = ctx.h_eta_bin_views[2 * binPair.first];
        int end_bin1 = ctx.h_eta_bin_views[2 * binPair.first + 1];
        // skip empty pairs
        if (begin_bin1 == end_bin1)
            continue;
        if (ctx.h_eta_bin_views[2 * binPair.second] ==
            ctx.h_eta_bin_views[2 * binPair.second + 1])
            continue;

        float rb2 = ctx.h_bin_rads[2 * binPair.second + 1];  // max radius

        // max radius of bin2 - min radius of bin1
        float maxDeltaR = std::fabs(rb2 - rb1);

        float deltaPhi = m_config.graph_building_params.min_delta_phi +
                         m_config.graph_building_params.dphi_coeff * maxDeltaR;

        if (maxDeltaR < 60) {
            deltaPhi =
                m_config.graph_building_params.min_delta_phi_low_dr +
                m_config.graph_building_params.dphi_coeff_low_dr * maxDeltaR;
        }
        // splitting large bins into more consistent sizes

        int currBegin_bin1 = begin_bin1;

        int currEnd_bin1 =
            end_bin1 < traccc::device::gbts_consts::node_buffer_length
                ? end_bin1
                : begin_bin1 + traccc::device::gbts_consts::node_buffer_length;

        for (; currEnd_bin1 < end_bin1;
             currEnd_bin1 += traccc::device::gbts_consts::node_buffer_length,
             pairIdx++) {
            unsigned int offset = 4 * pairIdx;

            ctx.h_bin_pair_views[offset] = currBegin_bin1;
            ctx.h_bin_pair_views[1 + offset] = currEnd_bin1;
            ctx.h_bin_pair_views[2 + offset] =
                ctx.h_eta_bin_views[2 * binPair.second];
            ctx.h_bin_pair_views[3 + offset] =
                ctx.h_eta_bin_views[2 * binPair.second + 1];
            ctx.h_bin_pair_dphi[pairIdx] = deltaPhi;

            currBegin_bin1 = currEnd_bin1;
        }
        currEnd_bin1 = end_bin1;

        unsigned int offset = 4 * pairIdx;

        ctx.h_bin_pair_views[offset] = currBegin_bin1;
        ctx.h_bin_pair_views[1 + offset] = currEnd_bin1;
        ctx.h_bin_pair_views[2 + offset] =
            ctx.h_eta_bin_views[2 * binPair.second];
        ctx.h_bin_pair_views[3 + offset] =
            ctx.h_eta_bin_views[2 * binPair.second + 1];
        ctx.h_bin_pair_dphi[pairIdx] = deltaPhi;
        pairIdx++;
    }
    ctx.nUsedBinPairs = pairIdx;
    if (ctx.nUsedBinPairs == 0)
        return {0, m_mr.main};
    ctx.h_eta_bin_views.reset();
    // allocate memory and copy bin pair views and phi cuts to GPU

    unsigned int data_size = ctx.nUsedBinPairs * sizeof(uint4);

    hipMalloc(&ctx.d_bin_pair_views, data_size);
    hipMemcpyAsync(&ctx.d_bin_pair_views[0], &ctx.h_bin_pair_views[0],
                    data_size, hipMemcpyHostToDevice, stream);

    data_size = ctx.nUsedBinPairs * sizeof(float);

    hipMalloc(&ctx.d_bin_pair_dphi, data_size);
    hipMemcpyAsync(&ctx.d_bin_pair_dphi[0], &ctx.h_bin_pair_dphi[0], data_size,
                    hipMemcpyHostToDevice, stream);

    hipMalloc(&ctx.d_counters, sizeof(unsigned int) * 12);
    hipMemsetAsync(ctx.d_counters, 0, sizeof(unsigned int) * 12, stream);

    hipStreamSynchronize(stream);

    // 2. Find edges between spacepoint pairs
    ctx.nMaxEdges = 8 * ctx.nNodes;
    hipMalloc(&ctx.d_edge_params, sizeof(kernels::half4) * ctx.nMaxEdges);
    hipMalloc(&ctx.d_edge_nodes, sizeof(int2) * ctx.nMaxEdges);

    hipMalloc(&ctx.d_num_incoming_edges, sizeof(int) * (ctx.nNodes + 1));
    hipMemsetAsync(ctx.d_num_incoming_edges, 0, sizeof(int) * (ctx.nNodes + 1),
                    stream);

    nBlocks = ctx.nUsedBinPairs;
    nThreads = 128;

    kernels::graphEdgeMakingKernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_bin_pair_views, ctx.d_bin_pair_dphi, ctx.d_node_params,
        ctx.d_graph_building_params, ctx.d_counters, ctx.d_edge_nodes,
        ctx.d_edge_params, ctx.d_num_incoming_edges, ctx.nMaxEdges,
        m_config.n_phi_bins);

    hipStreamSynchronize(stream);
    hipFree(ctx.d_node_params);
    hipFree(ctx.d_bin_pair_views);
    hipFree(ctx.d_bin_pair_dphi);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR("edge making: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    hipMemcpyAsync(&ctx.nEdges, ctx.d_counters, sizeof(unsigned int),
                    hipMemcpyDeviceToHost, stream);

    TRACCC_DEBUG("Created " << ctx.nEdges << " edges with a cap of "
                            << ctx.nMaxEdges);

    if (ctx.nEdges > ctx.nMaxEdges)
        ctx.nEdges = ctx.nMaxEdges;
    else if (ctx.nEdges == 0)
        return {0, m_mr.main};

    std::unique_ptr<int[]> cusum = std::make_unique<int[]>(ctx.nNodes + 1);

    data_size = (ctx.nNodes + 1) * sizeof(int);

    hipMemcpyAsync(&cusum[0], ctx.d_num_incoming_edges, data_size,
                    hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    for (unsigned int k = 0; k < ctx.nNodes; k++)
        cusum[k + 1] += cusum[k];

    hipMemcpyAsync(ctx.d_num_incoming_edges, &cusum[0], data_size,
                    hipMemcpyHostToDevice, stream);

    hipStreamSynchronize(stream);

    cusum.reset();

    // 3. link edges and nodes

    data_size = ctx.nEdges * sizeof(int);

    hipMalloc(&ctx.d_edge_links, data_size);

    nThreads = 256;
    nBlocks = 1 + (ctx.nEdges - 1) / nThreads;

    kernels::graphEdgeLinkingKernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_edge_nodes, ctx.d_edge_links, ctx.d_num_incoming_edges,
        ctx.nEdges);

    hipStreamSynchronize(stream);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR("edge linking: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    // 4. edge matching to create edge-to-edge connections

    data_size = ctx.nEdges * sizeof(unsigned char);

    hipMalloc(&ctx.d_num_neighbours, data_size);
    hipMemsetAsync(ctx.d_num_neighbours, 0, data_size, stream);

    data_size = ctx.nEdges * sizeof(int);

    hipMalloc(&ctx.d_reIndexer, data_size);
    hipMemsetAsync(ctx.d_reIndexer, 0xFF, data_size, stream);

    data_size = m_config.max_num_neighbours * ctx.nEdges * sizeof(int);
    hipMalloc(&ctx.d_neighbours, data_size);

    kernels::graphEdgeMatchingKernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_graph_building_params, ctx.d_edge_params, ctx.d_edge_nodes,
        ctx.d_num_incoming_edges, ctx.d_edge_links, ctx.d_num_neighbours,
        ctx.d_neighbours, ctx.d_reIndexer, ctx.d_counters, ctx.nEdges,
        m_config.max_num_neighbours);

    hipStreamSynchronize(stream);
    hipFree(ctx.d_num_incoming_edges);
    hipFree(ctx.d_edge_links);
    hipFree(ctx.d_edge_params);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR(
            "edge matching: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    // 5. Edge re-indexing to keep only edges involved in any connection

    kernels::edgeReIndexingKernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_reIndexer, ctx.d_counters, ctx.nEdges);

    hipStreamSynchronize(stream);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR(
            "edge re-indexing: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    unsigned int nStats[3];

    hipMemcpyAsync(&nStats[0], ctx.d_counters, 3 * sizeof(unsigned int),
                    hipMemcpyDeviceToHost, stream);

    ctx.nConnections = nStats[1];
    ctx.nConnectedEdges = nStats[2];

    TRACCC_DEBUG("created " << ctx.nConnections << " edge links, found "
                            << ctx.nConnectedEdges
                            << " connected edges for seed extraction");
    if (ctx.nConnectedEdges == 0)
        return {0, m_mr.main};

    unsigned int nIntsPerEdge = 2 + 1 + m_config.max_num_neighbours;

    data_size = ctx.nConnectedEdges * nIntsPerEdge * sizeof(int);

    hipMalloc(&ctx.d_output_graph, data_size);

    nThreads = 256;
    unsigned int nEdgesPerBlock = nThreads * 64;

    nBlocks = 1 + (ctx.nEdges - 1) / nEdgesPerBlock;

    kernels::graphCompressionKernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_node_index, ctx.d_edge_nodes, ctx.d_num_neighbours,
        ctx.d_neighbours, ctx.d_reIndexer, ctx.d_output_graph, nEdgesPerBlock,
        ctx.nEdges, m_config.max_num_neighbours);

    hipStreamSynchronize(stream);

    hipFree(ctx.d_edge_nodes);
    hipFree(ctx.d_node_index);
    hipFree(ctx.d_reIndexer);
    hipFree(ctx.d_num_neighbours);
    hipFree(ctx.d_neighbours);

    error = hipGetLastError();
    if (error != hipSuccess) {
        TRACCC_ERROR(
            "graph compression: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    // 6. Find longest segments with CCA

    data_size = ctx.nConnectedEdges * sizeof(int);

    hipMalloc(&ctx.d_active_edges, data_size);
    hipMemsetAsync(ctx.d_active_edges, 0xFF, data_size,
                    stream);  // initialize to -1

    data_size = 2 * ctx.nConnectedEdges * sizeof(unsigned char);

    // old levels and new levels are kept in opposite halves of the array
    hipMalloc(&ctx.d_levels, data_size);
    // initalize to 1 so level counts the maxium number of edge segments
    //  for a seed originating at the edge
    hipMemsetAsync(ctx.d_levels, 0x1, data_size, stream);

    hipMalloc(&ctx.d_outgoing_paths, ctx.nConnectedEdges * sizeof(short2));

    unsigned int nEdgesLeft = ctx.nConnectedEdges;

    hipMemcpyAsync(&ctx.d_counters[3], &nEdgesLeft, sizeof(unsigned int),
                    hipMemcpyHostToDevice, stream);

    if (nEdgesLeft == 0)
        return {0, m_mr.main};

    hipStreamSynchronize(stream);

    nThreads = 128;
    nBlocks = 1 + (nEdgesLeft - 1) / nThreads;
    for (int iter = 0; iter < traccc::device::gbts_consts::max_cca_iter;
         iter++) {
        kernels::CCA_IterationKernel<<<nBlocks, nThreads, 0, stream>>>(
            ctx.d_output_graph, ctx.d_levels, ctx.d_active_edges,
            ctx.d_outgoing_paths, ctx.d_counters, iter, ctx.nConnectedEdges,
            m_config.max_num_neighbours, m_config.minLevel);

        hipStreamSynchronize(stream);
    }

    hipStreamSynchronize(stream);

    hipFree(ctx.d_active_edges);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR(
            "message-passing CCA: CUDA error: " << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    nThreads = 128;
    nBlocks = 1 + (ctx.nConnectedEdges - 1) / nThreads;

    hipStreamSynchronize(stream);

    kernels::count_terminus_edges<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_path_store, ctx.d_outgoing_paths, ctx.d_counters,
        ctx.nConnectedEdges);

    hipStreamSynchronize(stream);

    // nPaths to terminus, nTerminusEdges
    int path_sizes[2];
    hipMemcpyAsync(&path_sizes, &ctx.d_counters[6], 2 * sizeof(unsigned int),
                    hipMemcpyDeviceToHost, stream);

    TRACCC_DEBUG(path_sizes[0] << "nPath | nTerminus " << path_sizes[1]);

    hipMalloc(&ctx.d_path_store,
               (path_sizes[0] + path_sizes[1]) * sizeof(int2));
    hipMalloc(&ctx.d_seed_proposals, path_sizes[0] * sizeof(int2));
    hipMalloc(&ctx.d_seed_ambiguity, path_sizes[0] * sizeof(char));

    hipMalloc(&ctx.d_edge_bids,
               ctx.nConnectedEdges * sizeof(unsigned long long int));
    hipMemsetAsync(ctx.d_edge_bids, 0,
                    ctx.nConnectedEdges * sizeof(unsigned long long int),
                    stream);

    kernels::add_terminus_to_path_store<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_path_store, ctx.d_outgoing_paths, ctx.d_counters,
        ctx.nConnectedEdges);

    nBlocks = 1 + (path_sizes[1] - 1) / 16;
    nThreads = 128;

    kernels::fill_path_store<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_path_store, ctx.d_output_graph, ctx.d_levels, ctx.d_counters,
        path_sizes[1], m_config.max_num_neighbours);

    nThreads = 128;
    nBlocks = 1 + (path_sizes[0] + path_sizes[1] - 1) / nThreads;

    kernels::fit_segments<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_reducedSP, ctx.d_output_graph, ctx.d_path_store,
        ctx.d_seed_proposals, ctx.d_edge_bids, ctx.d_seed_ambiguity,
        ctx.d_levels, ctx.d_counters, path_sizes[0], path_sizes[1],
        m_config.minLevel, m_config.max_num_neighbours,
        m_config.seed_extraction_params);

    int nProps = 0;
    hipMemcpyAsync(&nProps, &ctx.d_counters[8], sizeof(unsigned int),
                    hipMemcpyDeviceToHost, stream);

    TRACCC_DEBUG("nProps " << nProps);

    hipStreamSynchronize(stream);

    hipFree(ctx.d_levels);
    hipFree(ctx.d_outgoing_paths);
    hipFree(ctx.d_reducedSP);

    if (nProps == 0) {
        return {0, m_mr.main};
    }

    nThreads = 128;
    nBlocks = 1 + (nProps - 1) / nThreads;

    kernels::reset_edge_bids<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_path_store, ctx.d_seed_proposals, ctx.d_edge_bids,
        ctx.d_seed_ambiguity, ctx.d_counters, -1);

    for (int round = 0; round < 5; ++round) {

        hipMemsetAsync(ctx.d_edge_bids, 0,
                        ctx.nConnectedEdges * sizeof(unsigned long long int),
                        stream);

        kernels::seeds_rebid_for_edges<<<nBlocks, nThreads, 0, stream>>>(
            ctx.d_path_store, ctx.d_seed_proposals, ctx.d_edge_bids,
            ctx.d_seed_ambiguity, nProps);

        kernels::reset_edge_bids<<<nBlocks, nThreads, 0, stream>>>(
            ctx.d_path_store, ctx.d_seed_proposals, ctx.d_edge_bids,
            ctx.d_seed_ambiguity, ctx.d_counters, round);
    }

    hipFree(ctx.d_edge_bids);
    hipFree(ctx.d_counters);
    hipFree(ctx.d_graph_building_params);

    // 8. convert to 3sp seeds and make output buffer

    edm::seed_collection::buffer output_seeds(
        nProps, m_mr.main, vecmem::data::buffer_type::resizable);
    m_copy.get().setup(output_seeds)->ignore();

    kernels::gbts_seed_conversion_kernel<<<nBlocks, nThreads, 0, stream>>>(
        ctx.d_seed_proposals, ctx.d_seed_ambiguity, ctx.d_path_store,
        ctx.d_output_graph, output_seeds, nProps, m_config.max_num_neighbours);

    hipStreamSynchronize(stream);

    hipFree(ctx.d_output_graph);
    hipFree(ctx.d_path_store);
    hipFree(ctx.d_seed_proposals);
    hipFree(ctx.d_seed_ambiguity);

    error = hipGetLastError();

    if (error != hipSuccess) {
        TRACCC_ERROR("seed-extracting kalman filter: CUDA error: "
                     << hipGetErrorString(error));
        return {0, m_mr.main};
    }

    TRACCC_DEBUG("GBTS found " << ctx.nSeeds << " seeds");
    return output_seeds;
}

}  // namespace traccc::cuda
