#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "../utils/cuda_error_handling.hpp"
#include "../utils/utils.hpp"
#include "./kernels/GbtsNodesMakingKernels.cuh"
//#include "./kernels/GbtsGraphMaking.cuh"
//#include "./kernels/GbtsGraphProcessing.cuh"

#include "traccc/cuda/gbts_seeding/gbts_seeding_algorithm.hpp"

namespace traccc::cuda {

struct gpu_gbts_layerInfo {
	bool* isEndcap;
	int*

};

struct gbts_ctx {
	//counters
	unsigned int nSp{};
	unsigned int nEdges{};
	unsigned int nConnectedEdges{};
	unsigned int nSeeds{};
	//nEdges, nConnections, nConnectedEdges, .., nSeeds
	unsigned int* d_counters;	

	//node making
	unsigned int* d_layerCounts{};
	unsigned short* d_spacepointsLayer{};
	//begin_idx + 1 for the surfaceToLayerMap or -layerBin if one to one
	int* d_volumeToLayerMap{};	
	//surface_id, layerBin
	uint2* d_surfaceToLayerMap{};
	bool* d_layerIEndcap{};
	
	//x,y,z,cluster width in eta
	float4* d_reducedSP{};
	//output of layer binning
	float4* d_sp_params{};	
	
	int4* d_layerInfo{};
	float2* d_layerGeo{};

	//GraphMaking

	//GraphProccessing

	//output
};

gbts_seeding_algorithm::gbts_seeding_algorithm(const gbts_seedfinder_config& cfg, traccc::memory_resource& mr, vecmem::copy& copy, stream& str, std::unique_ptr<const Logger> logger)
                                               : messaging(logger->clone()), m_config(cfg), m_mr(mr), m_copy(copy), m_stream(str) {}

gbts_seeding_algorithm::output_type gbts_seeding_algorithm::operator()(const traccc::edm::spacepoint_collection::const_view& spacepoints, const traccc::measurement_collection_types::const_view& measurements) const {

	gbts_seeding_algorithm::output_type output_seeds;

	gbts_ctx ctx;

	hipStream_t stream = details::get_stream(m_stream);

	//0. bin spacepoints by layer(disk) or any other maping supplied to the config.m_surfaceToLayerMap
	ctx.nSp = m_copy.get().get_size(spacepoints); //why is get needed?

	unsigned int nThreads = 1024;
	unsigned int nBlocks = 1+(ctx.nSp-1)/nThreads;

	hipMalloc(&ctx.d_layerCounts, m_config.nLayers*sizeof(unsigned int));	
	hipMalloc(&ctx.d_spacepointsLayer, ctx.nSp*sizeof(unsigned char));	
	hipMalloc(&ctx.d_reducedSP, ctx.nSp*sizeof(float4));	

	hipMalloc(&ctx.d_volumeToLayerMap, sizeof(int)*m_config.maxVolIndex);	
	hipMalloc(&ctx.d_surfaceToLayerMap, sizeof(uint2)*m_config.surfaceMapSize);	

	hipMemcpyAsync(ctx.d_volumeToLayerMap, m_config.volumeToLayerMap.get(), sizeof(int)*m_config.maxVolIndex, hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(ctx.d_surfaceToLayerMap, m_config.surfaceToLayerMap.data(), sizeof(uint2)*m_config.surfaceMapSize, hipMemcpyHostToDevice, stream);

	hipMalloc(&ctx.d_layerIsEndcap, sizeof(bool)*m_config.nLayers);
	hipMemcpyAsync(ctx.d_layerIsEndcap, m_config.layerInfo.isEndcap.data(), sizeof(bool)*m_config.nLayers, hipMemcpyHostToDevice, stream);	
	
	kernels::count_sp_by_layer<<<nBlocks,nThreads,0,stream>>>(spacepoints,measurements,
	                            ctx.d_volumeToLayerMap,ctx.d_surfaceToLayerMap,ctx.d_layerIsEndcap, 
                                ctx.d_reducedSP, ctx.d_layerCounts, ctx.d_spacepointsLayer,
								ctx.nSp, m_config.surfaceMapSize);
	//prefix sum layerCounts
	std::unique_ptr<unsigned int[]> layerCounts = std::make_unique<unsigned int[]>(m_config.nLayers);

	hipMemcpyAsync(ctx.d_layerCounts, layerCounts.get(), m_config.nLayers*sizeof(unsigned int), hipMemcpyDeviceToHost, stream);	
	for(int layer = 1; layer<m_config.nLayers; layer++) {
		layerCounts[layer] += layerCounts[layer-1];
	}
	hipMemcpyAsync(layerCounts.get(), ctx.d_layerCounts, m_config.nLayers*sizeof(unsigned int), hipMemcpyHostToDevice, stream);	

	hipMalloc(&ctx.d_sp_params, ctx.nSp*sizeof(float4));	

	kernels::bin_sp_by_layer<<<nBlocks, nThreads, 0, stream>>>(ctx.d_sp_params, ctx.d_reducedSP, ctx.d_layerCounts, ctx.d_spacepointsLayer, ctx.nSp);	

	//1. histogram spacepoints by layer->eta->phi and convert to nodes phi,r,z,tau_min,tau_max
	//do this in config setup?
	hipMalloc(&ctx.d_layerInfo, sizeof(int2)*m_config.nLayers);
	hipMemcpyAsync(ctx.d_layerInfo, m_config.layerInfo.info.data(), sizeof(int2)*m_config.nLayers, hipMemcpyHostToDevice, stream);	
	
	hipMalloc(&ctx.d_layerGeo, sizeof(float2)*m_config.nLayers);
	hipMemcpyAsync(ctx.d_layerGeo, m_config.layerInfo.geo.data(), sizeof(float2)*m_config.nLayers, hipMemcpyHostToDevice, stream);	


	//2. Find edges between spacepoint pairs

	//3. Link edges into graph

	//4. Prune unlinked edges from graph

	//5. Find longest segments with CCA

	//6. extract seeds, longest segment first

	return output_seeds;
}

} //namespace traccc::cuda
