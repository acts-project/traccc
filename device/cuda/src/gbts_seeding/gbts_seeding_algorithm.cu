#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "../utils/cuda_error_handling.hpp"
#include "../utils/utils.hpp"
#include "./kernels/GbtsNodesMakingKernels.cuh"
#include "./kernels/GbtsGraphMakingKernels.cuh"
//#include "./kernels/GbtsGraphProcessingKernels.cuh"

#include "traccc/cuda/gbts_seeding/gbts_seeding_algorithm.hpp"

namespace traccc::cuda {

struct gbts_ctx {
	//counters
	unsigned int nSp{};
	unsigned int nNodes{};
	unsigned int nUsedBinPairs{};
	unsigned int nMaxEdges{};

	unsigned int nEdges{};
	unsigned int nConnections{};
	unsigned int nConnectedEdges{};
	unsigned int nSeeds{};
	//nEdges, nConnections, nConnectedEdges, .., nSeeds
	unsigned int* d_counters;	
	
	//device cut values
	gbts_algo_params* d_algo_params;

	//node making and binning
	unsigned int* d_layerCounts{};
	short* d_spacepointsLayer{}; 
	short* d_volumeToLayerMap{}; //begin_idx + 1 for the surfaceToLayerMap or -layerBin if one to one
	uint2* d_surfaceToLayerMap{}; //surface_index, layerBin
	char* d_layerIsEndcap{};
	int* d_original_sp_idx{}; // conversion to original sp from post layer binning index
	int* d_node_index{}; // conversion to orignal sp/node index from post binning index
	
	//x,y,z,cluster width in eta
	float4* d_reducedSP{};
	//layer binned reducedSP
	float4* d_sp_params{};	
	
	int2* d_layer_info{};
	float2* d_layer_geo{};
	
	int* d_node_eta_index{};
	int* d_node_phi_index{};

	unsigned int* d_eta_phi_histo{};//for data binning
	unsigned int* d_phi_cusums{};//for data binning
	unsigned int* d_eta_node_counter{};//for data binning

	int2* d_eta_bin_views{};//views of the nodes
	std::unique_ptr<int[]>   h_eta_bin_views{};//eta-bin views of the node_params array
	
	float2* d_bin_rads{};//minimum and maximum r of nodes inside an eta-bin
	std::unique_ptr<float[]> h_bin_rads{};

	uint4* d_bin_pair_views{};
	std::unique_ptr<unsigned int[]> h_bin_pair_views{};

	std::unique_ptr<float[]> h_bin_pair_dphi{};
	float* d_bin_pair_dphi{};
	//node making output
	float* d_node_params{};

	//GraphMaking
	int2* d_edge_nodes{};
	kernels::half4* d_edge_params{};	

	unsigned int* d_num_incoming_edges{}; 
	int* d_edge_links{};

	unsigned char* d_num_neighbours{};
	int* d_reIndexer{};
	int* d_neighbours{};
	int* d_output_graph{};

	//GraphProccessing

	//output
};

gbts_seeding_algorithm::gbts_seeding_algorithm(const gbts_seedfinder_config& cfg, traccc::memory_resource& mr, vecmem::copy& copy, stream& str, std::unique_ptr<const Logger> logger)
                                               : messaging(logger->clone()), m_config(cfg), m_mr(mr), m_copy(copy), m_stream(str) {}

gbts_seeding_algorithm::output_type gbts_seeding_algorithm::operator()(const traccc::edm::spacepoint_collection::const_view& spacepoints, const traccc::measurement_collection_types::const_view& measurements) const {
	
    edm::seed_collection::buffer output_seeds(0, m_mr.main, vecmem::data::buffer_type::resizable);

	gbts_ctx ctx;

	hipStream_t stream = details::get_stream(m_stream);
	
	hipMalloc(&ctx.d_algo_params, sizeof(m_config.algo_params));
	hipMemcpyAsync(ctx.d_algo_params, &m_config.algo_params , sizeof(m_config.algo_params), hipMemcpyHostToDevice);


	//0. bin spacepoints by layer(disk) or any other maping supplied to the config.m_surfaceToLayerMap
	ctx.nSp = m_copy.get().get_size(spacepoints); //why is get needed?
	if(ctx.nSp == 0) return output_seeds;

	unsigned int nThreads = 1024;
	unsigned int nBlocks = 1+(ctx.nSp-1)/nThreads;
	
	hipMalloc(&ctx.d_layerCounts, (m_config.nLayers+1)*sizeof(unsigned int));
	hipMemset(ctx.d_layerCounts, 0 , (m_config.nLayers+1)*sizeof(unsigned int));	
	
	hipMalloc(&ctx.d_spacepointsLayer, ctx.nSp*sizeof(short));	
	hipMalloc(&ctx.d_reducedSP, ctx.nSp*sizeof(float4));	
	
	hipMalloc(&ctx.d_volumeToLayerMap, sizeof(short)*m_config.volumeMapSize);		
	hipMemcpyAsync(ctx.d_volumeToLayerMap, m_config.volumeToLayerMap.get(), sizeof(short)*m_config.volumeMapSize, hipMemcpyHostToDevice, stream);
	
	if(m_config.surfaceMapSize != 0) {
		hipMalloc(&ctx.d_surfaceToLayerMap, sizeof(uint2)*m_config.surfaceMapSize);	
		hipMemcpyAsync(ctx.d_surfaceToLayerMap, m_config.surfaceToLayerMap.data(), sizeof(uint2)*m_config.surfaceMapSize, hipMemcpyHostToDevice, stream);
	} //may be zero and correct, volumeMapSize and nLayers are checked at config

	hipMalloc(&ctx.d_layerIsEndcap, sizeof(char)*m_config.nLayers);
	hipMemcpyAsync(ctx.d_layerIsEndcap, m_config.layerInfo.isEndcap.data(), sizeof(char)*m_config.nLayers, hipMemcpyHostToDevice, stream);	
	
	kernels::count_sp_by_layer<<<nBlocks,nThreads,0,stream>>>(spacepoints,measurements,
								ctx.d_volumeToLayerMap,ctx.d_surfaceToLayerMap,ctx.d_layerIsEndcap, 
                                ctx.d_reducedSP, ctx.d_layerCounts, ctx.d_spacepointsLayer,
								ctx.nSp, m_config.volumeMapSize, m_config.surfaceMapSize);

	hipStreamSynchronize(stream);
	
	hipFree(ctx.d_volumeToLayerMap);
	hipFree(ctx.d_surfaceToLayerMap);
	hipFree(ctx.d_layerIsEndcap);

	//prefix sum layerCounts
	std::unique_ptr<unsigned int[]> layerCounts = std::make_unique<unsigned int[]>(m_config.nLayers+1);

	hipMemcpyAsync(layerCounts.get(), ctx.d_layerCounts, (m_config.nLayers+1)*sizeof(unsigned int), hipMemcpyDeviceToHost, stream);	
	for(int layer = 1; layer < m_config.nLayers + 1; layer++) {
		layerCounts[layer] += layerCounts[layer-1];
	}
	hipMemcpyAsync(ctx.d_layerCounts, layerCounts.get(), m_config.nLayers*sizeof(unsigned int), hipMemcpyHostToDevice, stream);	
	ctx.nNodes = layerCounts[m_config.nLayers];
	if(ctx.nNodes == 0) return output_seeds;
	layerCounts.reset();
	
	hipMalloc(&ctx.d_sp_params, ctx.nSp*sizeof(float4));	
	hipMalloc(&ctx.d_original_sp_idx, ctx.nSp*sizeof(int));	

	kernels::bin_sp_by_layer<<<nBlocks, nThreads, 0, stream>>>(ctx.d_sp_params, ctx.d_reducedSP, ctx.d_layerCounts, ctx.d_spacepointsLayer, ctx.d_original_sp_idx, ctx.nSp);	
   
	hipStreamSynchronize(stream);
	hipError_t error = hipGetLastError();

	if(error != hipSuccess) {
		TRACCC_ERROR("spacepoint layer binning: CUDA error: " << hipGetErrorString(error));
		return output_seeds;
	}

	hipFree(ctx.d_spacepointsLayer);

	//1. histogram spacepoints by layer->eta->phi and convert to nodes phi,r,z,tau_min,tau_max
	//do this in config setup?
	hipMalloc(&ctx.d_layer_info, sizeof(int2)*m_config.nLayers);
	hipMemcpyAsync(ctx.d_layer_info, m_config.layerInfo.info.data(), sizeof(int2)*m_config.nLayers, hipMemcpyHostToDevice, stream);	
	
	hipMalloc(&ctx.d_layer_geo, sizeof(float2)*m_config.nLayers);
	hipMemcpyAsync(ctx.d_layer_geo, m_config.layerInfo.geo.data(), sizeof(float2)*m_config.nLayers, hipMemcpyHostToDevice, stream);	
	
	hipMalloc(&ctx.d_node_phi_index, sizeof(int)*ctx.nNodes);

	nThreads = 256;
	int nNodesPerBlock = nThreads*64;
     
	nBlocks = 1+(ctx.nNodes-1)/nNodesPerBlock;
   
	kernels::node_phi_binning_kernel<<<nBlocks, nThreads, 0, stream>>>(ctx.d_sp_params, ctx.d_node_phi_index, nNodesPerBlock, ctx.nNodes, m_config.n_phi_bins);

	hipStreamSynchronize(stream);
	
	hipMalloc(&ctx.d_node_eta_index, sizeof(int)*ctx.nNodes);

	nBlocks = m_config.nLayers;

	kernels::node_eta_binning_kernel<<<nBlocks, nThreads, 0, stream>>>(ctx.d_sp_params, ctx.d_layer_info, ctx.d_layer_geo, ctx.d_node_eta_index, ctx.d_layerCounts ,m_config.nLayers);
	
	hipStreamSynchronize(stream);
	
	hipFree(ctx.d_layerCounts);
	hipFree(ctx.d_layer_info);
	hipFree(ctx.d_layer_geo);
	
	error = hipGetLastError();

	if(error != hipSuccess) {
		TRACCC_ERROR("eta-phi binning: CUDA error: " << hipGetErrorString(error));
		return output_seeds;
	}
	
	unsigned int hist_size = static_cast<unsigned int>(sizeof(unsigned int))*m_config.n_eta_bins*m_config.n_phi_bins;
	hipMalloc(&ctx.d_eta_phi_histo, hist_size);
	hipMemset(ctx.d_eta_phi_histo, 0, hist_size); 
	hipMalloc(&ctx.d_phi_cusums, hist_size);	

    nBlocks = 1 + (ctx.nNodes-1)/nNodesPerBlock;

    kernels::eta_phi_histo_kernel<<<nBlocks, nThreads, 0, stream>>>(ctx.d_node_phi_index, ctx.d_node_eta_index, ctx.d_eta_phi_histo, nNodesPerBlock, ctx.nNodes, m_config.n_phi_bins);

    hipStreamSynchronize(stream);
 
    error = hipGetLastError();

    if(error != hipSuccess) {
        TRACCC_ERROR("eta-phi histo: CUDA error: " << hipGetErrorString(error));
        return output_seeds;
    }
	
	hipMalloc(&ctx.d_eta_node_counter, sizeof(unsigned int)*m_config.n_eta_bins);

    int nBinsPerBlock = 128;
        
    nThreads = nBinsPerBlock;

    nBlocks = 1 + (m_config.n_eta_bins - 1)/nBinsPerBlock;

    kernels::eta_phi_counting_kernel<<<nBlocks, nThreads, 0, stream>>>(ctx.d_eta_phi_histo, ctx.d_eta_node_counter, ctx.d_phi_cusums, nBinsPerBlock, m_config.n_eta_bins, m_config.n_phi_bins);

    hipStreamSynchronize(stream);
	hipFree(ctx.d_eta_phi_histo);	

    error = hipGetLastError();

    if(error != hipSuccess) {
        TRACCC_ERROR("eta-phi counting: CUDA error: " << hipGetErrorString(error));
        return output_seeds;
    }
	
	std::unique_ptr<unsigned int[]> eta_sums = std::make_unique<unsigned int[]>(m_config.n_eta_bins);

	hipMemcpyAsync(&eta_sums[0], &ctx.d_eta_node_counter[0], sizeof(unsigned int)*m_config.n_eta_bins, hipMemcpyDeviceToHost, stream);

	hipStreamSynchronize(stream);

	for(int k=1;k<m_config.n_eta_bins;k++) eta_sums[k] += eta_sums[k-1];

	//send back
	hipMemcpyAsync(&ctx.d_eta_node_counter[0], &eta_sums[0], sizeof(unsigned int)*m_config.n_eta_bins, hipMemcpyHostToDevice, stream);

	ctx.h_eta_bin_views = std::make_unique<int[]>(2*m_config.n_eta_bins);

	for(int view_idx = 0; view_idx < m_config.n_eta_bins; view_idx++) {
	   int pos = 2*view_idx;
	   ctx.h_eta_bin_views[pos]   = (view_idx == 0) ? 0 : eta_sums[view_idx-1];
	   ctx.h_eta_bin_views[pos+1] = eta_sums[view_idx];
	} 
	eta_sums.reset();

	hipStreamSynchronize(stream);

	kernels::eta_phi_prefix_sum_kernel<<<nBlocks, nThreads, 0, stream>>>(ctx.d_eta_node_counter, ctx.d_phi_cusums, nBinsPerBlock, m_config.n_eta_bins, m_config.n_phi_bins);

	hipStreamSynchronize(stream);
    hipFree(ctx.d_eta_node_counter);

	error = hipGetLastError();

	if(error != hipSuccess) {
	   TRACCC_ERROR("eta-phi cusum: CUDA error: " << hipGetErrorString(error));
	   return output_seeds;
	}
		
	hipMalloc(&ctx.d_node_params, 5*sizeof(float)*ctx.nNodes);
	hipMalloc(&ctx.d_node_index, sizeof(int)*ctx.nNodes);

	nThreads = 256;
	nNodesPerBlock = nThreads*64;
	   
	nBlocks = 1 + (ctx.nNodes-1)/nNodesPerBlock;

	kernels::node_sorting_kernel<<<nBlocks, nThreads, 0, stream>>>(ctx.d_sp_params, ctx.d_node_eta_index, ctx.d_node_phi_index, 
														           ctx.d_phi_cusums, ctx.d_node_params, ctx.d_node_index, ctx.d_original_sp_idx,
                                                                   nNodesPerBlock, ctx.nNodes, m_config.n_phi_bins);

	hipStreamSynchronize(stream);
	hipFree(ctx.d_sp_params);
	hipFree(ctx.d_original_sp_idx);
    hipFree(ctx.d_phi_cusums);
	hipFree(ctx.d_node_eta_index);
	hipFree(ctx.d_node_phi_index);

	error = hipGetLastError();

	if(error != hipSuccess) {
	   TRACCC_ERROR("node sorting: CUDA error: " << hipGetErrorString(error));
	   return output_seeds;
	}
	
	hipMalloc(&ctx.d_eta_bin_views, sizeof(int2)*m_config.n_eta_bins);
	hipMalloc(&ctx.d_bin_rads, sizeof(float2)*m_config.n_eta_bins);

	hipMemcpyAsync(&ctx.d_eta_bin_views[0], ctx.h_eta_bin_views.get(), 2*m_config.n_eta_bins*sizeof(int), hipMemcpyHostToDevice, stream);

	hipStreamSynchronize(stream);

	nBinsPerBlock = 128;
	   
	nThreads = nBinsPerBlock;

	nBlocks = 1 + (m_config.n_eta_bins-1)/nBinsPerBlock;

	kernels::minmax_rad_kernel<<<nBlocks, nThreads, 0, stream>>>(ctx.d_eta_bin_views, ctx.d_node_params,
															 ctx.d_bin_rads, nBinsPerBlock, m_config.n_eta_bins);

	hipStreamSynchronize(stream);
	hipFree(ctx.d_eta_bin_views);

	error = hipGetLastError();

	if(error != hipSuccess) {
	   TRACCC_ERROR("node sorting: CUDA error: " << hipGetErrorString(error));
	   return output_seeds;
	}

	ctx.h_bin_rads = std::make_unique<float[]>(2*m_config.n_eta_bins);

	hipMemcpyAsync(ctx.h_bin_rads.get(), &ctx.d_bin_rads[0], 2*sizeof(float)*m_config.n_eta_bins, hipMemcpyDeviceToHost, stream);

	hipStreamSynchronize(stream);

	hipFree(ctx.d_bin_rads);

	//2. prepare input for the graph making part of the code:

	unsigned int nBinPairs = 0;//the number of eta bin pairs

	for(std::pair<int, int> binPair : m_config.binTables) {//loop over bin pairs defined by the layer connection table and geometry settings

	   int bin1_begin = ctx.h_eta_bin_views[2*binPair.first];
	   int bin1_end   = ctx.h_eta_bin_views[2*binPair.first+1];

	   //large bins will be split into smaller sub-views
	   
	   unsigned int nNodesInBin1 = bin1_end - bin1_begin;

	   nBinPairs += 1 + (nNodesInBin1-1)/traccc::device::node_buffer_length;
	}

	ctx.h_bin_pair_views = std::make_unique<unsigned int[]>(4*nBinPairs);
	ctx.h_bin_pair_dphi  = std::make_unique<float[]>(nBinPairs);

	int pairIdx = 0;
	for(std::pair<int, int> binPair : m_config.binTables) {
	   
	   float rb1 = ctx.h_bin_rads[2*binPair.first];//min radius

	   unsigned int begin_bin1 = ctx.h_eta_bin_views[2*binPair.first];
	   unsigned int end_bin1    = ctx.h_eta_bin_views[2*binPair.first+1];
	   //skip empty pairs
	   if(begin_bin1 == end_bin1) continue;
	   if(ctx.h_eta_bin_views[2*binPair.second] == ctx.h_eta_bin_views[2*binPair.second+1]) continue;

	   float rb2 = ctx.h_bin_rads[2*binPair.second+1];//max radius
	   
	   float maxDeltaR = std::fabs(rb2 - rb1);// max radius of bin2 - min radius of bin1
			   
	   float deltaPhi = m_config.algo_params.min_delta_phi + m_config.algo_params.dphi_coeff*maxDeltaR;
	   if(maxDeltaR < 60) deltaPhi = m_config.algo_params.min_delta_phi_low_dr + m_config.algo_params.dphi_coeff_low_dr*maxDeltaR;

	   //splitting large bins into more consistent sizes
			   
	   unsigned int currBegin_bin1 = begin_bin1;

	   unsigned int currEnd_bin1 = end_bin1 < traccc::device::node_buffer_length ? end_bin1 : begin_bin1 + traccc::device::node_buffer_length;
	   
	   for(;currEnd_bin1 < end_bin1; currEnd_bin1 += traccc::device::node_buffer_length, pairIdx++) {
		   unsigned int offset = 4*pairIdx;
		   
		   ctx.h_bin_pair_views[offset] = currBegin_bin1;
		   ctx.h_bin_pair_views[1 + offset] = currEnd_bin1;
		   ctx.h_bin_pair_views[2 + offset] = ctx.h_eta_bin_views[2*binPair.second];
		   ctx.h_bin_pair_views[3 + offset] = ctx.h_eta_bin_views[2*binPair.second + 1];
		   ctx.h_bin_pair_dphi[pairIdx]     = deltaPhi;
						   
		   currBegin_bin1 = currEnd_bin1;
	   }
	   currEnd_bin1 = end_bin1;
	   
	   unsigned int offset = 4*pairIdx;

	   ctx.h_bin_pair_views[offset]     = currBegin_bin1;
	   ctx.h_bin_pair_views[1 + offset] = currEnd_bin1;
	   ctx.h_bin_pair_views[2 + offset] = ctx.h_eta_bin_views[2*binPair.second];
	   ctx.h_bin_pair_views[3 + offset] = ctx.h_eta_bin_views[2*binPair.second + 1];
	   ctx.h_bin_pair_dphi[pairIdx]     = deltaPhi;
	   pairIdx++;
	   
	}
	ctx.nUsedBinPairs = pairIdx;
	if(pairIdx == 0) return output_seeds;
	ctx.h_eta_bin_views.reset();	

	// allocate memory and copy bin pair views and phi cuts to GPU

	size_t data_size = ctx.nUsedBinPairs*4*sizeof(unsigned int);
	   
	hipMalloc(&ctx.d_bin_pair_views, data_size);
	hipMemcpyAsync(&ctx.d_bin_pair_views[0], &ctx.h_bin_pair_views[0], data_size, hipMemcpyHostToDevice, stream);

	data_size = ctx.nUsedBinPairs*sizeof(float);

	hipMalloc(&ctx.d_bin_pair_dphi, data_size);
	hipMemcpyAsync(&ctx.d_bin_pair_dphi[0], &ctx.h_bin_pair_dphi[0], data_size, hipMemcpyHostToDevice, stream);

	hipMalloc(&ctx.d_counters, sizeof(unsigned int)*12);
	hipMemset(ctx.d_counters, 0, 12*sizeof(unsigned int));

	hipStreamSynchronize(stream);
	
	//2. Find edges between spacepoint pairs
	ctx.nMaxEdges = 7*ctx.nNodes;
	hipMalloc(&ctx.d_edge_params, sizeof(kernels::half4)*ctx.nMaxEdges);
	hipMalloc(&ctx.d_edge_nodes, sizeof(int2)*ctx.nMaxEdges);
	hipMalloc(&ctx.d_num_incoming_edges, sizeof(unsigned int)*(ctx.nNodes+1));


	nBlocks = ctx.nUsedBinPairs;
	nThreads = 128;

	kernels::graphEdgeMakingKernel_ITk<<<nBlocks, nThreads, 0, stream>>>(ctx.d_bin_pair_views,
													ctx.d_bin_pair_dphi, ctx.d_node_params,
													ctx.d_algo_params, ctx.d_counters, ctx.d_edge_nodes, 
													ctx.d_edge_params, ctx.d_num_incoming_edges, ctx.nMaxEdges, m_config.n_phi_bins);

	hipStreamSynchronize(stream);
	hipFree(ctx.d_node_params);
	hipFree(ctx.d_bin_pair_views);
	hipFree(ctx.d_bin_pair_dphi);	

	error = hipGetLastError();

	if(error != hipSuccess) {
	   TRACCC_ERROR("edge making: CUDA error: " << hipGetErrorString(error));
	   return output_seeds;
	}

	hipMemcpyAsync(&ctx.nEdges, ctx.d_counters, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);

	TRACCC_INFO("Created " << ctx.nEdges << " edges under a cap of " << ctx.nMaxEdges);
	   
	if(ctx.nEdges >= ctx.nMaxEdges) ctx.nEdges = ctx.nMaxEdges-1;
	else if(ctx.nEdges == 0) return output_seeds;

	std::unique_ptr<unsigned int[]> cusum = std::make_unique<unsigned int[]>(ctx.nNodes+1);

	data_size = (ctx.nNodes+1)*sizeof(unsigned int);

	hipMemcpyAsync(&cusum[0], ctx.d_num_incoming_edges, data_size, hipMemcpyDeviceToHost, stream);

	hipStreamSynchronize(stream);

	for(int k=0;k<ctx.nNodes;k++) cusum[k+1] += cusum[k];

	hipMemcpyAsync(ctx.d_num_incoming_edges, &cusum[0], data_size, hipMemcpyHostToDevice, stream);

	cusum.reset();

	hipStreamSynchronize(stream);

	//3. link edges and nodes

	data_size = ctx.nEdges*sizeof(int);

	hipMalloc(&ctx.d_edge_links, data_size);

	nThreads = 256;
	nBlocks = 1 + (ctx.nEdges - 1)/nThreads;

	kernels::graphEdgeLinkingKernel_ITk<<<nBlocks, nThreads, 0, stream>>>(ctx.d_edge_nodes, 
																	 ctx.d_edge_links, ctx.d_num_incoming_edges,
																	 ctx.nEdges);

	hipStreamSynchronize(stream);
	
	error = hipGetLastError();

	if(error != hipSuccess) {
	   TRACCC_ERROR("edge linking: CUDA error: " << hipGetErrorString(error));
	   return output_seeds;
	}

	//4. edge matching to create edge-to-edge connections

	data_size = ctx.nEdges*sizeof(unsigned char); 

	hipMalloc(&ctx.d_num_neighbours, data_size);
	hipMemset(ctx.d_num_neighbours, 0, data_size);

	data_size = ctx.nEdges*sizeof(int);

	hipMalloc(&ctx.d_reIndexer, data_size);
	hipMemset(ctx.d_reIndexer, 0xFF, data_size);	

	data_size = m_config.max_num_neighbours*ctx.nEdges * sizeof(int);
	hipMalloc(&ctx.d_neighbours, data_size);	

	kernels::graphEdgeMatchingKernel_ITk<<<nBlocks, nThreads, 0, stream>>>(ctx.d_algo_params, ctx.d_edge_params,
											ctx.d_edge_nodes, ctx.d_num_incoming_edges, ctx.d_edge_links,
											ctx.d_num_neighbours, ctx.d_neighbours, ctx.d_reIndexer, ctx.d_counters, ctx.nEdges, m_config.max_num_neighbours);

	hipStreamSynchronize(stream);
	hipFree(ctx.d_edge_links);
	hipFree(ctx.d_edge_params);	

	error = hipGetLastError();

	if(error != hipSuccess) {
	   TRACCC_ERROR("edge matching: CUDA error: " << hipGetErrorString(error));
	   return output_seeds;
	}

	//5. Edge re-indexing to keep only edges involved in any connection

	kernels::edgeReIndexingKernel_ITk<<<nBlocks, nThreads, 0, stream>>>(ctx.d_reIndexer, ctx.d_counters, ctx.nEdges);

	hipStreamSynchronize(stream);

	error = hipGetLastError();

	if(error != hipSuccess) {
	   TRACCC_ERROR("edge re-indexing: CUDA error: " << hipGetErrorString(error));
	   return output_seeds;
	}

	unsigned int nStats[3];

	hipMemcpy(&nStats[0], ctx.d_counters, 3*sizeof(unsigned int), hipMemcpyDeviceToHost);

	ctx.nConnections = nStats[1];
	ctx.nConnectedEdges = nStats[2];

	TRACCC_INFO("created " << ctx.nConnections << " edge links, found " << ctx.nConnectedEdges << " connected edges for seed extraction");
	if(ctx.nConnectedEdges == 0) return output_seeds;

	int nIntsPerEdge = 2 + 1 + m_config.max_num_neighbours;

	data_size = ctx.nConnectedEdges*nIntsPerEdge*sizeof(int);

	hipMalloc(&ctx.d_output_graph, data_size);	

	nThreads = 256;
	int nEdgesPerBlock = nThreads*64;

	nBlocks = 1 + (ctx.nEdges-1)/nEdgesPerBlock;
	   
	kernels::graphCompressionKernel_ITk<<<nBlocks, nThreads, 0, stream>>>(ctx.d_sp_params, ctx.d_node_index, 
																ctx.d_edge_nodes, ctx.d_num_neighbours, ctx.d_neighbours,
																ctx.d_reIndexer, ctx.d_output_graph, nEdgesPerBlock, ctx.nEdges, m_config.max_num_neighbours);

	hipStreamSynchronize(stream);
	
	hipFree(ctx.d_edge_nodes);
	hipFree(ctx.d_node_index);
	hipFree(ctx.d_reIndexer);
	hipFree(ctx.d_num_neighbours);
	hipFree(ctx.d_neighbours);
	hipFree(ctx.d_node_index);

	error = hipGetLastError();
	if(error != hipSuccess) {
	   TRACCC_ERROR("graph compression: CUDA error: " << hipGetErrorString(error));
       return output_seeds;
	}
		

	//6. Find longest segments with CCA

	//7. extract seeds, longest segment first

	hipFree(ctx.d_reducedSP);
	hipFree(ctx.d_counters);
	hipFree(ctx.d_output_graph);	
	hipFree(ctx.d_algo_params);	

	TRACCC_INFO("ended!");

	return output_seeds;
	}

} //namespace traccc::cuda
