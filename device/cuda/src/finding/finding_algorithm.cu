/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2023-2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "../sanity/contiguous_on.cuh"
#include "../utils/barrier.hpp"
#include "../utils/cuda_error_handling.hpp"
#include "../utils/thread_id.hpp"
#include "../utils/utils.hpp"
#include "./kernels/apply_interaction.cuh"
#include "./kernels/build_tracks.cuh"
#include "./kernels/fill_sort_keys.cuh"
#include "./kernels/find_tracks.cuh"
#include "./kernels/make_barcode_sequence.cuh"
#include "./kernels/propagate_to_next_surface.cuh"
#include "./kernels/prune_tracks.cuh"
#include "traccc/cuda/finding/finding_algorithm.hpp"
#include "traccc/definitions/primitives.hpp"
#include "traccc/definitions/qualifiers.hpp"
#include "traccc/edm/device/sort_key.hpp"
#include "traccc/finding/candidate_link.hpp"
#include "traccc/geometry/detector.hpp"
#include "traccc/utils/projections.hpp"

// detray include(s).
#include <detray/detectors/bfield.hpp>
#include <detray/navigation/navigator.hpp>
#include <detray/propagator/rk_stepper.hpp>

// VecMem include(s).
#include <vecmem/containers/data/vector_buffer.hpp>
#include <vecmem/containers/data/vector_view.hpp>
#include <vecmem/containers/device_vector.hpp>
#include <vecmem/containers/jagged_device_vector.hpp>
#include <vecmem/containers/vector.hpp>
#include <vecmem/memory/unique_ptr.hpp>

// Thrust include(s).
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

// System include(s).
#include <cassert>
#include <memory_resource>
#include <vector>

namespace traccc::cuda {

template <typename stepper_t, typename navigator_t>
finding_algorithm<stepper_t, navigator_t>::finding_algorithm(
    const config_type& cfg, const traccc::memory_resource& mr,
    vecmem::copy& copy, stream& str, std::unique_ptr<const Logger> logger)
    : messaging(std::move(logger)),
      m_cfg(cfg),
      m_mr(mr),
      m_copy(copy),
      m_stream(str),
      m_warp_size(details::get_warp_size(str.device())) {}

template <typename stepper_t, typename navigator_t>
track_candidate_container_types::buffer
finding_algorithm<stepper_t, navigator_t>::operator()(
    const typename detector_type::view_type& det_view,
    const bfield_type& field_view,
    const typename measurement_collection_types::view& measurements,
    const bound_track_parameters_collection_types::buffer& seeds_buffer) const {

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // Copy setup
    m_copy.setup(seeds_buffer)->ignore();

    // The Thrust policy to use.
    auto thrust_policy =
        thrust::cuda::par_nosync(std::pmr::polymorphic_allocator(&(m_mr.main)))
            .on(stream);

    /*****************************************************************
     * Measurement Operations
     *****************************************************************/

    unsigned int n_modules;
    measurement_collection_types::const_view::size_type n_measurements =
        m_copy.get_size(measurements);

    // Get copy of barcode uniques
    measurement_collection_types::buffer uniques_buffer{n_measurements,
                                                        m_mr.main};
    m_copy.setup(uniques_buffer)->ignore();

    {
        assert(is_contiguous_on<measurement_collection_types::const_device>(
            measurement_module_projection(), m_mr.main, m_copy, m_stream,
            measurements));

        measurement_collection_types::device uniques(uniques_buffer);

        measurement* uniques_end =
            thrust::unique_copy(thrust_policy, measurements.ptr(),
                                measurements.ptr() + n_measurements,
                                uniques.begin(), measurement_equal_comp());
        m_stream.synchronize();
        n_modules = static_cast<unsigned int>(uniques_end - uniques.begin());
    }

    // Get upper bounds of unique elements
    vecmem::data::vector_buffer<unsigned int> upper_bounds_buffer{n_modules,
                                                                  m_mr.main};
    m_copy.setup(upper_bounds_buffer)->ignore();

    {
        vecmem::device_vector<unsigned int> upper_bounds(upper_bounds_buffer);

        measurement_collection_types::device uniques(uniques_buffer);

        thrust::upper_bound(thrust_policy, measurements.ptr(),
                            measurements.ptr() + n_measurements,
                            uniques.begin(), uniques.begin() + n_modules,
                            upper_bounds.begin(), measurement_sort_comp());
    }

    /*****************************************************************
     * Kernel1: Create barcode sequence
     *****************************************************************/

    vecmem::data::vector_buffer<detray::geometry::barcode> barcodes_buffer{
        n_modules, m_mr.main};
    m_copy.setup(barcodes_buffer)->ignore();

    {
        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks =
            (barcodes_buffer.size() + nThreads - 1) / nThreads;

        kernels::make_barcode_sequence<<<nBlocks, nThreads, 0, stream>>>(
            {uniques_buffer, barcodes_buffer});

        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());
    }

    const unsigned int n_seeds = m_copy.get_size(seeds_buffer);

    // Prepare input parameters with seeds
    bound_track_parameters_collection_types::buffer in_params_buffer(n_seeds,
                                                                     m_mr.main);
    m_copy.setup(in_params_buffer)->ignore();
    m_copy(vecmem::get_data(seeds_buffer), vecmem::get_data(in_params_buffer))
        ->ignore();
    vecmem::data::vector_buffer<unsigned int> param_liveness_buffer(n_seeds,
                                                                    m_mr.main);
    m_copy.setup(param_liveness_buffer)->ignore();
    m_copy.memset(param_liveness_buffer, 1)->ignore();

    // Number of tracks per seed
    vecmem::data::vector_buffer<unsigned int> n_tracks_per_seed_buffer(
        n_seeds, m_mr.main);
    m_copy.setup(n_tracks_per_seed_buffer)->ignore();

    // Create a map for links
    std::map<unsigned int, vecmem::data::vector_buffer<candidate_link>>
        link_map;

    // Create a buffer of tip links
    vecmem::data::vector_buffer<typename candidate_link::link_index_type>
        tips_buffer{m_cfg.max_num_branches_per_seed * n_seeds, m_mr.main,
                    vecmem::data::buffer_type::resizable};
    m_copy.setup(tips_buffer)->wait();

    // Link size
    std::vector<std::size_t> n_candidates_per_step;
    n_candidates_per_step.reserve(m_cfg.max_track_candidates_per_track);

    unsigned int n_in_params = n_seeds;

    for (unsigned int step = 0;
         step < m_cfg.max_track_candidates_per_track && n_in_params > 0;
         step++) {

        /*****************************************************************
         * Kernel2: Apply material interaction
         ****************************************************************/

        {
            const unsigned int nThreads = m_warp_size * 2;
            const unsigned int nBlocks =
                (n_in_params + nThreads - 1) / nThreads;

            kernels::apply_interaction<std::decay_t<detector_type>>
                <<<nBlocks, nThreads, 0, stream>>>(
                    m_cfg, {det_view, n_in_params, in_params_buffer,
                            param_liveness_buffer});
            TRACCC_CUDA_ERROR_CHECK(hipGetLastError());
        }

        /*****************************************************************
         * Kernel3: Find valid tracks
         *****************************************************************/

        unsigned int n_candidates = 0;

        {
            // Previous step
            const unsigned int prev_step = (step == 0 ? 0 : step - 1);

            // Buffer for kalman-updated parameters spawned by the measurement
            // candidates
            const unsigned int n_max_candidates =
                n_in_params * m_cfg.max_num_branches_per_surface;

            bound_track_parameters_collection_types::buffer
                updated_params_buffer(
                    n_in_params * m_cfg.max_num_branches_per_surface,
                    m_mr.main);
            m_copy.setup(updated_params_buffer)->ignore();

            vecmem::data::vector_buffer<unsigned int> updated_liveness_buffer(
                n_in_params * m_cfg.max_num_branches_per_surface, m_mr.main);
            m_copy.setup(updated_liveness_buffer)->ignore();

            // Create the link map
            link_map[step] = {n_in_params * m_cfg.max_num_branches_per_surface,
                              m_mr.main};
            m_copy.setup(link_map[step])->ignore();

            const unsigned int nThreads = m_warp_size * 2;
            const unsigned int nBlocks =
                (n_in_params + nThreads - 1) / nThreads;

            vecmem::unique_alloc_ptr<unsigned int> n_candidates_device =
                vecmem::make_unique_alloc<unsigned int>(m_mr.main);
            TRACCC_CUDA_ERROR_CHECK(hipMemsetAsync(
                n_candidates_device.get(), 0, sizeof(unsigned int), stream));

            kernels::find_tracks<std::decay_t<detector_type>>
                <<<nBlocks, nThreads,
                   nThreads * sizeof(unsigned int) +
                       2 * nThreads *
                           sizeof(std::pair<unsigned int, unsigned int>),
                   stream>>>(
                    m_cfg, {det_view, measurements, in_params_buffer,
                            param_liveness_buffer, n_in_params, barcodes_buffer,
                            upper_bounds_buffer, link_map[prev_step], step,
                            n_max_candidates, updated_params_buffer,
                            updated_liveness_buffer, link_map[step],
                            n_candidates_device.get()});
            TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

            std::swap(in_params_buffer, updated_params_buffer);
            std::swap(param_liveness_buffer, updated_liveness_buffer);

            TRACCC_CUDA_ERROR_CHECK(hipMemcpyAsync(
                &n_candidates, n_candidates_device.get(), sizeof(unsigned int),
                hipMemcpyDeviceToHost, stream));

            m_stream.synchronize();
        }

        if (n_candidates > 0) {
            /*****************************************************************
             * Kernel4: Get key and value for parameter sorting
             *****************************************************************/

            vecmem::data::vector_buffer<unsigned int> param_ids_buffer(
                n_candidates, m_mr.main);
            m_copy.setup(param_ids_buffer)->ignore();

            {
                vecmem::data::vector_buffer<device::sort_key> keys_buffer(
                    n_candidates, m_mr.main);
                m_copy.setup(keys_buffer)->ignore();

                const unsigned int nThreads = m_warp_size * 2;
                const unsigned int nBlocks =
                    (n_candidates + nThreads - 1) / nThreads;
                kernels::fill_sort_keys<<<nBlocks, nThreads, 0, stream>>>(
                    {in_params_buffer, keys_buffer, param_ids_buffer});
                TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

                // Sort the key and values
                vecmem::device_vector<device::sort_key> keys_device(
                    keys_buffer);
                vecmem::device_vector<unsigned int> param_ids_device(
                    param_ids_buffer);
                thrust::sort_by_key(thrust_policy, keys_device.begin(),
                                    keys_device.end(),
                                    param_ids_device.begin());

                m_stream.synchronize();
            }

            /*****************************************************************
             * Kernel5: Propagate to the next surface
             *****************************************************************/

            {
                // Reset the number of tracks per seed
                m_copy.memset(n_tracks_per_seed_buffer, 0)->ignore();

                const unsigned int nThreads = m_warp_size * 2;
                const unsigned int nBlocks =
                    (n_candidates + nThreads - 1) / nThreads;
                kernels::propagate_to_next_surface<
                    std::decay_t<propagator_type>, std::decay_t<bfield_type>>
                    <<<nBlocks, nThreads, 0, stream>>>(
                        m_cfg, {det_view, field_view, in_params_buffer,
                                param_liveness_buffer, param_ids_buffer,
                                link_map[step], step, n_candidates, tips_buffer,
                                n_tracks_per_seed_buffer});
                TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

                m_stream.synchronize();
            }
        }

        // Fill the candidate size vector
        n_candidates_per_step.push_back(n_candidates);

        n_in_params = n_candidates;
    }

    // Create link buffer
    vecmem::data::jagged_vector_buffer<candidate_link> links_buffer(
        n_candidates_per_step, m_mr.main, m_mr.host);
    m_copy.setup(links_buffer)->ignore();

    // Copy link map to link buffer
    const auto n_steps = n_candidates_per_step.size();
    for (unsigned int it = 0; it < n_steps; it++) {

        vecmem::device_vector<candidate_link> in(link_map[it]);
        vecmem::device_vector<candidate_link> out(
            *(links_buffer.host_ptr() + it));

        thrust::copy(thrust_policy, in.begin(),
                     in.begin() + n_candidates_per_step[it], out.begin());
    }

    /*****************************************************************
     * Kernel6: Build tracks
     *****************************************************************/

    // Get the number of tips
    auto n_tips_total = m_copy.get_size(tips_buffer);

    // Create track candidate buffer
    track_candidate_container_types::buffer track_candidates_buffer{
        {n_tips_total, m_mr.main},
        {std::vector<std::size_t>(n_tips_total,
                                  m_cfg.max_track_candidates_per_track),
         m_mr.main, m_mr.host, vecmem::data::buffer_type::resizable}};

    m_copy.setup(track_candidates_buffer.headers)->ignore();
    m_copy.setup(track_candidates_buffer.items)->ignore();

    // Create buffer for valid indices
    vecmem::data::vector_buffer<unsigned int> valid_indices_buffer(n_tips_total,
                                                                   m_mr.main);

    unsigned int n_valid_tracks = 0;

    // @Note: nBlocks can be zero in case there is no tip. This happens when
    // chi2_max config is set tightly and no tips are found
    if (n_tips_total > 0) {
        vecmem::unique_alloc_ptr<unsigned int> n_valid_tracks_device =
            vecmem::make_unique_alloc<unsigned int>(m_mr.main);
        TRACCC_CUDA_ERROR_CHECK(hipMemsetAsync(n_valid_tracks_device.get(), 0,
                                                sizeof(unsigned int), stream));

        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks = (n_tips_total + nThreads - 1) / nThreads;

        kernels::build_tracks<<<nBlocks, nThreads, 0, stream>>>(
            m_cfg, {measurements, seeds_buffer, links_buffer, tips_buffer,
                    track_candidates_buffer, valid_indices_buffer,
                    n_valid_tracks_device.get()});
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

        // Global counter object: Device -> Host
        TRACCC_CUDA_ERROR_CHECK(hipMemcpyAsync(
            &n_valid_tracks, n_valid_tracks_device.get(), sizeof(unsigned int),
            hipMemcpyDeviceToHost, stream));

        m_stream.synchronize();
    }

    // Create pruned candidate buffer
    track_candidate_container_types::buffer prune_candidates_buffer{
        {n_valid_tracks, m_mr.main},
        {std::vector<std::size_t>(n_valid_tracks,
                                  m_cfg.max_track_candidates_per_track),
         m_mr.main, m_mr.host, vecmem::data::buffer_type::resizable}};

    m_copy.setup(prune_candidates_buffer.headers)->ignore();
    m_copy.setup(prune_candidates_buffer.items)->ignore();

    if (n_valid_tracks > 0) {
        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks = (n_valid_tracks + nThreads - 1) / nThreads;

        kernels::prune_tracks<<<nBlocks, nThreads, 0, stream>>>(
            {track_candidates_buffer, valid_indices_buffer,
             prune_candidates_buffer});
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());
    }

    return prune_candidates_buffer;
}

// Explicit template instantiation
using default_detector_type = traccc::default_detector::device;
using default_stepper_type = detray::rk_stepper<
    covfie::field<detray::bfield::const_bknd_t<
        default_detector_type::scalar_type>>::view_t,
    default_detector_type::algebra_type,
    detray::constrained_step<default_detector_type::scalar_type>>;
using default_navigator_type = detray::navigator<const default_detector_type>;
template class finding_algorithm<default_stepper_type, default_navigator_type>;

}  // namespace traccc::cuda
