#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2023-2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "find_tracks_src.cuh"

// Project include(s).
#include "traccc/geometry/detector.hpp"

namespace traccc::cuda::kernels {
template __global__ void find_tracks<traccc::default_detector::device>(
    const finding_config cfg,
    device::find_tracks_payload<traccc::default_detector::device> payload);
}
