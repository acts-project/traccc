#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2023-2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../../utils/global_index.hpp"
#include "build_tracks.cuh"

// Project include(s).
#include "traccc/edm/measurement.hpp"
#include "traccc/edm/track_candidate.hpp"
#include "traccc/edm/track_parameters.hpp"
#include "traccc/finding/candidate_link.hpp"
#include "traccc/finding/device/build_tracks.hpp"
#include "traccc/finding/finding_config.hpp"

namespace traccc::cuda::kernels {

__global__ void build_tracks(const finding_config cfg,
                             device::build_tracks_payload payload) {

    device::build_tracks(details::global_index1(), cfg, payload);
}
}  // namespace traccc::cuda::kernels
