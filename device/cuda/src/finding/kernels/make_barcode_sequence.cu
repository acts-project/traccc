#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2023-2024 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "make_barcode_sequence.cuh"
#include "traccc/edm/measurement.hpp"
#include "traccc/finding/device/make_barcode_sequence.hpp"

namespace traccc::cuda::kernels {

__global__ void make_barcode_sequence(
    device::make_barcode_sequence_payload payload) {

    device::make_barcode_sequence(threadIdx.x + blockIdx.x * blockDim.x,
                                  payload);
}
}  // namespace traccc::cuda::kernels
