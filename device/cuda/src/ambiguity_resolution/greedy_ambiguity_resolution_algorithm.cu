#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "../utils/cuda_error_handling.hpp"
#include "../utils/utils.hpp"
#include "./kernels/add_block_offset.cuh"
#include "./kernels/block_inclusive_scan.cuh"
#include "./kernels/count_removable_tracks.cuh"
#include "./kernels/count_shared_measurements.cuh"
#include "./kernels/exclusive_scan.cuh"
#include "./kernels/fill_inverted_ids.cuh"
#include "./kernels/fill_track_candidates.cuh"
#include "./kernels/fill_tracks_per_measurement.cuh"
#include "./kernels/fill_unique_meas_id_map.cuh"
#include "./kernels/fill_vectors.cuh"
#include "./kernels/find_max_shared.cuh"
#include "./kernels/gather_tracks.cuh"
#include "./kernels/rearrange_tracks.cuh"
#include "./kernels/remove_tracks.cuh"
#include "./kernels/reset_status.cuh"
#include "./kernels/scan_block_offsets.cuh"
#include "./kernels/sort_updated_tracks.cuh"
#include "traccc/cuda/ambiguity_resolution/greedy_ambiguity_resolution_algorithm.hpp"

// Thrust include(s).
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>
namespace traccc::cuda {

// Device operator to calculate relative number of shared measurements
struct devide_op {
    TRACCC_HOST_DEVICE
    traccc::scalar operator()(unsigned int a, unsigned int b) const {
        return static_cast<traccc::scalar>(a) / static_cast<traccc::scalar>(b);
    }
};

// Track comparator to sort the track ids
struct track_comparator {
    const traccc::scalar* rel_shared;
    const traccc::scalar* pvals;

    TRACCC_HOST_DEVICE track_comparator(const traccc::scalar* rel_shared_,
                                        const traccc::scalar* pvals_)
        : rel_shared(rel_shared_), pvals(pvals_) {}

    TRACCC_HOST_DEVICE bool operator()(unsigned int a, unsigned int b) const {
        if (rel_shared[a] != rel_shared[b]) {
            return rel_shared[a] < rel_shared[b];
        }
        return pvals[a] > pvals[b];
    }
};

struct measurement_id_comparator {
    TRACCC_HOST_DEVICE bool operator()(const measurement& a,
                                       const measurement& b) const {
        return a.measurement_id < b.measurement_id;
    }
};

greedy_ambiguity_resolution_algorithm::greedy_ambiguity_resolution_algorithm(
    const config_type& cfg, const traccc::memory_resource& mr,
    vecmem::copy& copy, stream& str, std::unique_ptr<const Logger> logger)
    : messaging(std::move(logger)),
      m_config(cfg),
      m_mr(mr),
      m_copy(copy),
      m_stream(str),
      m_warp_size(details::get_warp_size(str.device())) {}

greedy_ambiguity_resolution_algorithm::output_type
greedy_ambiguity_resolution_algorithm::operator()(
    const edm::track_candidate_container<default_algebra>::const_view&
        track_candidates_view) const {

    measurement_collection_types::const_device measurements(
        track_candidates_view.measurements);

    auto n_meas_total =
        m_copy.get().get_size(track_candidates_view.measurements);

    // Make sure that max_measurement_id = number_of_measurement -1
    // @TODO: More robust way is to assert that measurement id ranges from 0, 1,
    // ..., number_of_measurement - 1
    [[maybe_unused]] auto max_meas_it = thrust::max_element(
        thrust::device, track_candidates_view.measurements.ptr(),
        track_candidates_view.measurements.ptr() + n_meas_total,
        measurement_id_comparator{});

    measurement max_meas;
    hipMemcpy(&max_meas, thrust::raw_pointer_cast(&(*max_meas_it)),
               sizeof(measurement), hipMemcpyDeviceToHost);

    if (max_meas.measurement_id != n_meas_total - 1) {
        throw std::runtime_error(
            "max measurement id should be equal to (the number of measurements "
            "- 1)");
    }

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // The Thrust policy to use.
    auto thrust_policy =
        thrust::cuda::par_nosync(std::pmr::polymorphic_allocator(&(m_mr.main)))
            .on(stream);

    const unsigned int n_tracks = track_candidates_view.tracks.capacity();

    if (n_tracks == 0) {
        return {};
    }

    // Make sure that max_shared_meas is largen than zero
    assert(m_config.max_shared_meas > 0u);

    // Status (1 = Accept, 0 = Reject)
    vecmem::data::vector_buffer<int> status_buffer{n_tracks, m_mr.main};

    vecmem::device_vector<int> status_device(status_buffer);
    thrust::fill(thrust_policy, status_device.begin(), status_device.end(), 1);

    // Get the sizes of the measurement index vector in each track
    const std::vector<unsigned int> candidate_sizes =
        m_copy.get().get_sizes(track_candidates_view.tracks);

    // Make measurement ID, pval and n_measurement vector
    vecmem::data::jagged_vector_buffer<measurement_id_type> meas_ids_buffer{
        candidate_sizes, m_mr.main, m_mr.host,
        vecmem::data::buffer_type::resizable};
    m_copy.get().setup(meas_ids_buffer)->ignore();

    const unsigned int n_cands_total =
        std::accumulate(candidate_sizes.begin(), candidate_sizes.end(), 0u);

    vecmem::data::vector_buffer<measurement_id_type> flat_meas_ids_buffer{
        n_cands_total, m_mr.main, vecmem::data::buffer_type::resizable};
    m_copy.get().setup(flat_meas_ids_buffer)->ignore();
    vecmem::data::vector_buffer<traccc::scalar> pvals_buffer{n_tracks,
                                                             m_mr.main};
    vecmem::data::vector_buffer<unsigned int> n_meas_buffer{n_tracks,
                                                            m_mr.main};
    thrust::fill(thrust_policy, n_meas_buffer.ptr(),
                 n_meas_buffer.ptr() + n_tracks, 0);

    {
        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks = (n_tracks + nThreads - 1) / nThreads;

        // Fill the vectors
        kernels::fill_vectors<<<nBlocks, nThreads, 0, stream>>>(
            m_config, device::fill_vectors_payload{
                          .track_candidates_view = track_candidates_view,
                          .meas_ids_view = meas_ids_buffer,
                          .flat_meas_ids_view = flat_meas_ids_buffer,
                          .pvals_view = pvals_buffer,
                          .n_meas_view = n_meas_buffer,
                          .status_view = status_buffer});
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

        m_stream.get().synchronize();
    }

    unsigned int n_accepted = static_cast<unsigned int>(thrust::count(
        thrust_policy, status_buffer.ptr(), status_buffer.ptr() + n_tracks, 1));

    vecmem::unique_alloc_ptr<unsigned int> n_accepted_device =
        vecmem::make_unique_alloc<unsigned int>(m_mr.main);
    TRACCC_CUDA_ERROR_CHECK(hipMemcpyAsync(n_accepted_device.get(),
                                            &n_accepted, sizeof(unsigned int),
                                            hipMemcpyHostToDevice, stream));

    m_stream.get().synchronize();

    if (n_accepted == 0) {
        return {};
    }

    // Make accepted ids vector
    vecmem::data::vector_buffer<unsigned int> pre_accepted_ids_buffer{
        n_accepted, m_mr.main};

    m_copy.get().setup(pre_accepted_ids_buffer)->ignore();

    // Fill the accepted ids vector using counting iterator
    auto cit_begin = thrust::counting_iterator<int>(0);
    auto cit_end = cit_begin + n_tracks;
    thrust::copy_if(thrust_policy, cit_begin, cit_end, status_buffer.ptr(),
                    pre_accepted_ids_buffer.ptr(), thrust::identity<int>());

    // Sort the flat measurement id vector
    thrust::sort(thrust_policy, flat_meas_ids_buffer.ptr(),
                 flat_meas_ids_buffer.ptr() + n_cands_total);

    // Count the number of unique measurements
    const unsigned int meas_count = static_cast<unsigned int>(
        thrust::unique_count(thrust_policy, flat_meas_ids_buffer.ptr(),
                             flat_meas_ids_buffer.ptr() + n_cands_total,
                             thrust::equal_to<int>()));

    // Unique measurement ids
    vecmem::data::vector_buffer<measurement_id_type> unique_meas_buffer{
        meas_count, m_mr.main};

    // Counts of unique measurement id in flat id vector
    vecmem::data::vector_buffer<std::size_t> unique_meas_counts_buffer{
        meas_count, m_mr.main};
    m_copy.get().setup(unique_meas_counts_buffer)->ignore();

    // Counting can be done using reduce_by_key and constant iterator
    thrust::reduce_by_key(thrust_policy, flat_meas_ids_buffer.ptr(),
                          flat_meas_ids_buffer.ptr() + n_cands_total,
                          thrust::make_constant_iterator(1),
                          unique_meas_buffer.ptr(),
                          unique_meas_counts_buffer.ptr());

    // Sort unique meas ids
    thrust::sort_by_key(thrust_policy, unique_meas_buffer.ptr(),
                        unique_meas_buffer.ptr() + meas_count,
                        unique_meas_counts_buffer.ptr());

    // Unique measurement ids
    vecmem::data::vector_buffer<measurement_id_type>
        meas_id_to_unique_id_buffer{max_meas.measurement_id, m_mr.main};

    // Make meas_id to meas vector
    {
        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks = (meas_count + nThreads - 1) / nThreads;

        kernels::fill_unique_meas_id_map<<<nBlocks, nThreads, 0, stream>>>(
            device::fill_unique_meas_id_map_payload{
                .unique_meas_view = unique_meas_buffer,
                .meas_id_to_unique_id_view = meas_id_to_unique_id_buffer});
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

        m_stream.get().synchronize();
    }

    // Retreive the counting vector to host
    std::vector<std::size_t> unique_meas_counts;
    m_copy
        .get()(unique_meas_counts_buffer, unique_meas_counts,
               vecmem::copy::type::device_to_host)
        ->wait();

    // Make the tracks per measurement vector
    vecmem::data::jagged_vector_buffer<unsigned int>
        tracks_per_measurement_buffer(unique_meas_counts, m_mr.main, m_mr.host,
                                      vecmem::data::buffer_type::resizable);
    m_copy.get().setup(tracks_per_measurement_buffer)->ignore();

    // Make the track status per measurement vector
    vecmem::data::jagged_vector_buffer<int> track_status_per_measurement_buffer(
        unique_meas_counts, m_mr.main, m_mr.host,
        vecmem::data::buffer_type::resizable);

    m_copy.get().setup(track_status_per_measurement_buffer)->ignore();

    // Make the number of accetped tracks per measurement vector
    vecmem::data::vector_buffer<unsigned int>
        n_accepted_tracks_per_measurement_buffer(meas_count, m_mr.main);
    thrust::fill(thrust_policy, n_accepted_tracks_per_measurement_buffer.ptr(),
                 n_accepted_tracks_per_measurement_buffer.ptr() + meas_count,
                 0);

    // Fill tracks per measurement vector
    {
        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks = (n_accepted + nThreads - 1) / nThreads;

        kernels::fill_tracks_per_measurement<<<nBlocks, nThreads, 0, stream>>>(
            device::fill_tracks_per_measurement_payload{
                .accepted_ids_view = pre_accepted_ids_buffer,
                .meas_ids_view = meas_ids_buffer,
                .meas_id_to_unique_id_view = meas_id_to_unique_id_buffer,
                .tracks_per_measurement_view = tracks_per_measurement_buffer,
                .track_status_per_measurement_view =
                    track_status_per_measurement_buffer,
                .n_accepted_tracks_per_measurement_view =
                    n_accepted_tracks_per_measurement_buffer});
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

        m_stream.get().synchronize();
    }

    // Make shared number of measurements vector
    vecmem::data::vector_buffer<unsigned int> n_shared_buffer{n_tracks,
                                                              m_mr.main};
    thrust::fill(thrust_policy, n_shared_buffer.ptr(),
                 n_shared_buffer.ptr() + n_tracks, 0);
    m_copy.get().setup(n_shared_buffer)->ignore();

    // Count shared number of measurements
    {
        const unsigned int nThreads = m_warp_size * 2;
        const unsigned int nBlocks = (n_accepted + nThreads - 1) / nThreads;

        kernels::count_shared_measurements<<<nBlocks, nThreads, 0, stream>>>(
            device::count_shared_measurements_payload{
                .accepted_ids_view = pre_accepted_ids_buffer,
                .meas_ids_view = meas_ids_buffer,
                .meas_id_to_unique_id_view = meas_id_to_unique_id_buffer,
                .n_accepted_tracks_per_measurement_view =
                    n_accepted_tracks_per_measurement_buffer,
                .n_shared_view = n_shared_buffer});
        TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

        m_stream.get().synchronize();
    }

    // Make relative shared number of measurements vector
    vecmem::data::vector_buffer<traccc::scalar> rel_shared_buffer{n_tracks,
                                                                  m_mr.main};

    // Fill the relative shared number of measurements vector
    thrust::transform(thrust_policy, n_shared_buffer.ptr(),
                      n_shared_buffer.ptr() + n_tracks, n_meas_buffer.ptr(),
                      rel_shared_buffer.ptr(), devide_op{});

    // Make sorted ids vector
    vecmem::data::vector_buffer<unsigned int> sorted_ids_buffer{n_accepted,
                                                                m_mr.main};
    m_copy.get().setup(sorted_ids_buffer)->ignore();
    vecmem::data::vector_buffer<unsigned int> temp_sorted_ids_buffer{n_accepted,
                                                                     m_mr.main};
    m_copy.get().setup(temp_sorted_ids_buffer)->ignore();

    // track id to the index of sorted ids
    vecmem::data::vector_buffer<unsigned int> inverted_ids_buffer{n_tracks,
                                                                  m_mr.main};
    m_copy.get().setup(inverted_ids_buffer)->ignore();

    // Whether track id is updated after an iteration
    vecmem::data::vector_buffer<int> is_updated_buffer{n_tracks, m_mr.main};
    m_copy.get().setup(inverted_ids_buffer)->ignore();

    // Prefix sum buffer
    vecmem::data::vector_buffer<int> prefix_sums_buffer{n_tracks, m_mr.main};
    m_copy.get().setup(prefix_sums_buffer)->ignore();

    // Fill and sort the sorted ids vector
    thrust::copy(thrust_policy, pre_accepted_ids_buffer.ptr(),
                 pre_accepted_ids_buffer.ptr() + n_accepted,
                 sorted_ids_buffer.ptr());
    m_stream.get().synchronize();

    track_comparator trk_comp(rel_shared_buffer.ptr(), pvals_buffer.ptr());
    thrust::sort(thrust_policy, sorted_ids_buffer.ptr(),
                 sorted_ids_buffer.ptr() + n_accepted, trk_comp);

    // Update track ids
    vecmem::data::vector_buffer<unsigned int> updated_tracks_buffer{n_accepted,
                                                                    m_mr.main};
    m_copy.get().setup(updated_tracks_buffer)->ignore();

    // Measurements to remove for each iteration
    vecmem::data::vector_buffer<measurement_id_type> meas_to_remove_buffer{
        1024, m_mr.main};
    vecmem::data::vector_buffer<unsigned int> threads_buffer{1024, m_mr.main};

    vecmem::unique_alloc_ptr<unsigned int> n_removable_tracks_device =
        vecmem::make_unique_alloc<unsigned int>(m_mr.main);
    vecmem::unique_alloc_ptr<unsigned int> n_meas_to_remove_device =
        vecmem::make_unique_alloc<unsigned int>(m_mr.main);

    // Device objects
    int is_first_iteration = 1;
    vecmem::unique_alloc_ptr<int> is_first_iteration_device =
        vecmem::make_unique_alloc<int>(m_mr.main);
    hipMemcpyAsync(is_first_iteration_device.get(), &is_first_iteration,
                    sizeof(int), hipMemcpyHostToDevice, stream);
    int terminate = 0;
    vecmem::unique_alloc_ptr<int> terminate_device =
        vecmem::make_unique_alloc<int>(m_mr.main);
    vecmem::unique_alloc_ptr<unsigned int> max_shared_device =
        vecmem::make_unique_alloc<unsigned int>(m_mr.main);
    vecmem::unique_alloc_ptr<unsigned int> n_updated_tracks_device =
        vecmem::make_unique_alloc<unsigned int>(m_mr.main);

    // Thread block size
    unsigned int nThreads_adaptive = 1024;
    unsigned int nBlocks_adaptive = (n_accepted + 1023) / 1024;

    unsigned int nThreads_warp = m_warp_size;
    unsigned int nBlocks_warp =
        (n_accepted + nThreads_warp - 1) / nThreads_warp;

    unsigned int nThreads_full = 1024;
    unsigned int nBlocks_full = (n_tracks + 1023) / 1024;

    unsigned int nThreads_scan = 1024;
    unsigned int nBlocks_scan = (n_accepted + 1023) / 1024;

    assert(nBlocks_scan <= 1024 &&
           "nBlocks_scan larger than 1024 will cause invalid arguments in "
           "scan_block_offsets kernel");

    // block offsets buffer
    vecmem::data::vector_buffer<int> block_offsets_buffer{nBlocks_scan,
                                                          m_mr.main};
    m_copy.get().setup(block_offsets_buffer)->ignore();
    vecmem::data::vector_buffer<int> scanned_block_offsets_buffer{nBlocks_scan,
                                                                  m_mr.main};
    m_copy.get().setup(block_offsets_buffer)->ignore();

    while (!terminate && n_accepted > 0) {
        nBlocks_adaptive = (n_accepted + 1023) / 1024;
        nBlocks_warp = (n_accepted + nThreads_warp - 1) / nThreads_warp;
        nBlocks_scan = (n_accepted + 1023) / 1024;

        // Make CUDA Graph
        hipGraph_t graph;
        hipGraphExec_t graphExec;

        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

        kernels::reset_status<<<1, 1, 0, stream>>>(device::reset_status_payload{
            .is_first_iteration = is_first_iteration_device.get(),
            .terminate = terminate_device.get(),
            .n_accepted = n_accepted_device.get(),
            .max_shared = max_shared_device.get(),
            .n_updated_tracks = n_updated_tracks_device.get()});

        kernels::find_max_shared<<<nBlocks_warp, nThreads_warp, 0, stream>>>(
            device::find_max_shared_payload{
                .sorted_ids_view = sorted_ids_buffer,
                .n_accepted = n_accepted_device.get(),
                .n_shared_view = n_shared_buffer,
                .terminate = terminate_device.get(),
                .max_shared = max_shared_device.get(),
                .is_updated_view = is_updated_buffer});

        kernels::count_removable_tracks<<<
            1, 1024,
            sizeof(int) * 1024 + sizeof(measurement_id_type) * 1024 +
                sizeof(unsigned int) * 1024 + sizeof(unsigned int) * 6 +
                sizeof(bool),
            stream>>>(device::count_removable_tracks_payload{
            .terminate = terminate_device.get(),
            .max_shared = max_shared_device.get(),
            .sorted_ids_view = sorted_ids_buffer,
            .n_accepted = n_accepted_device.get(),
            .meas_ids_view = meas_ids_buffer,
            .n_meas_view = n_meas_buffer,
            .meas_id_to_unique_id_view = meas_id_to_unique_id_buffer,
            .n_accepted_tracks_per_measurement_view =
                n_accepted_tracks_per_measurement_buffer,
            .n_removable_tracks = n_removable_tracks_device.get(),
            .n_meas_to_remove = n_meas_to_remove_device.get(),
            .meas_to_remove_view = meas_to_remove_buffer,
            .threads_view = threads_buffer});

        kernels::exclusive_scan<<<1, 1024,
                                  sizeof(int) * 1024 +
                                      sizeof(measurement_id_type) * 1024 +
                                      sizeof(unsigned int) * 1024,
                                  stream>>>(device::exclusive_scan_payload{
            .terminate = terminate_device.get(),
            .n_removable_tracks = n_removable_tracks_device.get(),
            .n_meas_to_remove = n_meas_to_remove_device.get(),
            .meas_to_remove_view = meas_to_remove_buffer,
            .threads_view = threads_buffer});

        kernels::remove_tracks<<<1, 1024,
                                 1024 * (2 * sizeof(unsigned int) +
                                         sizeof(measurement_id_type)),
                                 stream>>>(device::remove_tracks_payload{
            .sorted_ids_view = sorted_ids_buffer,
            .n_accepted = n_accepted_device.get(),
            .meas_ids_view = meas_ids_buffer,
            .n_meas_view = n_meas_buffer,
            .meas_id_to_unique_id_view = meas_id_to_unique_id_buffer,
            .tracks_per_measurement_view = tracks_per_measurement_buffer,
            .track_status_per_measurement_view =
                track_status_per_measurement_buffer,
            .n_accepted_tracks_per_measurement_view =
                n_accepted_tracks_per_measurement_buffer,
            .n_shared_view = n_shared_buffer,
            .rel_shared_view = rel_shared_buffer,
            .n_removable_tracks = n_removable_tracks_device.get(),
            .n_meas_to_remove = n_meas_to_remove_device.get(),
            .meas_to_remove_view = meas_to_remove_buffer,
            .threads_view = threads_buffer,
            .terminate = terminate_device.get(),
            .n_updated_tracks = n_updated_tracks_device.get(),
            .updated_tracks_view = updated_tracks_buffer,
            .is_updated_view = is_updated_buffer});

        // The seven kernels below are to keep sorted_ids sorted based on
        // the relative shared measurements and pvalues. This can be reduced
        // into thrust::sort():
        /*
        hipMemcpyAsync(&n_accepted, n_accepted_device.get(),
                        sizeof(unsigned int), hipMemcpyDeviceToHost,
        stream); thrust::sort(thrust_policy, sorted_ids_buffer.ptr(),
                     sorted_ids_buffer.ptr() + n_accepted,
                     trk_comp);
        */
        // Disadvantage: we need to do device-host copy which has large
        // overhead and CUDA graph is not available anymore
        // Advantage: This works for all cases (The below kernels only work
        // when the number of updated tracks <= 1024) and might be faster
        // with large number of updated tracks

        kernels::sort_updated_tracks<<<1, 1024, 1024 * sizeof(unsigned int),
                                       stream>>>(
            device::sort_updated_tracks_payload{
                .rel_shared_view = rel_shared_buffer,
                .pvals_view = pvals_buffer,
                .terminate = terminate_device.get(),
                .n_updated_tracks = n_updated_tracks_device.get(),
                .updated_tracks_view = updated_tracks_buffer,
            });

        kernels::fill_inverted_ids<<<nBlocks_adaptive, nThreads_adaptive, 0,
                                     stream>>>(
            device::fill_inverted_ids_payload{
                .sorted_ids_view = sorted_ids_buffer,
                .terminate = terminate_device.get(),
                .n_accepted = n_accepted_device.get(),
                .n_updated_tracks = n_updated_tracks_device.get(),
                .inverted_ids_view = inverted_ids_buffer,
            });

        kernels::block_inclusive_scan<<<nBlocks_scan, nThreads_scan,
                                        nThreads_scan * sizeof(int), stream>>>(
            device::block_inclusive_scan_payload{
                .sorted_ids_view = sorted_ids_buffer,
                .terminate = terminate_device.get(),
                .n_accepted = n_accepted_device.get(),
                .n_updated_tracks = n_updated_tracks_device.get(),
                .is_updated_view = is_updated_buffer,
                .block_offsets_view = block_offsets_buffer,
                .prefix_sums_view = prefix_sums_buffer});

        kernels::scan_block_offsets<<<1, nBlocks_scan,
                                      nBlocks_scan * sizeof(int), stream>>>(
            device::scan_block_offsets_payload{
                .terminate = terminate_device.get(),
                .n_accepted = n_accepted_device.get(),
                .n_updated_tracks = n_updated_tracks_device.get(),
                .block_offsets_view = block_offsets_buffer,
                .scanned_block_offsets_view = scanned_block_offsets_buffer});

        kernels::add_block_offset<<<nBlocks_scan, nThreads_scan, 0, stream>>>(
            device::add_block_offset_payload{
                .terminate = terminate_device.get(),
                .n_accepted = n_accepted_device.get(),
                .n_updated_tracks = n_updated_tracks_device.get(),
                .block_offsets_view = scanned_block_offsets_buffer,
                .prefix_sums_view = prefix_sums_buffer});

        kernels::rearrange_tracks<<<nBlocks_adaptive, nThreads_adaptive, 0,
                                    stream>>>(device::rearrange_tracks_payload{
            .sorted_ids_view = sorted_ids_buffer,
            .inverted_ids_view = inverted_ids_buffer,
            .rel_shared_view = rel_shared_buffer,
            .pvals_view = pvals_buffer,
            .terminate = terminate_device.get(),
            .n_accepted = n_accepted_device.get(),
            .n_updated_tracks = n_updated_tracks_device.get(),
            .updated_tracks_view = updated_tracks_buffer,
            .is_updated_view = is_updated_buffer,
            .prefix_sums_view = prefix_sums_buffer,
            .temp_sorted_ids_view = temp_sorted_ids_buffer,
        });

        kernels::gather_tracks<<<nBlocks_full, nThreads_full, 0, stream>>>(
            device::gather_tracks_payload{
                .terminate = terminate_device.get(),
                .n_accepted = n_accepted_device.get(),
                .n_updated_tracks = n_updated_tracks_device.get(),
                .temp_sorted_ids_view = temp_sorted_ids_buffer,
                .sorted_ids_view = sorted_ids_buffer,
                .is_updated_view = is_updated_buffer});

        hipStreamEndCapture(stream, &graph);
        hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);

        // TODO: Make n_it adaptive based on the average track length, bound
        // value in count_removable_tracks, etc.
        const unsigned int n_it = 100;
        for (unsigned int iter = 0; iter < n_it; iter++) {
            hipGraphLaunch(graphExec, stream);
        }

        hipMemcpyAsync(&terminate, terminate_device.get(), sizeof(int),
                        hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(&n_accepted, n_accepted_device.get(),
                        sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
    }

    hipMemcpyAsync(&n_accepted, n_accepted_device.get(), sizeof(unsigned int),
                    hipMemcpyDeviceToHost, stream);

    auto max_it =
        std::max_element(candidate_sizes.begin(), candidate_sizes.end());
    const unsigned int max_cands_size = *max_it;

    // Create resolved candidate buffer
    edm::track_candidate_collection<default_algebra>::buffer
        res_track_candidates_buffer{
            std::vector<std::size_t>(n_accepted, max_cands_size), m_mr.main,
            m_mr.host, vecmem::data::buffer_type::resizable};
    m_copy.get().setup(res_track_candidates_buffer)->ignore();

    // Fill the output track candidates
    {
        if (n_accepted > 0) {
            kernels::fill_track_candidates<<<
                static_cast<unsigned int>((n_accepted + 63) / 64), 64, 0,
                stream>>>(device::fill_track_candidates_payload{
                .track_candidates_view = track_candidates_view.tracks,
                .n_accepted = n_accepted,
                .sorted_ids_view = sorted_ids_buffer,
                .res_track_candidates_view = res_track_candidates_buffer});
            TRACCC_CUDA_ERROR_CHECK(hipGetLastError());

            m_stream.get().synchronize();
        }
    }

    return res_track_candidates_buffer;
}

}  // namespace traccc::cuda
