#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../../utils/global_index.hpp"
#include "fill_tracks_per_measurement.cuh"

// VecMem include(s).
#include <vecmem/containers/device_vector.hpp>
#include <vecmem/containers/jagged_device_vector.hpp>

// Thrust include(s).
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/find.h>

namespace traccc::cuda::kernels {

__global__ void fill_tracks_per_measurement(
    device::fill_tracks_per_measurement_payload payload) {

    vecmem::device_vector<const unsigned int> accepted_ids(
        payload.accepted_ids_view);

    const auto globalIndex = details::global_index1();
    if (globalIndex >= accepted_ids.size()) {
        return;
    }

    vecmem::jagged_device_vector<const measurement_id_type> meas_ids(
        payload.meas_ids_view);
    vecmem::device_vector<const unsigned int> meas_id_to_unique_id(
        payload.meas_id_to_unique_id_view);
    vecmem::jagged_device_vector<unsigned int> tracks_per_measurement(
        payload.tracks_per_measurement_view);
    vecmem::jagged_device_vector<int> track_status_per_measurement(
        payload.track_status_per_measurement_view);
    vecmem::device_vector<unsigned int> n_accepted_tracks_per_measurement(
        payload.n_accepted_tracks_per_measurement_view);

    const unsigned int id = accepted_ids.at(globalIndex);

    for (unsigned int i = 0; i < meas_ids[id].size(); i++) {
        auto meas_id = meas_ids[id][i];

        if (thrust::find(thrust::seq, meas_ids[id].begin(),
                         meas_ids[id].begin() + i,
                         meas_id) != (meas_ids[id].begin() + i)) {
            continue;
        }

        const auto unique_meas_idx = meas_id_to_unique_id.at(meas_id);

        auto tracks = tracks_per_measurement.at(unique_meas_idx);

        tracks_per_measurement.at(unique_meas_idx).push_back(id);
        track_status_per_measurement.at(unique_meas_idx).push_back(1);

        vecmem::device_atomic_ref<unsigned int> n_accepted(
            n_accepted_tracks_per_measurement.at(
                static_cast<unsigned int>(unique_meas_idx)));
        n_accepted.fetch_add(1u);
    }
}
}  // namespace traccc::cuda::kernels
