#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../../utils/global_index.hpp"
#include "fill_tracks_per_measurement.cuh"

namespace traccc::cuda::kernels {

__global__ void fill_tracks_per_measurement(
    device::fill_tracks_per_measurement_payload payload) {

    device::fill_tracks_per_measurement(details::global_index1(), payload);
}
}  // namespace traccc::cuda::kernels
