#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../../utils/global_index.hpp"
#include "fill_vectors.cuh"

// Project include(s).
#include "traccc/ambiguity_resolution/ambiguity_resolution_config.hpp"

namespace traccc::cuda::kernels {

__global__ void fill_vectors(const ambiguity_resolution_config cfg,
                             device::fill_vectors_payload payload) {

    device::fill_vectors(details::global_index1(), cfg, payload);
}
}  // namespace traccc::cuda::kernels
