#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../../utils/global_index.hpp"
#include "fill_track_candidates.cuh"

// VecMem include(s).
#include <vecmem/containers/device_vector.hpp>

namespace traccc::cuda::kernels {

__global__ void fill_track_candidates(
    device::fill_track_candidates_payload payload) {

    const auto globalIndex = details::global_index1();
    if (globalIndex >= payload.n_accepted) {
        return;
    }

    // Set up the device objects.
    vecmem::device_vector<const unsigned int> sorted_ids(
        payload.sorted_ids_view);
    edm::track_candidate_collection<default_algebra>::const_device
        track_candidates(payload.track_candidates_view);
    edm::track_candidate_collection<default_algebra>::device
        res_track_candidates(payload.res_track_candidates_view);

    // Copy the appropriate track candidate.
    res_track_candidates.at(globalIndex) =
        track_candidates.at(sorted_ids.at(globalIndex));
}

}  // namespace traccc::cuda::kernels
