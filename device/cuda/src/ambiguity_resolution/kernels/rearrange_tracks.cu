#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include "traccc/definitions/qualifiers.hpp"

// Local include(s).
#include "../../utils/global_index.hpp"
#include "rearrange_tracks.cuh"

// VecMem include(s).
#include <vecmem/containers/device_vector.hpp>

namespace traccc::cuda::kernels {

TRACCC_DEVICE inline bool find_valid_index(
    unsigned int& idx, const int lower_bound, const int upper_bound,
    const vecmem::device_vector<const unsigned int>& sorted_ids,
    const vecmem::device_vector<const int>& is_updated) {

    for (int i = idx; i <= upper_bound; i++) {
        if (!is_updated[sorted_ids[i]]) {
            idx = i;
            return true;
        }
    }

    for (int i = idx; i >= lower_bound; i--) {
        if (!is_updated[sorted_ids[i]]) {
            idx = i;
            return true;
        }
    }

    return false;
}

__global__ void rearrange_tracks(device::rearrange_tracks_payload payload) {

    if (*(payload.terminate) == 1 || *(payload.n_updated_tracks) == 0) {
        return;
    }

    auto gid = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int n_accepted = *(payload.n_accepted);

    if (gid >= n_accepted) {
        return;
    }

    vecmem::device_vector<const unsigned int> sorted_ids(
        payload.sorted_ids_view);
    vecmem::device_vector<const unsigned int> inverted_ids(
        payload.inverted_ids_view);
    vecmem::device_vector<const traccc::scalar> rel_shared(
        payload.rel_shared_view);
    vecmem::device_vector<const traccc::scalar> pvals(payload.pvals_view);
    vecmem::device_vector<const unsigned int> updated_tracks(
        payload.updated_tracks_view);
    vecmem::device_vector<const int> is_updated(payload.is_updated_view);
    vecmem::device_vector<const int> prefix_sums(payload.prefix_sums_view);
    vecmem::device_vector<unsigned int> temp_sorted_ids(
        payload.temp_sorted_ids_view);

    const auto tid = sorted_ids[gid];
    auto rel_sh_ref = rel_shared[tid];
    auto pval_ref = pvals[tid];
    int shifted_idx = static_cast<int>(gid);
    auto N = *(payload.n_updated_tracks);

    if (is_updated[tid]) {

        if (gid > 0) {

            unsigned int left = 0;
            unsigned int right = gid;

            bool first_iteration = true;
            while (right > left) {

                const bool find_left =
                    find_valid_index(left, 0, gid, sorted_ids, is_updated);

                if (!find_left) {
                    break;
                }

                const bool find_right =
                    find_valid_index(right, 0, gid, sorted_ids, is_updated);

                if (!find_right) {
                    break;
                }

                if (first_iteration) {
                    auto rel_sh = rel_shared[sorted_ids[right]];
                    auto pval = pvals[sorted_ids[right]];

                    if (rel_sh < rel_sh_ref ||
                        (rel_sh == rel_sh_ref && pval >= pval_ref)) {
                        left = gid;
                        break;
                    }
                }

                first_iteration = false;

                unsigned int mid = left + (right - left) / 2;

                const bool find_mid = find_valid_index(mid, left, right - 1,
                                                       sorted_ids, is_updated);

                if (find_mid) {

                    auto rel_sh = rel_shared[sorted_ids[mid]];
                    auto pval = pvals[sorted_ids[mid]];

                    if (rel_sh < rel_sh_ref ||
                        (rel_sh == rel_sh_ref && pval >= pval_ref)) {

                        left = mid + 1;
                    } else {
                        right = mid;
                    }
                }
            }

            int delta = 0;

            if (is_updated[sorted_ids[left]]) {
                delta = gid - left - (prefix_sums[gid] - prefix_sums[left]);
            } else {
                delta = gid - left - (prefix_sums[gid] - prefix_sums[left] - 1);
            }

            shifted_idx -= delta;
        }

        for (int i = 0; i < N; i++) {

            auto id = updated_tracks[i];
            auto rel_sh = rel_shared[id];
            auto pval = pvals[id];

            if (inverted_ids[id] < gid) {
                shifted_idx--;
            }
        }

        int offset = 0;
        for (int i = 0; i < N; i++) {
            if (updated_tracks[i] == tid) {
                offset = i;
                break;
            }
        }
        shifted_idx += offset;

    } else {
        for (int i = 0; i < N; i++) {

            auto id = updated_tracks[i];
            auto rel_sh = rel_shared[id];
            auto pval = pvals[id];

            if (inverted_ids[id] > gid) {
                if (rel_sh < rel_sh_ref) {
                    shifted_idx++;
                } else if (rel_sh == rel_sh_ref && pval > pval_ref) {
                    shifted_idx++;
                }
            }
        }
    }

    temp_sorted_ids.at(shifted_idx) = tid;
}

}  // namespace traccc::cuda::kernels
