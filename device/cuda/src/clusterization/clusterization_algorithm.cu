#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// CUDA Library include(s).
#include "../utils/utils.hpp"
#include "traccc/cuda/clusterization/clusterization_algorithm.hpp"
#include "traccc/cuda/utils/barrier.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// Project include(s)
#include "traccc/clusterization/device/aggregate_cluster.hpp"
#include "traccc/clusterization/device/ccl_kernel.hpp"
#include "traccc/clusterization/device/form_spacepoints.hpp"
#include "traccc/clusterization/device/reduce_problem_cell.hpp"

// Vecmem include(s).
#include <vecmem/utils/copy.hpp>

// System include(s).
#include <algorithm>

namespace traccc::cuda {

namespace {
/// These indices in clusterization will only range from 0 to
/// max_cells_per_partition, so we only need a short.
using index_t = unsigned short;

static constexpr int TARGET_CELLS_PER_THREAD = 8;
static constexpr int MAX_CELLS_PER_THREAD = 12;
}  // namespace

namespace kernels {

/// CUDA kernel for running @c traccc::device::ccl_kernel
__global__ void ccl_kernel(
    const cell_collection_types::const_view cells_view,
    const cell_module_collection_types::const_view modules_view,
    const index_t max_cells_per_partition,
    const index_t target_cells_per_partition,
    alt_measurement_collection_types::view measurements_view,
    unsigned int& measurement_count,
    vecmem::data::vector_view<unsigned int> cell_links) {
    __shared__ unsigned int partition_start, partition_end;
    __shared__ unsigned int outi;
    extern __shared__ index_t shared_v[];
    index_t* f = &shared_v[0];
    index_t* f_next = &shared_v[max_cells_per_partition];
    traccc::cuda::barrier barry_r;

    device::ccl_kernel(threadIdx.x, blockDim.x, blockIdx.x, cells_view,
                       modules_view, max_cells_per_partition,
                       target_cells_per_partition, partition_start,
                       partition_end, outi, f, f_next, barry_r,
                       measurements_view, measurement_count, cell_links);
}

__global__ void form_spacepoints(
    alt_measurement_collection_types::const_view measurements_view,
    cell_module_collection_types::const_view modules_view,
    const unsigned int measurement_count,
    spacepoint_collection_types::view spacepoints_view) {

    device::form_spacepoints(threadIdx.x + blockIdx.x * blockDim.x,
                             measurements_view, modules_view, measurement_count,
                             spacepoints_view);
}

}  // namespace kernels

clusterization_algorithm::clusterization_algorithm(
    const traccc::memory_resource& mr, vecmem::copy& copy, stream& str,
    const unsigned short target_cells_per_partition)
    : m_mr(mr),
      m_copy(copy),
      m_stream(str),
      m_target_cells_per_partition(target_cells_per_partition) {}

clusterization_algorithm::output_type clusterization_algorithm::operator()(
    const cell_collection_types::const_view& cells,
    const cell_module_collection_types::const_view& modules) const {

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // Number of cells
    const cell_collection_types::view::size_type num_cells =
        m_copy.get_size(cells);

    if (num_cells == 0) {
        return {output_type::first_type{0, m_mr.main},
                output_type::second_type{0, m_mr.main}};
    }

    // Create result object for the CCL kernel with size overestimation
    alt_measurement_collection_types::buffer measurements_buffer(num_cells,
                                                                 m_mr.main);
    m_copy.setup(measurements_buffer);

    // Counter for number of measurements
    vecmem::unique_alloc_ptr<unsigned int> num_measurements_device =
        vecmem::make_unique_alloc<unsigned int>(m_mr.main);
    CUDA_ERROR_CHECK(hipMemsetAsync(num_measurements_device.get(), 0,
                                     sizeof(unsigned int), stream));

    const unsigned short max_cells_per_partition =
        (m_target_cells_per_partition * MAX_CELLS_PER_THREAD +
         TARGET_CELLS_PER_THREAD - 1) /
        TARGET_CELLS_PER_THREAD;
    const unsigned int threads_per_partition =
        (m_target_cells_per_partition + TARGET_CELLS_PER_THREAD - 1) /
        TARGET_CELLS_PER_THREAD;
    const unsigned int num_partitions =
        (num_cells + m_target_cells_per_partition - 1) /
        m_target_cells_per_partition;

    // Create buffer for linking cells to their spacepoints.
    vecmem::data::vector_buffer<unsigned int> cell_links(num_cells, m_mr.main);
    m_copy.setup(cell_links);

    // Launch ccl kernel. Each thread will handle a single cell.
    kernels::
        ccl_kernel<<<num_partitions, threads_per_partition,
                     2 * max_cells_per_partition * sizeof(index_t), stream>>>(
            cells, modules, max_cells_per_partition,
            m_target_cells_per_partition, measurements_buffer,
            *num_measurements_device, cell_links);

    CUDA_ERROR_CHECK(hipGetLastError());

    // Copy number of measurements to host
    vecmem::unique_alloc_ptr<unsigned int> num_measurements_host =
        vecmem::make_unique_alloc<unsigned int>(
            (m_mr.host != nullptr) ? *(m_mr.host) : m_mr.main);
    CUDA_ERROR_CHECK(hipMemcpyAsync(
        num_measurements_host.get(), num_measurements_device.get(),
        sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
    m_stream.synchronize();

    spacepoint_collection_types::buffer spacepoints_buffer(
        *num_measurements_host, m_mr.main);
    m_copy.setup(spacepoints_buffer);

    // For the following kernel, we can now use whatever the desired number of
    // threads per block.
    auto spacepointsLocalSize = 1024;
    const unsigned int num_blocks =
        (*num_measurements_host + spacepointsLocalSize - 1) /
        spacepointsLocalSize;

    // Turn 2D measurements into 3D spacepoints
    kernels::form_spacepoints<<<num_blocks, spacepointsLocalSize, 0, stream>>>(
        measurements_buffer, modules, *num_measurements_host,
        spacepoints_buffer);

    CUDA_ERROR_CHECK(hipGetLastError());

    return {std::move(spacepoints_buffer), std::move(cell_links)};
}

}  // namespace traccc::cuda