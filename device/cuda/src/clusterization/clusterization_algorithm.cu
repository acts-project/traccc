#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// CUDA Library include(s).
#include "../utils/utils.hpp"
#include "traccc/cuda/clusterization/clusterization_algorithm.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// Project include(s)
#include "traccc/clusterization/device/aggregate_cluster.hpp"
#include "traccc/clusterization/device/form_spacepoints.hpp"
#include "traccc/clusterization/device/reduce_problem_cell.hpp"

// Vecmem include(s).
#include <vecmem/utils/copy.hpp>

// System include(s).
#include <algorithm>

namespace traccc::cuda {

namespace {
/// These indices in clusterization will only range from 0 to
/// max_cells_per_partition, so we only need a short.
using index_t = unsigned short;

static constexpr int TARGET_CELLS_PER_THREAD = 8;
static constexpr int MAX_CELLS_PER_THREAD = 12;
}  // namespace

namespace kernels {

/// Implementation of a FastSV algorithm with the following steps:
///   1) mix of stochastic and aggressive hooking
///   2) shortcutting
///
/// The implementation corresponds to an adapted versiion of Algorithm 3 of
/// the following paper:
/// https://www.sciencedirect.com/science/article/pii/S0743731520302689
///
/// @param[inout] f     array holding the parent cell ID for the current
/// iteration.
/// @param[inout] gf    array holding grandparent cell ID from the previous
/// iteration.
///                     This array only gets updated at the end of the iteration
///                     to prevent race conditions.
/// @param[in] adjc     The number of adjacent cells
/// @param[in] adjv     Vector of adjacent cells
/// @param[in] tid      The thread index
///
__device__ void fast_sv_1(index_t* f, index_t* gf,
                          unsigned char adjc[MAX_CELLS_PER_THREAD],
                          index_t adjv[MAX_CELLS_PER_THREAD][8], index_t tid,
                          const index_t blckDim) {
    /*
     * The algorithm finishes if an iteration leaves the arrays unchanged.
     * This varible will be set if a change is made, and dictates if another
     * loop is necessary.
     */
    bool gf_changed;

    do {
        /*
         * Reset the end-parameter to false, so we can set it to true if we
         * make a change to the gf array.
         */
        gf_changed = false;

        /*
         * The algorithm executes in a loop of three distinct parallel
         * stages. In this first one, a mix of stochastic and aggressive
         * hooking, we examine adjacent cells and copy their grand parents
         * cluster ID if it is lower than ours, essentially merging the two
         * together.
         */
        for (index_t tst = 0; tst < MAX_CELLS_PER_THREAD; ++tst) {
            const index_t cid = tst * blckDim + tid;

            __builtin_assume(adjc[tst] <= 8);
            for (unsigned char k = 0; k < adjc[tst]; ++k) {
                index_t q = gf[adjv[tst][k]];

                if (gf[cid] > q) {
                    f[f[cid]] = q;
                    f[cid] = q;
                }
            }
        }

        /*
         * Each stage in this algorithm must be preceded by a
         * synchronization barrier!
         */
        __syncthreads();

#pragma unroll
        for (index_t tst = 0; tst < MAX_CELLS_PER_THREAD; ++tst) {
            const index_t cid = tst * blckDim + tid;
            /*
             * The second stage is shortcutting, which is an optimisation that
             * allows us to look at any shortcuts in the cluster IDs that we
             * can merge without adjacency information.
             */
            if (f[cid] > gf[cid]) {
                f[cid] = gf[cid];
            }
        }

        /*
         * Synchronize before the final stage.
         */
        __syncthreads();

#pragma unroll
        for (index_t tst = 0; tst < MAX_CELLS_PER_THREAD; ++tst) {
            const index_t cid = tst * blckDim + tid;
            /*
             * Update the array for the next generation, keeping track of any
             * changes we make.
             */
            if (gf[cid] != f[f[cid]]) {
                gf[cid] = f[f[cid]];
                gf_changed = true;
            }
        }

        /*
         * To determine whether we need another iteration, we use block
         * voting mechanics. Each thread checks if it has made any changes
         * to the arrays, and votes. If any thread votes true, all threads
         * will return a true value and go to the next iteration. Only if
         * all threads return false will the loop exit.
         */
    } while (__syncthreads_or(gf_changed));
}

__global__ void ccl_kernel(
    const cell_collection_types::const_view cells_view,
    const cell_module_collection_types::const_view modules_view,
    const unsigned short max_cells_per_partition,
    const unsigned short target_cells_per_partition,
    alt_measurement_collection_types::view measurements_view,
    unsigned int& measurement_count,
    vecmem::data::vector_view<unsigned int> cell_links) {

    const index_t tid = threadIdx.x;
    const index_t blckDim = blockDim.x;

    const cell_collection_types::const_device cells_device(cells_view);
    const unsigned int num_cells = cells_device.size();
    __shared__ unsigned int start, end;
    /*
     * This variable will be used to write to the output later.
     */
    __shared__ unsigned int outi;

    /*
     * First, we determine the exact range of cells that is to be examined by
     * this block of threads. We start from an initial range determined by the
     * block index multiplied by the target number of cells per block. We then
     * shift both the start and the end of the block forward (to a later point
     * in the array); start and end may be moved different amounts.
     */
    if (tid == 0) {
        /*
         * Initialize shared variables.
         */
        start = blockIdx.x * target_cells_per_partition;
        assert(start < num_cells);
        end = std::min(num_cells, start + target_cells_per_partition);
        outi = 0;

        /*
         * Next, shift the starting point to a position further in the array;
         * the purpose of this is to ensure that we are not operating on any
         * cells that have been claimed by the previous block (if any).
         */
        while (start != 0 &&
               cells_device[start - 1].module_link ==
                   cells_device[start].module_link &&
               cells_device[start].channel1 <=
                   cells_device[start - 1].channel1 + 1) {
            ++start;
        }

        /*
         * Then, claim as many cells as we need past the naive end of the
         * current block to ensure that we do not end our partition on a cell
         * that is not a possible boundary!
         */
        while (end < num_cells &&
               cells_device[end - 1].module_link ==
                   cells_device[end].module_link &&
               cells_device[end].channel1 <=
                   cells_device[end - 1].channel1 + 1) {
            ++end;
        }
    }
    __syncthreads();

    const index_t size = end - start;
    assert(size <= max_cells_per_partition);

    // Check if any work needs to be done
    if (tid >= size) {
        return;
    }

    const cell_module_collection_types::const_device modules_device(
        modules_view);

    alt_measurement_collection_types::device measurements_device(
        measurements_view);

    // Vector of indices of the adjacent cells
    index_t adjv[MAX_CELLS_PER_THREAD][8];
    /*
     * The number of adjacent cells for each cell must start at zero, to
     * avoid uninitialized memory. adjv does not need to be zeroed, as
     * we will only access those values if adjc indicates that the value
     * is set.
     */
    // Number of adjacent cells
    unsigned char adjc[MAX_CELLS_PER_THREAD];

#pragma unroll
    for (index_t tst = 0; tst < MAX_CELLS_PER_THREAD; ++tst) {
        adjc[tst] = 0;
    }

    for (index_t tst = 0, cid; (cid = tst * blckDim + tid) < size; ++tst) {
        /*
         * Look for adjacent cells to the current one.
         */
        device::reduce_problem_cell(cells_device, cid, start, end, adjc[tst],
                                    adjv[tst]);
    }

    /*
     * These arrays are the meat of the pudding of this algorithm, and we
     * will constantly be writing and reading from them which is why we
     * declare them to be in the fast shared memory. Note that this places a
     * limit on the maximum contiguous activations per module, as the amount of
     * shared memory is limited. These could always be moved to global memory,
     * but the algorithm would be decidedly slower in that case.
     */
    extern __shared__ index_t shared_v[];
    index_t* f = &shared_v[0];
    index_t* f_next = &shared_v[max_cells_per_partition];

#pragma unroll
    for (index_t tst = 0; tst < MAX_CELLS_PER_THREAD; ++tst) {
        const index_t cid = tst * blckDim + tid;
        /*
         * At the start, the values of f and f_next should be equal to the
         * ID of the cell.
         */
        f[cid] = cid;
        f_next[cid] = cid;
    }

    /*
     * Now that the data has initialized, we synchronize again before we
     * move onto the actual processing part.
     */
    __syncthreads();

    /*
     * Run FastSV algorithm, which will update the father index to that of the
     * cell belonging to the same cluster with the lowest index.
     */
    fast_sv_1(f, f_next, adjc, adjv, tid, blckDim);

    __syncthreads();

    /*
     * Count the number of clusters by checking how many cells have
     * themself assigned as a parent.
     */
    for (index_t tst = 0, cid; (cid = tst * blckDim + tid) < size; ++tst) {
        if (f[cid] == cid) {
            atomicAdd(&outi, 1);
        }
    }

    __syncthreads();

    /*
     * Add the number of clusters of each thread block to the total
     * number of clusters. At the same time, a cluster id is retrieved
     * for the next data processing step.
     * Note that this might be not the same cluster as has been treated
     * previously. However, since each thread block spawns a the maximum
     * amount of threads per block, this has no sever implications.
     */
    if (tid == 0) {
        outi = atomicAdd(&measurement_count, outi);
    }

    __syncthreads();

    /*
     * Get the position to fill the measurements found in this thread group.
     */
    const unsigned int groupPos = outi;

    __syncthreads();

    if (tid == 0) {
        outi = 0;
    }

    __syncthreads();

    vecmem::data::vector_view<index_t> f_view(max_cells_per_partition, f);

    for (index_t tst = 0, cid; (cid = tst * blckDim + tid) < size; ++tst) {
        if (f[cid] == cid) {
            /*
             * If we are a cluster owner, atomically claim a position in the
             * output array which we can write to.
             */
            const unsigned int id = atomicAdd(&outi, 1);
            device::aggregate_cluster(
                cells_device, modules_device, f_view, start, end, cid,
                measurements_device[groupPos + id], cell_links, groupPos + id);
        }
    }
}

__global__ void form_spacepoints(
    alt_measurement_collection_types::const_view measurements_view,
    cell_module_collection_types::const_view modules_view,
    const unsigned int measurement_count,
    spacepoint_collection_types::view spacepoints_view) {

    device::form_spacepoints(threadIdx.x + blockIdx.x * blockDim.x,
                             measurements_view, modules_view, measurement_count,
                             spacepoints_view);
}

}  // namespace kernels

clusterization_algorithm::clusterization_algorithm(
    const traccc::memory_resource& mr, vecmem::copy& copy, stream& str,
    const unsigned short target_cells_per_partition)
    : m_mr(mr),
      m_copy(copy),
      m_stream(str),
      m_target_cells_per_partition(target_cells_per_partition) {}

clusterization_algorithm::output_type clusterization_algorithm::operator()(
    const cell_collection_types::const_view& cells,
    const cell_module_collection_types::const_view& modules) const {

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // Number of cells
    const cell_collection_types::view::size_type num_cells =
        m_copy.get_size(cells);

    // Create result object for the CCL kernel with size overestimation
    alt_measurement_collection_types::buffer measurements_buffer(num_cells,
                                                                 m_mr.main);

    // Counter for number of measurements
    vecmem::unique_alloc_ptr<unsigned int> num_measurements_device =
        vecmem::make_unique_alloc<unsigned int>(m_mr.main);
    CUDA_ERROR_CHECK(hipMemsetAsync(num_measurements_device.get(), 0,
                                     sizeof(unsigned int), stream));

    const unsigned short max_cells_per_partition =
        (m_target_cells_per_partition * MAX_CELLS_PER_THREAD +
         TARGET_CELLS_PER_THREAD - 1) /
        TARGET_CELLS_PER_THREAD;
    const unsigned int threads_per_partition =
        (m_target_cells_per_partition + TARGET_CELLS_PER_THREAD - 1) /
        TARGET_CELLS_PER_THREAD;
    const unsigned int num_partitions =
        (num_cells + m_target_cells_per_partition - 1) /
        m_target_cells_per_partition;

    // Create buffer for linking cells to their spacepoints.
    vecmem::data::vector_buffer<unsigned int> cell_links(num_cells, m_mr.main);

    // Launch ccl kernel. Each thread will handle a single cell.
    kernels::
        ccl_kernel<<<num_partitions, threads_per_partition,
                     2 * max_cells_per_partition * sizeof(index_t), stream>>>(
            cells, modules, max_cells_per_partition,
            m_target_cells_per_partition, measurements_buffer,
            *num_measurements_device, cell_links);

    CUDA_ERROR_CHECK(hipGetLastError());

    // Copy number of measurements to host
    vecmem::unique_alloc_ptr<unsigned int> num_measurements_host =
        vecmem::make_unique_alloc<unsigned int>(*(m_mr.host));
    CUDA_ERROR_CHECK(hipMemcpyAsync(
        num_measurements_host.get(), num_measurements_device.get(),
        sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
    m_stream.synchronize();

    spacepoint_collection_types::buffer spacepoints_buffer(
        *num_measurements_host, m_mr.main);

    // For the following kernel, we can now use whatever the desired number of
    // threads per block.
    auto spacepointsLocalSize = 1024;
    const unsigned int num_blocks =
        (*num_measurements_host + spacepointsLocalSize - 1) /
        spacepointsLocalSize;

    // Turn 2D measurements into 3D spacepoints
    kernels::form_spacepoints<<<num_blocks, spacepointsLocalSize, 0, stream>>>(
        measurements_buffer, modules, *num_measurements_host,
        spacepoints_buffer);

    CUDA_ERROR_CHECK(hipGetLastError());
    m_stream.synchronize();

    return {std::move(spacepoints_buffer), std::move(cell_links)};
}

}  // namespace traccc::cuda