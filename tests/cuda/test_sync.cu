#include "hip/hip_runtime.h"
/**
 * TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "../../device/cuda/src/utils/sync.cuh"

#include <gtest/gtest.h>

__global__ void testWarpIndexedBallotSyncBasicKernel(uint32_t *vts,
                                                     uint32_t *vis) {
    auto [vt, vi] =
        traccc::cuda::warp_indexed_ballot_sync(threadIdx.x % 2 == 0);

    vts[threadIdx.x] = vt;
    vis[threadIdx.x] = vi;
}

__global__ void testWarpIndexedBallotSyncWithExitKernel(uint32_t *vts,
                                                        uint32_t *vis) {
    if (threadIdx.x < 16) {
        return;
    }

    auto [vt, vi] =
        traccc::cuda::warp_indexed_ballot_sync(threadIdx.x % 2 == 0);

    vts[threadIdx.x] = vt;
    vis[threadIdx.x] = vi;
}

TEST(CUDASync, WarpIndexedBallotSyncBasic) {
    uint32_t *dev_vt = nullptr, *dev_vi = nullptr;
    uint32_t host_vt[32], host_vi[32];

    ASSERT_EQ(hipMalloc(&dev_vt, 32u * sizeof(uint32_t)), hipSuccess);
    ASSERT_EQ(hipMalloc(&dev_vi, 32u * sizeof(uint32_t)), hipSuccess);
    ASSERT_NE(dev_vt, nullptr);
    ASSERT_NE(dev_vi, nullptr);

    testWarpIndexedBallotSyncBasicKernel<<<1, 32u>>>(dev_vt, dev_vi);

    ASSERT_EQ(hipPeekAtLastError(), hipSuccess);

    ASSERT_EQ(hipMemcpy(host_vt, dev_vt, 32u * sizeof(uint32_t),
                         hipMemcpyDeviceToHost),
              hipSuccess);
    ASSERT_EQ(hipMemcpy(host_vi, dev_vi, 32u * sizeof(uint32_t),
                         hipMemcpyDeviceToHost),
              hipSuccess);

    for (uint32_t i = 0; i < 32u; ++i) {
        ASSERT_EQ(host_vt[i], 16u);
    }

    for (uint32_t i = 0; i < 16u; ++i) {
        ASSERT_EQ(host_vi[i * 2], i);
    }

    ASSERT_EQ(hipFree(dev_vt), hipSuccess);
    ASSERT_EQ(hipFree(dev_vi), hipSuccess);
}

TEST(CUDASync, WarpIndexedBallotSyncWithExit) {
    uint32_t *dev_vt = nullptr, *dev_vi = nullptr;
    uint32_t host_vt[32], host_vi[32];

    ASSERT_EQ(hipMalloc(&dev_vt, 32u * sizeof(uint32_t)), hipSuccess);
    ASSERT_EQ(hipMalloc(&dev_vi, 32u * sizeof(uint32_t)), hipSuccess);
    ASSERT_NE(dev_vt, nullptr);
    ASSERT_NE(dev_vi, nullptr);

    testWarpIndexedBallotSyncWithExitKernel<<<1, 32u>>>(dev_vt, dev_vi);

    ASSERT_EQ(hipPeekAtLastError(), hipSuccess);

    ASSERT_EQ(hipMemcpy(host_vt, dev_vt, 32u * sizeof(uint32_t),
                         hipMemcpyDeviceToHost),
              hipSuccess);
    ASSERT_EQ(hipMemcpy(host_vi, dev_vi, 32u * sizeof(uint32_t),
                         hipMemcpyDeviceToHost),
              hipSuccess);

    for (uint32_t i = 16; i < 32u; ++i) {
        ASSERT_EQ(host_vt[i], 8u);
    }

    for (uint32_t i = 0; i < 8u; ++i) {
        ASSERT_EQ(host_vi[i * 2 + 16], i);
    }

    ASSERT_EQ(hipFree(dev_vt), hipSuccess);
    ASSERT_EQ(hipFree(dev_vi), hipSuccess);
}
