#include "hip/hip_runtime.h"
/**
 * traccc library, part of the ACTS project (R&D line)
 *
 * (c) 2024 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <gtest/gtest.h>

#include <vecmem/memory/cuda/managed_memory_resource.hpp>
#include <vecmem/memory/unique_ptr.hpp>

#include "../../device/cuda/src/utils/cuda_error_handling.hpp"
#include "traccc/device/mutex.hpp"

__global__ void mutex_add_kernel(uint32_t *out, uint32_t *lock) {
    traccc::device::mutex m(*lock);

    if (threadIdx.x == 0) {
        m.lock();
        uint32_t tmp = *out;
        tmp += 1;
        *out = tmp;
        m.unlock();
    }
}

TEST(CUDAMutex, MassAdditionKernel) {
    vecmem::cuda::managed_memory_resource mr;

    vecmem::unique_alloc_ptr<uint32_t> out =
        vecmem::make_unique_alloc<uint32_t>(mr);
    vecmem::unique_alloc_ptr<uint32_t> lock =
        vecmem::make_unique_alloc<uint32_t>(mr);

    TRACCC_CUDA_ERROR_CHECK(hipMemset(lock.get(), 0, sizeof(uint32_t)));

    uint32_t n_blocks = 262144;
    uint32_t n_threads = 32;

    mutex_add_kernel<<<n_blocks, n_threads>>>(out.get(), lock.get());

    TRACCC_CUDA_ERROR_CHECK(hipGetLastError());
    TRACCC_CUDA_ERROR_CHECK(hipDeviceSynchronize());

    EXPECT_EQ(n_blocks, *out.get());
}
