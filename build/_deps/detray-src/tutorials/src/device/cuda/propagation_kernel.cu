#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2023 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/detail/cuda_definitions.hpp"
#include "propagation.hpp"

namespace detray::tutorial {

// Propagation configurations
inline constexpr detray::scalar path_limit{2.f *
                                           detray::unit<detray::scalar>::m};

/// Kernel that runs the entire propagation loop
__global__ void propagation_kernel(
    typename detray::tutorial::detector_host_t::view_type det_data,
    typename detray::tutorial::device_field_t::view_t field_data,
    const vecmem::data::vector_view<
        detray::free_track_parameters<detray::tutorial::algebra_t>>
        tracks_data) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // Setup device-side track collection
    vecmem::device_vector<
        detray::free_track_parameters<detray::tutorial::algebra_t>>
        tracks(tracks_data);

    if (gid >= tracks.size()) {
        return;
    }

    // Setup of the device-side detector
    detray::tutorial::detector_device_t det(det_data);

    // Create propagator from a stepper and a navigator
    propagation::config cfg{};
    cfg.navigation.search_window = {3u, 3u};
    detray::tutorial::propagator_t p{cfg};

    // Create actor states
    detray::pathlimit_aborter::state aborter_state{path_limit};
    detray::parameter_transporter<detray::tutorial::algebra_t>::state
        transporter_state{};
    detray::pointwise_material_interactor<detray::tutorial::algebra_t>::state
        interactor_state{};
    detray::parameter_resetter<detray::tutorial::algebra_t>::state
        resetter_state{};

    auto actor_states = detray::tie(aborter_state, transporter_state,
                                    interactor_state, resetter_state);

    // Create the propagator state for the track
    detray::tutorial::propagator_t::state state(tracks[gid], field_data, det);

    // Run propagation
    p.propagate(state, actor_states);
}

void propagation(typename detray::tutorial::detector_host_t::view_type det_data,
                 typename detray::tutorial::device_field_t::view_t field_data,
                 const vecmem::data::vector_view<
                     detray::free_track_parameters<detray::tutorial::algebra_t>>
                     tracks_data) {

    int thread_dim = 2 * WARP_SIZE;
    int block_dim = tracks_data.size() / thread_dim + 1;

    // run the tutorial kernel
    propagation_kernel<<<block_dim, thread_dim>>>(det_data, field_data,
                                                  tracks_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray::tutorial
