#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2021-2024 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/detail/cuda_definitions.hpp"

// Detray test include(s)
#include "mask_store_cuda_kernel.hpp"

namespace detray {

/// test kernel function to fill the output vector with is_inside function
/// return values
__global__ void mask_test_kernel(
    typename host_store_type::view_type store_data,
    vecmem::data::vector_view<point3> input_point3_data,
    vecmem::data::jagged_vector_view<int> output_data) {

    // get mask store
    device_store_type store(store_data);

    // get mask objects
    vecmem::device_vector<point3> input_point3(input_point3_data);
    vecmem::jagged_device_vector<int> output_device(output_data);

    const auto& rectangle_mask = store.get<mask_ids::e_rectangle2>()[0];
    const auto& trapezoid_mask = store.get<mask_ids::e_trapezoid2>()[0];
    const auto& ring_mask = store.get<mask_ids::e_ring2>()[0];
    const auto& cylinder_mask = store.get<mask_ids::e_cylinder2>()[0];
    const auto& annulus_mask = store.get<mask_ids::e_annulus2>()[0];

    // get device results from is_inside function
    for (int i = 0; i < n_points; i++) {
        output_device[0].push_back(rectangle_mask.is_inside(input_point3[i]));
        output_device[1].push_back(trapezoid_mask.is_inside(input_point3[i]));
        output_device[2].push_back(ring_mask.is_inside(input_point3[i]));
        output_device[3].push_back(cylinder_mask.is_inside(input_point3[i]));
        output_device[4].push_back(annulus_mask.is_inside(input_point3[i]));
    }
}

void mask_test(typename host_store_type::view_type store_data,
               vecmem::data::vector_view<point3> input_point3_data,
               vecmem::data::jagged_vector_view<int> output_data) {

    int block_dim = 1;
    int thread_dim = 1;

    // run the test kernel
    mask_test_kernel<<<block_dim, thread_dim>>>(store_data, input_point3_data,
                                                output_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray
