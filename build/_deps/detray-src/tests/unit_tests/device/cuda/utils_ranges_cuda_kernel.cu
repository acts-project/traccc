#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022-2024 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/detail/cuda_definitions.hpp"

// Detray test include(s)
#include "utils_ranges_cuda_kernel.hpp"

namespace detray {

//
// single
//
__global__ void single_kernel(const dindex value, dindex* result) {

    // single view should ony add the value 'i' once
    for (auto i : detray::views::single(value)) {
        *result += i;
    }
}

void test_single(const dindex value, dindex& check) {
    dindex* result{nullptr};
    hipMallocManaged(&result, sizeof(dindex));
    *result = 0u;

    // run the kernel
    single_kernel<<<1, 1>>>(value, result);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());

    check = *result;
    hipFree(result);
}

//
// pointer
//
__global__ void pointer_kernel(const dindex value, dindex* result) {

    // pointer view should ony add the value 'i' once
    for (auto i : detray::views::pointer(value)) {
        *result += i;
    }
}

void test_pointer(const dindex value, dindex& check) {
    dindex* result{nullptr};
    hipMallocManaged(&result, sizeof(dindex));
    *result = 0u;

    // run the kernel
    pointer_kernel<<<1, 1>>>(value, result);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());

    check = *result;
    hipFree(result);
}

//
// iota
//
__global__ void iota_kernel(const darray<dindex, 2> range,
                            vecmem::data::vector_view<dindex> check_data) {

    vecmem::device_vector<dindex> check(check_data);

    for (auto i : detray::views::iota(range)) {
        check.push_back(i);
    }
}

void test_iota(const darray<dindex, 2> range,
               vecmem::data::vector_view<dindex> check_data) {

    // run the kernel
    iota_kernel<<<1, 1>>>(range, check_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

//
// cartesian product
//
__global__ void cartesian_product_kernel(
    const darray<dindex, 2> range1, const darray<dindex, 2> range2,
    const darray<dindex, 2> range3,
    vecmem::data::vector_view<std::tuple<dindex, dindex, dindex>> check_data) {

    vecmem::device_vector<std::tuple<dindex, dindex, dindex>> check(check_data);

    auto seq1 = detray::views::iota(range1);
    auto seq2 = detray::views::iota(range2);
    auto seq3 = detray::views::iota(range3);

    for (const auto [i, j, k] : detray::views::cartesian_product(
             std::move(seq1), std::move(seq2), std::move(seq3))) {
        check.emplace_back(i, j, k);
    }
}

void test_cartesian_product(
    const darray<dindex, 2> range1, const darray<dindex, 2> range2,
    const darray<dindex, 2> range3,
    vecmem::data::vector_view<std::tuple<dindex, dindex, dindex>> check_data) {

    // run the kernel
    cartesian_product_kernel<<<1, 1>>>(range1, range2, range3, check_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

//
// enumerate
//
__global__ void enumerate_kernel(
    vecmem::data::vector_view<uint_holder> seq_data,
    vecmem::data::vector_view<dindex> check_idx_data,
    vecmem::data::vector_view<dindex> check_value_data) {

    vecmem::device_vector<uint_holder> seq(seq_data);
    vecmem::device_vector<dindex> check_idx(check_idx_data);
    vecmem::device_vector<dindex> check_value(check_value_data);

    for (auto [i, v] : detray::views::enumerate(seq)) {
        check_idx.push_back(i);
        check_value.push_back(v.ui);
    }
}

void test_enumerate(vecmem::data::vector_view<uint_holder> seq_data,
                    vecmem::data::vector_view<dindex> check_idx_data,
                    vecmem::data::vector_view<dindex> check_value_data) {

    // run the kernel
    enumerate_kernel<<<1, 1>>>(seq_data, check_idx_data, check_value_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

//
// pick
//
__global__ void pick_kernel(
    vecmem::data::vector_view<uint_holder> seq_data,
    vecmem::data::vector_view<dindex> idx_data,
    vecmem::data::vector_view<dindex> check_idx_data,
    vecmem::data::vector_view<dindex> check_value_data) {

    vecmem::device_vector<uint_holder> seq(seq_data);
    vecmem::device_vector<dindex> idx(idx_data);
    vecmem::device_vector<dindex> check_idx(check_idx_data);
    vecmem::device_vector<dindex> check_value(check_value_data);

    for (auto [i, v] : detray::views::pick(seq, idx)) {
        check_idx.push_back(i);
        check_value.push_back(v.ui);
    }
}

void test_pick(vecmem::data::vector_view<uint_holder> seq_data,
               vecmem::data::vector_view<dindex> idx_data,
               vecmem::data::vector_view<dindex> check_idx_data,
               vecmem::data::vector_view<dindex> check_value_data) {

    // run the kernel
    pick_kernel<<<1, 1>>>(seq_data, idx_data, check_idx_data, check_value_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

//
// join
//
__global__ void join_kernel(
    vecmem::data::vector_view<uint_holder> seq_data_1,
    vecmem::data::vector_view<uint_holder> seq_data_2,
    vecmem::data::vector_view<dindex> check_value_data) {

    vecmem::device_vector<uint_holder> seq_1(seq_data_1);
    vecmem::device_vector<uint_holder> seq_2(seq_data_2);
    vecmem::device_vector<dindex> check_value(check_value_data);
    std::array<vecmem::device_vector<uint_holder>, 2> vectors{seq_1, seq_2};

    for (auto v : detray::views::join(vectors)) {
        check_value.push_back(v.ui);
    }
}

void test_join(vecmem::data::vector_view<uint_holder> seq_data_1,
               vecmem::data::vector_view<uint_holder> seq_data_2,
               vecmem::data::vector_view<dindex> check_value_data) {

    // run the kernel
    join_kernel<<<1, 1>>>(seq_data_1, seq_data_2, check_value_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

//
// static_join
//
__global__ void static_join_kernel(
    vecmem::data::vector_view<uint_holder> seq_data_1,
    vecmem::data::vector_view<uint_holder> seq_data_2,
    vecmem::data::vector_view<dindex> check_value_data) {

    vecmem::device_vector<uint_holder> seq_1(seq_data_1);
    vecmem::device_vector<uint_holder> seq_2(seq_data_2);
    vecmem::device_vector<dindex> check_value(check_value_data);

    for (auto v : detray::views::static_join(seq_1, seq_2)) {
        check_value.push_back(v.ui);
    }
}

void test_static_join(vecmem::data::vector_view<uint_holder> seq_data_1,
                      vecmem::data::vector_view<uint_holder> seq_data_2,
                      vecmem::data::vector_view<dindex> check_value_data) {

    // run the kernel
    static_join_kernel<<<1, 1>>>(seq_data_1, seq_data_2, check_value_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

//
// subrange
//
__global__ void subrange_kernel(vecmem::data::vector_view<int> seq_data,
                                vecmem::data::vector_view<int> check_value_data,
                                const std::size_t begin,
                                const std::size_t end) {

    vecmem::device_vector<int> seq(seq_data);
    vecmem::device_vector<int> check(check_value_data);

    for (const auto& v : detray::ranges::subrange(
             seq, std::array<std::size_t, 2>{begin, end})) {
        check.push_back(v);
    }
}

void test_subrange(vecmem::data::vector_view<int> seq_data,
                   vecmem::data::vector_view<int> check_value_data,
                   const std::size_t begin, const std::size_t end) {

    // run the kernel
    subrange_kernel<<<1, 1>>>(seq_data, check_value_data, begin, end);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray
