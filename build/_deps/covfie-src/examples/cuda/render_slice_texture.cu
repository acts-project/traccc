#include "hip/hip_runtime.h"
/*
 * SPDX-PackageName: "covfie, a part of the ACTS project"
 * SPDX-FileCopyrightText: 2022 CERN
 * SPDX-License-Identifier: MPL-2.0
 */

#include <chrono>
#include <fstream>
#include <iostream>

#include <boost/log/trivial.hpp>
#include <boost/program_options.hpp>

#include <covfie/core/backend/transformer/affine.hpp>
#include <covfie/core/backend/transformer/linear.hpp>
#include <covfie/core/backend/transformer/strided.hpp>
#include <covfie/core/field.hpp>
#include <covfie/core/parameter_pack.hpp>
#include <covfie/cuda/backend/primitive/cuda_device_array.hpp>
#include <covfie/cuda/backend/primitive/cuda_texture.hpp>
#include <covfie/cuda/error_check.hpp>

#include "bitmap.hpp"

using cpu_field_t = covfie::field<
    covfie::backend::affine<covfie::backend::linear<covfie::backend::strided<
        covfie::vector::size3,
        covfie::backend::array<covfie::vector::float3>>>>>;

using cuda_field_t = covfie::field<covfie::backend::affine<
    covfie::backend::
        cuda_texture<covfie::vector::float3, covfie::vector::float3>>>;

void parse_opts(
    int argc, char * argv[], boost::program_options::variables_map & vm
)
{
    boost::program_options::options_description opts("general options");

    opts.add_options()("help", "produce help message")(
        "input,i",
        boost::program_options::value<std::string>()->required(),
        "input vector field to read"
    )("output,o",
      boost::program_options::value<std::string>()->required(),
      "output bitmap image to write"
    )("height,h",
      boost::program_options::value<unsigned int>()->default_value(1024),
      "height of the output image"
    )("width,w",
      boost::program_options::value<unsigned int>()->default_value(1024),
      "width of the output image"
    )("z",
      boost::program_options::value<float>()->default_value(0.f),
      "z-index of the vector field slice");

    boost::program_options::parsed_options parsed =
        boost::program_options::command_line_parser(argc, argv)
            .options(opts)
            .run();

    boost::program_options::store(parsed, vm);

    if (vm.count("help")) {
        std::cout << opts << std::endl;
        std::exit(0);
    }

    try {
        boost::program_options::notify(vm);
    } catch (boost::program_options::required_option & e) {
        BOOST_LOG_TRIVIAL(fatal) << e.what();
        std::exit(1);
    }
}

template <typename field_t>
__global__ void render(
    typename field_t::view_t vf,
    unsigned char * out,
    unsigned int width,
    unsigned int height,
    float z
)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        float fx = x / static_cast<float>(width);
        float fy = y / static_cast<float>(height);

        typename field_t::output_t p =
            vf.at(fx * 20000.f - 10000.f, fy * 20000.f - 10000.f, z);
        out[height * x + y] = static_cast<unsigned char>(std::lround(
            255.f * fmin(
                        ::sqrtf(
                            ::powf(p[0] / 0.000299792458f, 2.f) +
                            ::powf(p[1] / 0.000299792458f, 2.f) +
                            ::powf(p[2] / 0.000299792458f, 2.f)
                        ),
                        1.0f
                    )
        ));
    }
}

int main(int argc, char ** argv)
{
    boost::program_options::variables_map vm;
    parse_opts(argc, argv, vm);

    unsigned int width = vm["width"].as<unsigned int>();
    unsigned int height = vm["height"].as<unsigned int>();

    BOOST_LOG_TRIVIAL(info) << "Welcome to the covfie CUDA field renderer!";
    BOOST_LOG_TRIVIAL(info) << "Using magnetic field file \""
                            << vm["input"].as<std::string>() << "\"";
    BOOST_LOG_TRIVIAL(info) << "Starting read of input file...";

    std::ifstream ifs(vm["input"].as<std::string>(), std::ifstream::binary);

    if (!ifs.good()) {
        BOOST_LOG_TRIVIAL(fatal) << "Failed to open input file "
                                 << vm["input"].as<std::string>() << "!";
        std::exit(1);
    }

    cpu_field_t f(ifs);
    ifs.close();

    BOOST_LOG_TRIVIAL(info) << "Casting magnetic field into CUDA array...";

    cuda_field_t nf(covfie::make_parameter_pack(
        f.backend().get_configuration(), f.backend().get_backend().get_backend()
    ));

    BOOST_LOG_TRIVIAL(info) << "Allocating device memory for output image...";

    unsigned char * img_device;

    cudaErrorCheck(hipMalloc(
        reinterpret_cast<void **>(&img_device),
        width * height * sizeof(unsigned char)
    ));

    BOOST_LOG_TRIVIAL(info) << "Rendering magnetic field strength to image...";

    std::chrono::high_resolution_clock::time_point t1 =
        std::chrono::high_resolution_clock::now();

    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        width / dimBlock.x + (width % dimBlock.x != 0 ? 1 : 0),
        height / dimBlock.y + (height % dimBlock.y != 0 ? 1 : 0)
    );

    render<decltype(nf)><<<dimGrid, dimBlock>>>(
        nf, img_device, width, height, vm["z"].as<float>()
    );

    cudaErrorCheck(hipGetLastError());
    cudaErrorCheck(hipDeviceSynchronize());

    std::chrono::high_resolution_clock::time_point t2 =
        std::chrono::high_resolution_clock::now();

    BOOST_LOG_TRIVIAL(info
    ) << "Rendering took "
      << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count()
      << "us." << std::endl;

    BOOST_LOG_TRIVIAL(info) << "Allocating host memory for output image...";

    std::unique_ptr<unsigned char[]> img_host =
        std::make_unique<unsigned char[]>(width * height);

    BOOST_LOG_TRIVIAL(info) << "Copying image from device to host...";

    cudaErrorCheck(hipMemcpy(
        img_host.get(),
        img_device,
        width * height * sizeof(unsigned char),
        hipMemcpyDeviceToHost
    ));

    BOOST_LOG_TRIVIAL(info) << "Deallocating device memory...";

    cudaErrorCheck(hipFree(img_device));

    BOOST_LOG_TRIVIAL(info) << "Saving image to file \""
                            << vm["output"].as<std::string>() << "\"...";

    render_bitmap(
        img_host.get(), width, height, vm["output"].as<std::string>()
    );

    BOOST_LOG_TRIVIAL(info) << "Procedure complete, goodbye!";
}
